#include <iostream>
#include <float.h> 
#include <hipblas.h>

#include <fstream>
#include <sstream>
#include <vector>
#include <cstdlib>
#include <stdio.h>
#include <parallel/algorithm>  
#include <unordered_map>
#include <random>
#include <iostream>
#include <cstring> 
#include <chrono> 
#include <hip/hip_runtime.h>
#include <string>
#include <omp.h>
#include <unordered_set>
#include <set>
#include <map>
#include <algorithm>
#include <numeric>
#include <utility>
#include <iomanip>
#include <chrono>

using std::count;
using std::cout;
using std::endl;

// nvcc -c kernel_copy.cu -o kernel.o
// CUDA核函数，执行向量加法
// __device__ std::vector<double> decp_data;
__device__ double* decp_data;
__device__ double* decp_data_copy ;
__device__ int directions1[36] =  {0,1,0,0,-1,0,1,0,0,-1,0,0,-1,1,0,1,-1,0,0,0, -1,  0,-1, 1, 0,0, 1,  0,1, -1,  -1,0, 1,   1, 0,-1};
__device__ int width;
__device__ int height;
__device__ int cnt = 0;
__device__ int depth;
__device__ int num;
__device__ int* adjacency;
__device__ int* all_max; 
__device__ int* all_min;
__device__ int* all_p_max; 
__device__ int* all_p_min;
__device__ int* unsigned_n;
__device__ int count_max;
__device__ int count_min;

__device__ int count_f_max;
__device__ int count_f_min;
__device__ int count_p_max;
__device__ int count_p_min;
__device__ int* maxi;
__device__ int* mini;
__device__ double bound;
__device__ int* or_maxi;
__device__ int* or_mini;
__device__ int* or_label;
__device__ int* dec_label;
__device__ int* lowgradientindices;
__device__ double* input_data;
__device__ int* de_direction_as;
__device__ int* de_direction_ds;
__device__ int maxNeighbors = 12;

__device__ int direction_to_index_mapping[12][3] = {{0,1,0},{0,-1,0},{1,0,0},{-1,0,0},{-1,1,0},{1,-1,0},{0,0, -1},  {0,-1, 1}, {0,0, 1},  {0,1, -1},  {-1,0, 1},   {1, 0,-1}};   


__device__ int getDirection(int x, int y, int z){
    
    for (int i = 0; i < 12; ++i) {
        if (direction_to_index_mapping[i][0] == x && direction_to_index_mapping[i][1] == y && direction_to_index_mapping[i][2] == z) {
            return i+1;  
        }
    }
    return -1;  

// 26302898,3378820
// 27930227,32438238
}
__device__ int from_direction_to_index1(int cur, int direc){
    
    if (direc==-1) return cur;
    int x = cur % width;
    int y = (cur / width) % height;
    int z = (cur/(width * height))%depth;
    // printf("%d %d\n", row, rank1);
    if (direc >= 1 && direc <= 12) {
        int delta_row = direction_to_index_mapping[direc-1][0];
        int delta_col = direction_to_index_mapping[direc-1][1];
        int delta_dep = direction_to_index_mapping[direc-1][2];
        
        
        int next_row = x + delta_row;
        int next_col = y + delta_col;
        int next_dep = z + delta_dep;
        // printf("%d \n", next_row * width + next_col);
        // return next_row * width + next_col + next_dep* (height * width);
        return next_row + next_col * width + next_dep* (height * width);
    }
    else {
        return -1;
    }
    // return 0;
};

__device__ void find_direction2 (int type, int index){
    double *data;
    int *direction_as;
    int *direction_ds;
    if(type==0){
        data = decp_data;
        direction_as = de_direction_as;
        direction_ds = de_direction_ds;
    }
    else{
        data = input_data;
        direction_as = or_maxi;
        direction_ds = or_mini;
    }
    
    double mini = 0;
    
    
    // std::vector<int> indexs = adjacency[index];
    int largetst_index = index;
    
    
        
    for(int j =0;j<12;++j){
        int i = adjacency[index*12+j];
        
        if(i==-1){
            break;
        }
        if(lowgradientindices[i]==1){
            continue;
        }
        if((data[i]>data[largetst_index] or (data[i]==data[largetst_index] and i>largetst_index))){
            mini = data[i]-data[index];
            
            largetst_index = i;
            // }
            
        };
    };
    int row_l = (largetst_index / (height)) % width;
    int row_i = (index / (height)) % width;
    
    int col_diff = row_l - row_i;
    int row_diff = (largetst_index % height) - (index % height);

    int dep_diff = (largetst_index /(width * height))%depth - (index /(width * height))%depth;
    direction_as[index] = getDirection(row_diff, col_diff,dep_diff);
    // if(index==8058 and type==0){
    //     printf("%d %d \n" ,direction_ds[index],or_mini[index]);
    // }
    
    

    mini = 0;
    largetst_index = index;
    for(int j =0;j<12;++j){
        int i = adjacency[index*12+j];
        
        if(i==-1){
            break;
        }
        if(lowgradientindices[i]==1){
            continue;
        }
        // if(i==8186 and index==8058 and type==0){
        //     printf("%.20f %.20f\n",data[i]-data[index],data[8057]-data[index]);
        //     // cout<<data[i]<<", "<<data[index]<<", "<<data[8057]<<endl;
        // }
        if((data[i]<data[largetst_index] or (data[i]==data[largetst_index] and i<largetst_index))){
            mini = data[i]-data[index];
            
            largetst_index = i;

            
        };
    };
    
    row_l = (largetst_index / (height)) % width;
    row_i = (index / (height)) % width;
    
    col_diff = row_l - row_i;
    row_diff = (largetst_index % height) - (index % height);

    dep_diff = (largetst_index /(width * height))%depth - (index /(width * height))%depth;
    // row_l = (largetst_index % (height * width)) / width;
    // row_i = (index % (height * width)) / width;
    
    // row_diff = row_l - row_i;
    // col_diff = (largetst_index % width) - (index % width);

    // dep_diff = (largetst_index /(width * height)) - (index /(width * height));
    
    direction_ds[index] = getDirection(row_diff, col_diff,dep_diff);
    
    
    
}
__global__ void find_direction (int type=0){
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if(index>=num or lowgradientindices[index]==1){
        return;
    }
    
    double *data;
    int *direction_as;
    int *direction_ds;
    if(type==0){
        data = decp_data;
        direction_as = de_direction_as;
        direction_ds = de_direction_ds;
    }
    else{
        data = input_data;
        direction_as = or_maxi;
        direction_ds = or_mini;
    }
    
    double mini = 0;
    
    
    // std::vector<int> indexs = adjacency[index];
    int largetst_index = index;
    
    
        
    for(int j =0;j<12;++j){
        int i = adjacency[index*12+j];
        
        if(i==-1){
            continue;
        }
        if(lowgradientindices[i]==1){
            continue;
        }
        if((data[i]>data[largetst_index] or (data[i]==data[largetst_index] and i>largetst_index))){
            mini = data[i]-data[index];
            
            largetst_index = i;
            // }
            
        };
    };
    // int row_l = (largetst_index % (height * width)) / width;
    // int row_i = (index % (height * width)) / width;
    
    // int row_diff = row_l - row_i;
    // int col_diff = (largetst_index % width) - (index % width);

    // int dep_diff = (largetst_index /(width * height)) - (index /(width * height));
    // int x_l = (largetst_index / (height)) % width;
    // int x_i = (index / (height)) % width;
    int y_diff = (largetst_index / (width)) % height - (index / (width)) % height;
    // int y_diff = row_l - row_i;
    int x_diff = (largetst_index % width) - (index % width);

    int z_diff = (largetst_index /(width * height)) % depth - (index /(width * height)) % depth;
    direction_as[index] = getDirection(x_diff, y_diff,z_diff);
    
    // if(index==24654784 and type==0){
        
    //     printf("值：");
    //     printf("%d %d %d\n",row_diff, col_diff,dep_diff);
    //     printf("%d %d \n", largetst_index % 750, index % 750);
    //     // printf("%f %f \n" ,decp_data[index],input_data[index]);
    //     // for(int i=0;i<12;i++){
    //     //     int j = adjacency[index*12+i];
    //     //     if(j==-1){
    //     //         break;
    //     //     }
    //     //     printf("%f %f \n" ,decp_data[j],input_data[j]);
    //     // }
        
    // }
    
    

    mini = 0;
    largetst_index = index;
    for(int j =0;j<12;++j){
        int i = adjacency[index*12+j];
        
        if(i==-1){
            break;
        }
        if(lowgradientindices[i]==1){
            continue;
        }
        // if(i==8186 and index==8058 and type==0){
        //     printf("%.20f %.20f\n",data[i]-data[index],data[8057]-data[index]);
        //     // cout<<data[i]<<", "<<data[index]<<", "<<data[8057]<<endl;
        // }
        if((data[i]<data[largetst_index] or (data[i]==data[largetst_index] and i<largetst_index))){
            mini = data[i]-data[index];
            
            largetst_index = i;

            
        };
    };
    
    
    // row_l = (largetst_index % (height * width)) / width;
    // row_i = (index % (height * width)) / width;
    
    // row_diff = row_l - row_i;
    // col_diff = (largetst_index % width) - (index % width);

    // dep_diff = (largetst_index /(width * height)) - (index /(width * height));
    y_diff = (largetst_index / (width)) % height - (index / (width)) % height;
    // int y_diff = row_l - row_i;
    x_diff = (largetst_index % width) - (index % width);

    z_diff = (largetst_index /(width * height)) % depth - (index /(width * height)) % depth;
    // direction_as[index] = getDirection(x_diff, y_diff,z_diff);
    direction_ds[index] = getDirection(x_diff, y_diff,z_diff);
    
    
    
    
    
    return;

};
__global__ void checkElementKernel(int* array, int size, int target, bool* result) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < size) {
        if (array[idx] == target) {
            *result = true;
        }
    }
}

__global__ void iscriticle(){
        
        int i = threadIdx.x + blockIdx.x * blockDim.x;
        
        if(i>=num or lowgradientindices[i]==1){
            
            return;
        }
        
        bool is_maxima = true;
        bool is_minima = true;
        
        for (int index=0;index<12;index++) {
            int j = adjacency[i*12+index];
            if(j==-1){
                break;
            }
            if(lowgradientindices[j]==1){
                continue;
            }
            
                
            if (decp_data[j] > decp_data[i]) {
                
                is_maxima = false;
                
                break;
            }
            else if(decp_data[j] == decp_data[i] and j>i){
                is_maxima = false;
                break;
            }
        }
        for (int index=0;index< 12;index++) {
            int j = adjacency[i*12+index];
            if(j==-1){
                break;
            }
            if(lowgradientindices[j]==1){
                    continue;
            }
            
            if (decp_data[j] < decp_data[i]) {
                is_minima = false;
                break;
            }
            else if(decp_data[j] == decp_data[i] and j<i){
                is_minima = false;
                break;
            }
        }
        
        
        if((is_maxima && or_maxi[i]!=-1) or (!is_maxima && or_maxi[i]==-1)){
            int idx_fp_max = atomicAdd(&count_f_max, 1);
            // if(i==6345199){
            //     printf("%d %d \n",is_maxima,or_maxi[i]);
            // }
            all_max[idx_fp_max] = i;
            
        }
        
        else if ((is_minima && or_mini[i]!=-1) or (!is_minima && or_mini[i]==-1)) {
            int idx_fp_min = atomicAdd(&count_f_min, 1);// in one instruction
            
            all_min[idx_fp_min] = i;
            
        } 
        
       
        
}

__global__ void get_wrong_index_path1(){

    
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    
    if(i>=num or lowgradientindices[i]==1){
        
        return;
    }
    if (or_label[i * 2 + 1] != dec_label[i * 2 + 1] || or_label[i * 2] != dec_label[i * 2]) {
        atomicAdd(&cnt, 1);
    }
    if (or_label[i * 2 + 1] != dec_label[i * 2 + 1]) {
        int idx_fp_max = atomicAdd(&count_p_max, 1);
        // printf("%d %d %d\n",i,or_label[i * 2 + 1],dec_label[i * 2 + 1]);
        all_p_max[idx_fp_max] = i;
            
    }
    if (or_label[i * 2] != dec_label[i * 2]) {
        int idx_fp_min = atomicAdd(&count_p_min, 1);
        all_p_min[idx_fp_min] = i;
        
    }
    
    

    return;
};

__global__ void freeDeviceMemory() {
    // 释放 decp_data 指向的内存
    if (decp_data != nullptr) {
        delete[] decp_data;
        decp_data = nullptr;  // 避免野指针
    }
} 
__global__ void freeDeviceMemory1() {
    // 释放 decp_data 指向的内存
    if (de_direction_as != nullptr) {
        delete[] de_direction_as;
        de_direction_as = nullptr;  // 避免野指针
    }
    if (de_direction_ds != nullptr) {
        delete[] de_direction_ds;
        de_direction_ds = nullptr;  // 避免野指针
    }
}
__global__ void computeAdjacency() {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i < num and lowgradientindices[i]==0) {
        
        int y = (i / (width)) % height; // Get the x coordinate
        int x = i % width; // Get the y coordinate
        int z = (i / (width * height)) % depth;
        int neighborIdx = 0;
        
        for (int d = 0; d < 12; d++) {
            
            int dirX = directions1[d * 3];     
            int dirY = directions1[d * 3 + 1]; 
            int dirZ = directions1[d * 3 + 2]; 
            int newX = x + dirX;
            int newY = y + dirY;
            int newZ = z + dirZ;
            int r = newX + newY * width + newZ* (height * width); // Calculate the index of the adjacent vertex
            // if(lowgradientindices[r]==1){
            //     continue;
            // }
            if (newX >= 0 && newX < width && newY >= 0 && newY < height && r < width*height*depth && newZ<depth && newZ>=0 && lowgradientindices[r]==0) {
                
                adjacency[i * maxNeighbors + neighborIdx] = r;
                neighborIdx++;

            }
        }

        // Fill the remaining slots with -1 or another placeholder value
        
        for (int j = neighborIdx; j < maxNeighbors; ++j) {
            adjacency[i * maxNeighbors + j] = -1;
        }
    }
}

__global__ void allocateDeviceMemory() {
    if (threadIdx.x == 0) { // 仅在一个线程上执行
        // printf("%d %d \n", threadIdx.x,num );
        all_max = new int[num];
        
        all_min = new int[num];
    }
    return;
}


__global__ void fix_maxi_critical1(int direction){
    int index_f = blockIdx.x * blockDim.x + threadIdx.x;
    
    
        
    
    if (direction == 0 && index_f<count_f_max && lowgradientindices[all_max[index_f]]==0){
        
        int index = all_max[index_f];
        // printf("%d\n",index);
        if (or_maxi[index]!=-1){
            // printf("%d\n",index);
            // find_direction2(1,index);
            int next_vertex = from_direction_to_index1(index,or_maxi[index]);
            
            int smallest_vertex = next_vertex;
            double threshold = -DBL_MAX;
            
            
            for(int j=0;j<12;j++){
                int i = adjacency[index*12+j];
                if(i==-1){
                    break;
                }
                if(lowgradientindices[i]==1){
                    continue;
                }
                
                if(input_data[i]<input_data[index] and input_data[i]>threshold and i!=next_vertex){
                    smallest_vertex = i;
                    threshold = input_data[i];
                }
            }
            
            threshold = decp_data[smallest_vertex];
            // 对的
            // d是把index还要降低
            // 如果是tthresh的话，那它的下限就是：input_data[index]-(abs(inaput_data[index]-decp_data[index]))
            // 之前的
            // double diff = (bound - (input_data[next_vertex]-decp_data[index]))/2.0;
            // double d = (decp_data[index] - input_data[index] + bound )/2.0;
            // // double d = (decp_data[index]-(input_data[index]-(abs(input_data[index]-decp_data_copy[index]))))/2.0;
            // double d1 = ((input_data[next_vertex] + bound) - decp_data[next_vertex])/2.0;
            // double diff1 = (bound - (input_data[next_vertex]-decp_data[index]))/2.0;
            double diff = (bound - (input_data[next_vertex]-decp_data[index]))/2.0;
            double d = (decp_data[index] - input_data[index] + bound )/2.0;
            double d1 = ((input_data[next_vertex] + bound) - decp_data[next_vertex])/2.0;
            double diff1 = (bound - (input_data[next_vertex]-decp_data[index]))/2.0;
            // double diff = ((input_data[next_vertex]-decp_data_copy[next_vertex]) - (input_data[next_vertex]-decp_data[index]))/2.0;
            // double d = (decp_data[index] - input_data[index] + (input_data[index]-decp_data_copy[index]))/2.0;
            // double d1 = ((input_data[next_vertex] + (input_data[next_vertex]-decp_data_copy[next_vertex])) - decp_data[next_vertex])/2.0;
            // double diff1 = ((input_data[next_vertex]-decp_data_copy[next_vertex]) - (input_data[next_vertex]-decp_data[index]))/2.0;
            // if(count_f_max==1){
            //     printf("改变后");
            //     printf("%d, %.17lf\n", index, decp_data[index]);
            //     printf("%d %.17lf\n", next_vertex, decp_data[next_vertex]);
            //     printf("%.17lf %.17lf \n",d1, d);
            //     printf("%.17lf %.17lf \n",input_data[index], input_data[next_vertex]);
            //     // cout<<"index: "<<index<<", "<<decp_data[index]<<endl;
            //     // cout<<"next_vertex: "<<next_vertex<<","<<decp_data[next_vertex]<<endl;
            //     // cout<<"smallest_vertex: "<<smallest_vertex<<", "<<decp_data[smallest_vertex]<<endl;
            //     // cout<<"diff: "<<d<<","<<"d: "<<d<<endl;
            // }
            if(decp_data[index]<decp_data[next_vertex] or (decp_data[index]==decp_data[next_vertex] and index<next_vertex)){
                
                de_direction_as[index]=or_maxi[index];
            
                return;
            }
            
            if(d>=1e-16 ){
                
                if(decp_data[index]==decp_data[next_vertex])
                    {
                        
                        while(abs(input_data[index]-decp_data[index]+d)>bound and d>=2e-16){
                            d/=2;
                        }
                        if (abs(input_data[index]-decp_data[index]+d)<=bound){
                            decp_data[index] -= d;
                        }
                    }
                else{
                    if(decp_data[index]>=decp_data[next_vertex]){
                        
                        while(abs(input_data[index]-decp_data[index]+d)>bound and d>=2e-16){
                                d/=2;
                        }
                        
                        if(decp_data[index]>=threshold and threshold<=decp_data[next_vertex]){
                            
                            while(decp_data[index] - d < threshold and d>=2e-16)
                            {
                                d/=2;
                            }
                            
                            
                        }
                        // else if(threshold>decp_data[next_vertex]){
                            
                            
                        //     double diff2 = (bound-(input_data[smallest_vertex]-decp_data[smallest_vertex]))/64;
                            
                        //     if(diff2>=1e-16){
                        //         while(abs(input_data[smallest_vertex]-decp_data[smallest_vertex]+diff2)>bound and diff2>=2e-16 and decp_data[smallest_vertex]-diff2>decp_data[next_vertex]){
                                    
                        //             diff2/=2;
                        //         }
                                
                        //         if(abs(input_data[smallest_vertex]-decp_data[smallest_vertex]+diff2)<=bound){
                        //             if(smallest_vertex==66783){cout<<"在这里11."<<endl;}
                        //             decp_data[smallest_vertex]-=diff2;
                        //             // if(index==97) cout<<"处理97的时候: "<<decp_data[next_vertex]<<", "<<decp_data[index]<<endl;
                        //         }
                                
                                
                        //     }
                            
                        // }

                        if(abs(input_data[index]-(decp_data[index]-d))<=bound and decp_data[index]>=decp_data[next_vertex] and d>=1e-16){
                            // if(index==1620477){
                            //     // cout<<"next_vertex: "<<decp_data[next_vertex]<<endl;
                            //     // cout<<"smallest_vertex: "<<decp_data[smallest_vertex]<<endl;
                            //     cout<<"before index: "<<decp_data[index]<<endl;
                                
                            // }
                            
                            decp_data[index] -= d;
                            
                            
                                            
                        }
                        // else if(abs(input_data[next_vertex]-(decp_data[next_vertex]+d1))<=bound and decp_data[index]>=decp_data[next_vertex] and d1>0){
                        //     // if(index==1620477){
                        //     //     // cout<<"next_vertex: "<<decp_data[next_vertex]<<endl;
                        //     //     // cout<<"smallest_vertex: "<<decp_data[smallest_vertex]<<endl;
                        //     //     cout<<"before index: "<<decp_data[index]<<endl;
                                
                        //     // }
                            
                        //     decp_data[next_vertex] += d1;
                            
                            
                                            
                        // }
                        
                        // if(count_f_max==1){
                        //     printf("改变后dd");
                        //     printf("%d, %.17lf\n", index, decp_data[index]);
                        //     printf("%d %.17lf\n", next_vertex, decp_data[next_vertex]);
                        //     printf("%.17lf %.17lf \n",d1, d);
                        //     printf("%.17lf %.17lf \n",input_data[index], input_data[next_vertex]);
                        //     // cout<<"index: "<<index<<", "<<decp_data[index]<<endl;
                        //     // cout<<"next_vertex: "<<next_vertex<<","<<decp_data[next_vertex]<<endl;
                        //     // cout<<"smallest_vertex: "<<smallest_vertex<<", "<<decp_data[smallest_vertex]<<endl;
                        //     // cout<<"diff: "<<d<<","<<"d: "<<d<<endl;
                        // }
                   
                };
                     }
            
                 
            
                
            }
            else{
                
                if(decp_data[index]>decp_data[next_vertex]){
                    double t = (decp_data[next_vertex]-(input_data[index]-bound))/2.0;
                    if(abs(input_data[index]-decp_data[next_vertex]+t)<=bound and t>=1e-16){
                            
                            
                            decp_data[index] = decp_data[next_vertex] - t;
                            // decp_data[next_vertex] = t;
                        }
                    else{
                        
                        decp_data[index] = input_data[index] - bound;
                        
                    }
                    // if(count_f_max==1){
                    //         printf("改变后dd");
                    //         printf("%d, %.17lf, %.17lf\n", index, decp_data[index],input_data[index]-bound);
                    //         printf("%d %.17lf\n", next_vertex, decp_data[next_vertex]);
                    //         printf("%.17lf %.17lf \n",d1, d);
                    //         printf("%.17lf %.17lf \n",input_data[index], input_data[next_vertex]);
                    //         // cout<<"index: "<<index<<", "<<decp_data[index]<<endl;
                    //         // cout<<"next_vertex: "<<next_vertex<<","<<decp_data[next_vertex]<<endl;
                    //         // cout<<"smallest_vertex: "<<smallest_vertex<<", "<<decp_data[smallest_vertex]<<endl;
                    //         // cout<<"diff: "<<d<<","<<"d: "<<d<<endl;
                    //     }
                }
                else if(decp_data[index]==decp_data[next_vertex]){
                    // double bound1 = abs(input_data[index]-decp_data[index]);
                    //
                    double d = (bound - (input_data[index]-decp_data[index]))/2.0;
                    double d1 = (bound - (input_data[next_vertex]-decp_data[next_vertex]))/2.0;
                    // while(abs(input_data[index]-decp_data[index]-d)>bound and d>=2e-16){
                    //         d/=2;
                    // }
                    // if(index==157569){
                    //     cout<<"在这时候d: "<<d<<endl;
                    // }   
                    // double d = 1e-16;
                    if(abs(input_data[index]-decp_data[index]+d)<=bound){
                        
                        decp_data[index]-=d;
                    }
                    
                    else if(abs(input_data[next_vertex]-decp_data[next_vertex]-d1)<=bound){
                        // if(next_vertex==78){cout<<"在这里21"<<endl;}
                        decp_data[next_vertex]+=d1;
                    }
                }
                
            }
            
            
        
        }
        else{
            // if(index==25026 and count_f_max<=770){
            //     cout<<"在这里"<<endl;
            // }
            // find_direction2(0,index);
            int largest_index = from_direction_to_index1(index,de_direction_as[index]);
            // 对的
            double diff = (bound-(input_data[index]-decp_data[index]))/2.0;
            double d = (bound-(input_data[largest_index]-decp_data[index]))/2.0;
            // double diff = (input_data[index]-decp_data[index])/2.0;
            // double d = (input_data[largest_index]-decp_data[index])/2.0;
            // double d1 = ((input_data[next_vertex] + (input_data[next_vertex]-decp_data_copy[next_vertex])) - decp_data[next_vertex])/2.0;
            // double diff1 = ((input_data[next_vertex]-decp_data_copy[next_vertex]) - (input_data[next_vertex]-decp_data[index]))/2.0;
            // if(index==25026 and count_f_max<=770){
            //     cout<<"改变前"<<endl;
            //     cout<<"index: "<<index<<", "<<decp_data[index]<<endl;
            //     cout<<"next_vertex: "<<largest_index<<","<<decp_data[largest_index]<<endl;
            //     // cout<<"smallest_vertex: "<<smallest_vertex<<", "<<decp_data[smallest_vertex]<<endl;
            //     cout<<"diff: "<<d<<","<<"d: "<<d<<endl;
            //     cout<<or_direction_as[25026]<<de_direction_as[25026]<<endl;
            // }
            // if(count_f_max==1 and count_f_min==0){
            //     printf("fp改变后");
            //     printf("%d, %f\n", index, decp_data[index]);
            //     printf("%d %f\n", largest_index, decp_data[largest_index]);
            //     printf("%f %f \n",diff, d);
            //     printf("%.17lf %.17lf \n",input_data[index], input_data[largetst_index]);
            //     // cout<<"index: "<<index<<", "<<decp_data[index]<<endl;
            //     // cout<<"next_vertex: "<<next_vertex<<","<<decp_data[next_vertex]<<endl;
            //     // cout<<"smallest_vertex: "<<smallest_vertex<<", "<<decp_data[smallest_vertex]<<endl;
            //     // cout<<"diff: "<<d<<","<<"d: "<<d<<endl;
            // }
            // if(index==6345199){
            //     printf("改变后");
            //     printf("%d, %f\n", index, decp_data[index]);
            //     printf("%d %f\n",largest_index, decp_data[largest_index]);
            //     printf("%f %f \n",diff, d);
            //     printf("%d %d \n",de_direction_as[index],or_maxi[index]);
            //     // cout<<"index: "<<index<<", "<<decp_data[index]<<endl;
            //     // cout<<"next_vertex: "<<next_vertex<<","<<decp_data[next_vertex]<<endl;
            //     // cout<<"smallest_vertex: "<<smallest_vertex<<", "<<decp_data[smallest_vertex]<<endl;
            //     // cout<<"diff: "<<d<<","<<"d: "<<d<<endl;
            // }
            if(decp_data[index]>decp_data[largest_index] or(decp_data[index]==decp_data[largest_index] and index>largest_index)){
                de_direction_as[index] = -1;
            }
            if(d>=1e-16){
                
                if (decp_data[index]<=decp_data[largest_index]){
                    if(abs(input_data[largest_index]-decp_data[index]+d)){
                        // if(largest_index==66783){cout<<"在这里17"<<endl;}
                        decp_data[largest_index] = decp_data[index]-d;
                    }
                }
                
            
                
            }
            
            else{
                if(decp_data[index]<=decp_data[largest_index]){
                    // if(index==78){
                    //         cout<<"在这里1"<<endl;
                    //     }
                    decp_data[index] = input_data[index] + bound;
                }
                    
            }

            // if(index==15885 and count_f_max==7){
            //     cout<<"改变后"<<endl;
            //     cout<<"index: "<<index<<", "<<decp_data[index]<<endl;
            //     cout<<"next_vertex: "<<largest_index<<","<<decp_data[largest_index]<<endl;
            //     // cout<<"smallest_vertex: "<<smallest_vertex<<", "<<decp_data[smallest_vertex]<<endl;
            //     cout<<"diff: "<<d<<","<<"d: "<<d<<endl;
            // }
            
        }
        
        
    
    }
    
    else if (direction != 0 && index_f<count_f_min && lowgradientindices[all_min[index_f]]==0){
        int index = all_min[index_f];
        if (or_mini[index]!=-1){
            // find_direction2(1,index);
            int next_vertex= from_direction_to_index1(index,or_mini[index]);
            
            double diff = (bound - (input_data[next_vertex]-decp_data[index]))/2.0;
            double d =  (bound+input_data[index]-decp_data[index])/2.0;
            // double d1 =  (bound-(input_data[next_vertex]-decp_data[next_vertex]))/2.0;
            
            double d1 = (decp_data[next_vertex]-input_data[next_vertex]+bound)/2.0;
            if(decp_data[index]>decp_data[next_vertex] or (decp_data[index]==decp_data[next_vertex] and index>next_vertex)){
                de_direction_ds[index]=or_mini[index];
                return;
            }

            // if(index == 6595 and count_f_min==5){
            //     cout<<"下降："<<endl;
            //     cout<<"next: "<<next_vertex<<", "<<decp_data[next_vertex]<<endl;
            //     cout<<"index: "<<index<<", "<<decp_data[index]<<endl;
            //     cout<<"daxiaoguanxi: "<<decp_data[index]-decp_data[next_vertex]<<endl;
            //     cout<<"diff: "<<diff<<endl;
            //     cout<<"d: "<<d<<endl;
            //     cout<<"d1: "<<d1<<endl;
            // }
            
            if(diff>=1e-16){
                
                if(decp_data[index]==decp_data[next_vertex]){
                    
                      
                    
                        while(abs(input_data[next_vertex]-decp_data[index]-diff)>bound and diff>=2e-16){
                            diff/=2;
                        }
                        
                        if(abs(input_data[next_vertex]-decp_data[index]+diff)<=bound and diff>=1e-16){
                            // if(index==344033 and count_f_min==2){cout<<"在这里22"<<d<<endl;}
                            decp_data[next_vertex]= decp_data[index]-diff;
                        }
                        else if(d1>=1e-16){
                            // if(index==344033 and count_f_min==2){cout<<"在这里23"<<d<<endl;}
                            decp_data[next_vertex]-=d1;
                        }
                        else if(d>=1e-16){
                            // if(index==344033 and count_f_min==2){cout<<"在这里24"<<d<<endl;}
                            decp_data[index]+=d;
                        }

                    
                    
                }
                else{
                    if(decp_data[index]<=decp_data[next_vertex]){
                        
                            while(abs(input_data[next_vertex]-decp_data[index]+diff)>bound and diff >= 2e-16){
                                    diff/=2;
                            }
                            
                            
                            if (abs(input_data[next_vertex]-decp_data[index]+diff)<=bound and decp_data[index]<=decp_data[next_vertex] and diff>=1e-16){
                                // while(abs(input_data[next_vertex]-decp_data[index]+diff)<=bound and diff<1e-17){
                                //     diff*=2;
                                // }
                                // if(index==270808 and count_f_min==1){cout<<"在这里2！"<< endl;}
                                while(abs(input_data[next_vertex]-decp_data[index]+diff)<bound and diff <= 1e-17){
                                    diff*=2;
                                }
                                if(abs(input_data[next_vertex]-decp_data[index]+diff)<=bound){
                                    decp_data[next_vertex] = decp_data[index]-diff;
                                }
                                // if(index == 6595 and count_f_min==5){
                                //     cout<<"在这里1！"<< diff <<", "<<index<<", "<<decp_data[index]<<","<<input_data[index]<<","<<input_data[next_vertex]<<endl;

                                // }
                                // if(next_vertex==66783){cout<<"在这里13"<<endl;}
                                // decp_data[next_vertex] = decp_data[index]-diff;
                                // if(index==89797){
                                //         cout<<"在这里2"<<diff<<", "<<d<<endl;
                                // }

                                // decp_data[index]+=d;
                            }
                            // else if(abs(input_data[index]-decp_data[index]-d)<=bound and decp_data[index]<=decp_data[next_vertex] and d>0){
                            //     if(index==135569){cout<<"在这里23"<<endl;}
                            //     decp_data[index]+=d;
                            // }
                            else if(abs(input_data[next_vertex]-decp_data[next_vertex]+d1)<=bound and decp_data[index]<=decp_data[next_vertex] and d1>=1e-16){
                                while(abs(input_data[next_vertex]-decp_data[next_vertex]+d1)<bound and d1<=1e-16){
                                    d1*=2;
                                }
                                // if(count_f_min<=12){cout<<"在这里2！"<<abs(input_data[next_vertex]-decp_data[next_vertex]+d1)<<"," <<d1<< endl;}
                                if(abs(input_data[next_vertex]-decp_data[next_vertex]+d1)<=bound and d1>=1e-16){
                                    decp_data[next_vertex]-=d1;
                                }
                                // else{
                                //     decp_data[index] += d;
                                // }
                                // else{
                                // decp_data[next_vertex] = input_data[next_vertex] - bound;}
                                
                            }
                            else{
                                decp_data[next_vertex] = input_data[next_vertex] - bound;
                                // if(index == 6595 and count_f_min==5){cout<<"在这里3！"<<abs(input_data[next_vertex]-bound-decp_data[next_vertex])<< endl;}
                            }
                            
                            
                        
                        
                };

                }
                
                

                
            }

            else{
                
                if(decp_data[index]<decp_data[next_vertex]){
                    // if(next_vertex==339928 and wrong_maxi_cp.size()==84){
                    //     cout<<"np下降："<<endl;
                    //     cout<<"next: "<<next_vertex<<", "<<decp_data[next_vertex]<<endl;
                    //     cout<<"index: "<<index<<", "<<decp_data[index]<<endl;
                    //     cout<<"daxiaoguanxi: "<<decp_data[index]-decp_data[next_vertex]<<endl;
                    //     cout<<"diff: "<<diff<<endl;
                    //     cout<<"d: "<<d<<endl;
                
                    //     }
                        
                        // if(abs(input_data[index]-(decp_data[next_vertex]))<=bound and abs(input_data[next_vertex]-decp_data[index])<=bound){
                        //     double t = decp_data[index];
                        //     decp_data[index] = decp_data[next_vertex];
                        //     if(next_vertex==66783){cout<<"在这里14"<<endl;}
                        //     decp_data[next_vertex] = t;
                            
                        // }
                        double t = (decp_data[index]-(input_data[index]-bound))/2.0;
                        if(abs(input_data[next_vertex]-decp_data[index]+t)<bound and t>=1e-16){
                            
                            // if(index==949999){cout<<"在这里24"<<endl;}
                            // decp_data[index] = decp_data[next_vertex];
                            // if(next_vertex==66783){cout<<"在这里14"<<endl;}
                            decp_data[next_vertex] = decp_data[index]-t;
                            
                        }
                        else{
                            // if(index==949999){cout<<"在这里29"<<endl;}
                            decp_data[index] = input_data[index] + bound;
                        }
                }
                
                else if(decp_data[index]==decp_data[next_vertex]){
                    double d = (bound - (input_data[index]-decp_data[index]))/2.0;
                    // while(abs(input_data[index]-decp_data[index]-d)>bound and d>=2e-16){
                    //         d/=2;
                    // }
                    // if(index==949999){
                    //     cout<<"在这里99 "<<d<<endl;
                    // }   
                    // double d = 1e-16;
                    if(abs(input_data[index]-decp_data[index]-d)<=bound){
                        decp_data[index]+=d;
                    }
                    else if(abs(input_data[next_vertex]-decp_data[next_vertex]+d)<=bound){
                        // if(next_vertex==66783){cout<<"在这里13"<<endl;}
                        decp_data[next_vertex]-=d;
                    }
                }
            }
            

            
            
            
        // if(index == 6595 and count_f_min==5){
        //         cout<<"下降后："<<endl;
        //         cout<<"next: "<<next_vertex<<", "<<decp_data[next_vertex]<<endl;
        //         cout<<"index: "<<index<<", "<<decp_data[index]<<endl;
        //         cout<<"daxiaoguanxi: "<<decp_data[index]-decp_data[next_vertex]<<endl;
        //         cout<<"diff: "<<diff<<endl;
        //         cout<<"d: "<<d<<endl;
        //         cout<<"d1: "<<d1<<endl;
        //         cout<<input_data[index]<<","<<input_data[next_vertex]<<endl;
        //     }
            
        
        }
    
        else{
            // find_direction2(0,index);
            int largest_index = from_direction_to_index1(index,de_direction_ds[index]);
            double diff = (bound-(input_data[index]-decp_data[index]))/2.0;
            double d = (bound-(input_data[largest_index]-decp_data[index]))/2.0;
            // if(count_f_min==84){
            //     cout<<"np下降："<<endl;
            //     cout<<"next: "<<largest_index<<", "<<decp_data[largest_index]<<endl;
            //     cout<<"index: "<<index<<", "<<decp_data[index]<<endl;
            //     cout<<"daxiaoguanxi: "<<decp_data[index]-decp_data[largest_index]<<endl;
            //     cout<<"diff: "<<diff<<endl;
            //     cout<<"d: "<<d<<endl;
                
            // }
            if(decp_data[index]<decp_data[largest_index] or (decp_data[index]==decp_data[largest_index] and index<largest_index)){
                de_direction_ds[index] = -1;
                return;
            }
            
            if (diff>=1e-16){
                if (decp_data[index]>=decp_data[largest_index]){
                    while(abs(input_data[index]-decp_data[index]+diff)>bound and diff>=2e-16){
                        diff/=2;
                    }
                    
                    
                    if(abs(input_data[index]-decp_data[index]+diff)<=bound){
                        // if(index==999973){
                        //     cout<<"在这里2！"<<endl;
                        // }
                        
                        decp_data[index] -= diff;
                    }
                    
                    
                }                    
            }
            
                    
            else{
                if (decp_data[index]>=decp_data[largest_index]){
                    
                    // if(index==66783){cout<<"在这里15"<<endl;}
                    decp_data[index] = input_data[index] - bound;
                }   
    
            }


               
        }

        
    }    
    return;
}

__global__ void fix_maxi_critical5(int direction){
    int index_f = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (direction == 0 && index_f<count_f_max && lowgradientindices[all_max[index_f]]==0){
        
        int index = all_max[index_f];
        
        if (or_maxi[index]!=-1){
            
            int next_vertex = from_direction_to_index1(index,or_maxi[index]);
            
            int smallest_vertex = next_vertex;
            double threshold = -DBL_MAX;
            
            
            for(int j=0;j<12;j++){
                int i = adjacency[index*12+j];
                if(lowgradientindices[i]==1){
                    continue;
                }
                if(i==-1){
                    break;
                }
                if(input_data[i]<input_data[index] and input_data[i]>threshold and i!=next_vertex){
                    smallest_vertex = i;
                    threshold = input_data[i];
                }
            }
            
            threshold = decp_data[smallest_vertex];
            // double diff = (bound - (input_data[next_vertex]-decp_data[index]))/2.0;
            double d = (bound - (input_data[index]-decp_data[index]))/2.0;
            
            if(decp_data[index]<decp_data[next_vertex] or (decp_data[index]==decp_data[next_vertex] and index<next_vertex)){
                
                de_direction_as[index]=or_maxi[index];
                
                return;
            }
            
            if(d>=1e-16){
                
                if(decp_data[index]==decp_data[next_vertex])
                    {
                        
                        while(abs(input_data[index]-decp_data[index]+d)>bound and d>=2e-16){
                            d/=2;
                        }
                        if (abs(input_data[index]-decp_data[index]+d)<=bound){
                            decp_data[index] -= d;
                        }

                    
                    }
                else{
                    if(decp_data[index]>=decp_data[next_vertex]){
                        
                        while(abs(input_data[index]-decp_data[index]+d)>bound and d>=2e-16){
                                d/=2;
                        }
                        
                        if(decp_data[index]>=threshold and threshold<=decp_data[next_vertex]){
                            
                            while(decp_data[index] - d < threshold and d>=2e-16)
                            {
                                d/=2;
                            }
                            
                            
                        }
                        else if(threshold>decp_data[next_vertex]){
                            
                            
                            double diff2 = (bound-(input_data[smallest_vertex]-decp_data[smallest_vertex]))/2;
                            
                            if(diff2>1e-16){
                                while(abs(input_data[smallest_vertex]-decp_data[smallest_vertex]+diff2)>bound and diff2>=2e-16 and decp_data[smallest_vertex]-diff2>decp_data[next_vertex]){
                                    
                                    diff2/=2;
                                }
                                
                                if(abs(input_data[smallest_vertex]-decp_data[smallest_vertex]+diff2)<=bound){
                                    decp_data[smallest_vertex]-=diff2;
                                    // if(index==97) cout<<"处理97的时候: "<<decp_data[next_vertex]<<", "<<decp_data[index]<<endl;
                                }
                                
                                
                            }
                            
                        }

                        if(abs(input_data[index]-(decp_data[index]-d))<=bound and decp_data[index]>=decp_data[next_vertex]){
                            decp_data[index] -= d;
                            
                        }
                        
                        
                   
                };
                     }

                 
            
                
            }
            else{
                
                if(decp_data[index]>=decp_data[next_vertex]){
                    if(abs(input_data[index]-(input_data[next_vertex] -bound+ decp_data[index])/2.0)<=bound){
                        decp_data[index] = (input_data[next_vertex] -bound + decp_data[index])/2.0;
                    }
                    else{
                        
                        decp_data[index] = input_data[index] - bound;
                    }
                    
                }
                
            }
            
            
        
        }
        else{
            // printf("%d \n",or_maxi[index]);
            int largest_index = from_direction_to_index1(index,de_direction_as[index]);
            // double diff = (bound-(input_data[index]-decp_data[index]))/2.0;
            double d = (bound-(input_data[largest_index]-decp_data[index]))/2.0;
            if(decp_data[index]>decp_data[largest_index] or(decp_data[index]==decp_data[largest_index] and index>largest_index)){
                de_direction_as[index] = -1;
            }
            if(d>=1e-16){
                
                if (decp_data[index]<=decp_data[largest_index]){
                    if(abs(input_data[largest_index]-decp_data[index]+d)){
                        decp_data[largest_index] = decp_data[index]-d;
                    }
                }
                
            
                
            }
            
            else{
                if(decp_data[index]<=decp_data[largest_index]){
                    decp_data[index] = input_data[index] + bound;
                }
                    
            }
            
        }
        
        
    
    }
    
    else if(direction == 1 && index_f<count_f_min && lowgradientindices[all_min[index_f]]==0){
        int index = all_min[index_f];
        if (or_mini[index]!=-1){
            int next_vertex= from_direction_to_index1(index,or_mini[index]);
            
            double diff = (bound - (input_data[next_vertex]-decp_data[index]))/2.0;
            double d =  (bound-(input_data[index]-decp_data[index]))/2.0;
            
            
            if(decp_data[index]>decp_data[next_vertex] or (decp_data[index]==decp_data[next_vertex] and index>next_vertex)){
                de_direction_ds[index]=or_mini[index];
                return;
            }
            
            if(diff>=1e-16 or d>=1e-16){
                if(decp_data[index]==decp_data[next_vertex]){
                    
                    
                        while(abs(input_data[next_vertex]-decp_data[index]-d)>bound and d>=2e-16){
                            d/=2;
                        }
                        
                        if(abs(input_data[index]-decp_data[index]-d)<=bound){
                            decp_data[index]+=d;
                        }
                    
                    
                    
                    
                }
                else{
                    if(decp_data[index]<=decp_data[next_vertex]){
                        
                            while(abs(input_data[next_vertex]-decp_data[index]+diff)>bound and diff >= 2e-16){
                                    diff/=2;
                            }
                            
                            if (abs(input_data[next_vertex]-decp_data[index]+d)<=bound and decp_data[index]<=decp_data[next_vertex]){
                                decp_data[next_vertex] = decp_data[index]-diff;
                            }
                            
                            
                        
                        
                };

                }
                
                

                
            }

            else{
                
                if(decp_data[index]<=decp_data[next_vertex]){
                    if(abs(input_data[index]-(input_data[next_vertex] + bound + decp_data[index])/2.0)<=bound){
                        decp_data[index] = (input_data[next_vertex] + bound + decp_data[index])/2.0;
                    }
                    else{
                        decp_data[index] = input_data[index] + bound;
                    }
                }
            }
            

            
            
            

            
        
        }
    
        else{
            
            int largest_index = from_direction_to_index1(index,de_direction_ds[index]);
            double diff = (bound-(input_data[index]-decp_data[index]))/2.0;
            // double d = (bound-(input_data[largest_index]-decp_data[index]))/2.0;
            
            if(decp_data[index]<decp_data[largest_index] or (decp_data[index]==decp_data[largest_index] and index<largest_index)){
                de_direction_ds[index] = -1;
                return;
            }
            
            if (diff>=1e-16){
                if (decp_data[index]>=decp_data[largest_index]){
                    while(abs(input_data[index]-decp_data[index]+diff)>bound and diff>=2e-16){
                        diff/=2;
                    }
                    
                    
                    if(abs(input_data[index]-decp_data[index]+diff)<=bound){
                        decp_data[index] -= diff;
                    }
                    
                    
                }                    
            }
            
                    
            else{
                if (decp_data[index]>=decp_data[largest_index]){
                    decp_data[index] = input_data[index] - bound;
                }   
    
            }


               
        }

        
    }    
    return;
};
__global__ void getlabel(int *un_sign_ds, int *un_sign_as, int type=0){
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int *direction_as;
    int *direction_ds;
    int *label;
    
    if(i>=num or lowgradientindices[i]==1){
        // printf("%d\n",num);
        
        return;
    }
    
    if(type==0){
        direction_as = de_direction_as;
        direction_ds = de_direction_ds;
        label = dec_label;
    }
    else{
        direction_as = or_maxi;
        direction_ds = or_mini;
        label = or_label;
    }
    
    int cur = label[i*2+1];
    
    
    int next_vertex;
    // cur!=-1就说明它首先不是cp，direction_as[cur]也说明他不是cp
    if (cur!=-1 and direction_as[cur]!=-1){
        
        int direc = direction_as[cur];
        // 找到他的下一个邻居
        
        next_vertex = from_direction_to_index1(cur, direc);
        
        // 检查下一个邻居是否为cp，如果是，直接把label换成邻居
        if(label[next_vertex*2+1] == -1){
            label[i*2+1] = next_vertex;
            
        }
        
        else{
            
            label[i*2+1] = label[next_vertex*2+1];
            
            
        }
        
        if (direction_as[label[i*2+1]] != -1){
            *un_sign_as+=1;  
        }
        
    }
    
    
    
    
    cur = label[i*2];
    int next_vertex1;
    
    
    if (cur!=-1 and label[cur*2]!=-1){
        
        int direc = direction_ds[cur];
        // 找到他的下一个邻居
        next_vertex1 = from_direction_to_index1(cur, direc);
        // 检查下一个邻居是否为cp，如果是，直接把label换成邻居
        if(label[next_vertex1*2] == -1){
            label[i*2] = next_vertex1;
            
        }
        // 如果不是cp，检查邻居是否找到cp，如果找到了，就换成邻居的label
        else if(label[label[next_vertex1*2]*2] == -1){
            label[i*2] = label[next_vertex1*2];  
        }
        
        else if(direction_ds[i]!=-1){
            // 如果邻居不是cp，那就替换成邻居的当前邻居
            if(label[next_vertex1*2]!=-1){
                label[i*2] = label[next_vertex1*2];
            }
            // 否则：下一个邻居是cp, 那么他的cp就是下一个邻居
            else{

                label[i*2] = next_vertex1;
            }
            
            
        }
        // if(i==66590){
        //     printf("%d %d %d %d %d\n",next_vertex,de_direction_as[next_vertex],de_direction_as[label[next_vertex*2+1]],label[next_vertex*2+1],label[i*2+1]);
        // }
        if (direction_ds[label[i*2]]!=-1){
            *un_sign_ds+=1;
        }
    } 

}
__global__ void fix_maxi_critical2(int direction){
    int index_f = blockIdx.x * blockDim.x + threadIdx.x;
    
    
        
    
    if (direction == 0 && index_f<count_f_max && lowgradientindices[all_max[index_f]]==0){
        
        int index = all_max[index_f];
      
	// printf("%d\n",index);
        if (or_maxi[index]!=-1){
            // printf("%d\n",index);
            // find_direction2(1,index);
            int next_vertex = from_direction_to_index1(index,or_maxi[index]);
            
            int smallest_vertex = next_vertex;
            double threshold = -DBL_MAX;
            
            
            for(int j=0;j<12;j++){
                int i = adjacency[index*12+j];
                if(i==-1){
                    break;
                }
                if(lowgradientindices[i]==1){
                    continue;
                }
                
                if(input_data[i]<input_data[index] and input_data[i]>threshold and i!=next_vertex){
                    smallest_vertex = i;
                    threshold = input_data[i];
                }
            }
            
            threshold = decp_data[smallest_vertex];
            // 对的
            // d是把index还要降低
            // 如果是tthresh的话，那它的下限就是：input_data[index]-(abs(inaput_data[index]-decp_data[index]))
            // 之前的
            // double diff = (bound - (input_data[next_vertex]-decp_data[index]))/2.0;
            // double d = (decp_data[index] - input_data[index] + bound )/2.0;
            // // double d = (decp_data[index]-(input_data[index]-(abs(input_data[index]-decp_data_copy[index]))))/2.0;
            // double d1 = ((input_data[next_vertex] + bound) - decp_data[next_vertex])/2.0;
            // double diff1 = (bound - (input_data[next_vertex]-decp_data[index]))/2.0;
            double diff = (bound - (input_data[next_vertex]-decp_data[index]))/2.0;
            double d = (decp_data[index] - input_data[index] + bound )/2.0;
            double d1 = ((input_data[next_vertex] + bound) - decp_data[next_vertex])/2.0;
            double diff1 = (bound - (input_data[next_vertex]-decp_data[index]))/2.0;
            // double diff = ((input_data[next_vertex]-decp_data_copy[next_vertex]) - (input_data[next_vertex]-decp_data[index]))/2.0;
            // double d = (decp_data[index] - input_data[index] + (input_data[index]-decp_data_copy[index]))/2.0;
            // double d1 = ((input_data[next_vertex] + (input_data[next_vertex]-decp_data_copy[next_vertex])) - decp_data[next_vertex])/2.0;
            // double diff1 = ((input_data[next_vertex]-decp_data_copy[next_vertex]) - (input_data[next_vertex]-decp_data[index]))/2.0;
            if(false){
                 printf("改变后");
                 printf("%d, %f\n", index, decp_data[index]);
                 printf("%d %f\n", next_vertex, decp_data[next_vertex]);
                 printf("%f %f \n",diff, d);
            //     // cout<<"index: "<<index<<", "<<decp_data[index]<<endl;
            //     // cout<<"next_vertex: "<<next_vertex<<","<<decp_data[next_vertex]<<endl;
            //     // cout<<"smallest_vertex: "<<smallest_vertex<<", "<<decp_data[smallest_vertex]<<endl;
            //     // cout<<"diff: "<<d<<","<<"d: "<<d<<endl;
             }
            if(decp_data[index]<decp_data[next_vertex] or (decp_data[index]==decp_data[next_vertex] and index<next_vertex)){
                
                de_direction_as[index]=or_maxi[index];
            
                return;
            }
            
            if(d>=1e-16 or d1>=1e-16){
                
                if(decp_data[index]==decp_data[next_vertex])
                    {
                        
                        while(abs(input_data[index]-decp_data[index]+d)>bound and d>=2e-16){
                            d/=2;
                        }
                        if (abs(input_data[index]-decp_data[index]+d/64)<=bound){
                            decp_data[index] -= d/64;
                        }
                    }
                else{
                    if(decp_data[index]>=decp_data[next_vertex]){
                        
                        while(abs(input_data[index]-decp_data[index]+d)>bound and d>=2e-16){
                                d/=2;
                        }
                        
                        if(decp_data[index]>=threshold and threshold<=decp_data[next_vertex]){
                            
                            while(decp_data[index] - d < threshold and d>=2e-16)
                            {
                                d/=2;
                            }
                            
                            
                        }
                        // else if(threshold>decp_data[next_vertex]){
                            
                            
                        //     double diff2 = (bound-(input_data[smallest_vertex]-decp_data[smallest_vertex]))/64;
                            
                        //     if(diff2>=1e-16){
                        //         while(abs(input_data[smallest_vertex]-decp_data[smallest_vertex]+diff2)>bound and diff2>=2e-16 and decp_data[smallest_vertex]-diff2>decp_data[next_vertex]){
                                    
                        //             diff2/=2;
                        //         }
                                
                        //         if(abs(input_data[smallest_vertex]-decp_data[smallest_vertex]+diff2)<=bound){
                        //             if(smallest_vertex==66783){cout<<"在这里11."<<endl;}
                        //             decp_data[smallest_vertex]-=diff2;
                        //             // if(index==97) cout<<"处理97的时候: "<<decp_data[next_vertex]<<", "<<decp_data[index]<<endl;
                        //         }
                                
                                
                        //     }
                            
                        // }

                        if(abs(input_data[index]-(decp_data[index]-d))<=bound and decp_data[index]>=decp_data[next_vertex] and d>=1e-16){
                            // if(index==1620477){
                            //     // cout<<"next_vertex: "<<decp_data[next_vertex]<<endl;
                            //     // cout<<"smallest_vertex: "<<decp_data[smallest_vertex]<<endl;
                            //     cout<<"before index: "<<decp_data[index]<<endl;
                                
                            // }
                            
                            decp_data[index] -= d;
                            
                            
                                            
                        }
                        // else if(abs(input_data[next_vertex]-(decp_data[next_vertex]+d1))<=bound and decp_data[index]>=decp_data[next_vertex] and d1>0){
                        //     // if(index==1620477){
                        //     //     // cout<<"next_vertex: "<<decp_data[next_vertex]<<endl;
                        //     //     // cout<<"smallest_vertex: "<<decp_data[smallest_vertex]<<endl;
                        //     //     cout<<"before index: "<<decp_data[index]<<endl;
                                
                        //     // }
                            
                        //     decp_data[next_vertex] += d1;
                            
                            
                                            
                        // }
                        
                        
                   
                };
                     }
            
                 
            
                
            }
            else{
                
                if(decp_data[index]>decp_data[next_vertex]){
                    
                    if(abs(input_data[index]-decp_data[next_vertex])<bound){
                            
                            double t = (decp_data[next_vertex]-(input_data[index]-bound))/2.0;
                            decp_data[index] = decp_data[next_vertex] + t;
                            // decp_data[next_vertex] = t;
                        }
                    else{
                        decp_data[index] = input_data[index] - bound;
                        
                    }
                    
                }
                else if(decp_data[index]==decp_data[next_vertex]){
                    // double bound1 = abs(input_data[index]-decp_data[index]);
                    //
                    double d = (bound - (input_data[index]-decp_data[index]))/64;
                    // while(abs(input_data[index]-decp_data[index]-d)>bound and d>=2e-16){
                    //         d/=2;
                    // }
                    // if(index==157569){
                    //     cout<<"在这时候d: "<<d<<endl;
                    // }   
                    // double d = 1e-16;
                    if(abs(input_data[index]-decp_data[index]+d)<=bound){
                        
                        decp_data[index]-=d;
                    }
                    else if(abs(input_data[next_vertex]-decp_data[next_vertex]-d)<=bound){
                        // if(next_vertex==78){cout<<"在这里21"<<endl;}
                        decp_data[next_vertex]+=d;
                    }
                }
                
            }
            
            
        
        }
        else{
            // if(index==25026 and count_f_max<=770){
            //     cout<<"在这里"<<endl;
            // }
            // find_direction2(0,index);
            int largest_index = from_direction_to_index1(index,de_direction_as[index]);
            // 对的
            double diff = (bound-(input_data[index]-decp_data[index]))/2.0;
            double d = (bound-(input_data[largest_index]-decp_data[index]))/2.0;
            // double diff = (input_data[index]-decp_data[index])/2.0;
            // double d = (input_data[largest_index]-decp_data[index])/2.0;
            // double d1 = ((input_data[next_vertex] + (input_data[next_vertex]-decp_data_copy[next_vertex])) - decp_data[next_vertex])/2.0;
            // double diff1 = ((input_data[next_vertex]-decp_data_copy[next_vertex]) - (input_data[next_vertex]-decp_data[index]))/2.0;
            // if(index==25026 and count_f_max<=770){
            //     cout<<"改变前"<<endl;
            //     cout<<"index: "<<index<<", "<<decp_data[index]<<endl;
            //     cout<<"next_vertex: "<<largest_index<<","<<decp_data[largest_index]<<endl;
            //     // cout<<"smallest_vertex: "<<smallest_vertex<<", "<<decp_data[smallest_vertex]<<endl;
            //     cout<<"diff: "<<d<<","<<"d: "<<d<<endl;
            //     cout<<or_direction_as[25026]<<de_direction_as[25026]<<endl;
            // }
            // if(index==6345199){
            //     printf("改变后");
            //     printf("%d, %f\n", index, decp_data[index]);
            //     printf("%d %f\n",largest_index, decp_data[largest_index]);
            //     printf("%f %f \n",diff, d);
            //     printf("%d %d \n",de_direction_as[index],or_maxi[index]);
            //     // cout<<"index: "<<index<<", "<<decp_data[index]<<endl;
            //     // cout<<"next_vertex: "<<next_vertex<<","<<decp_data[next_vertex]<<endl;
            //     // cout<<"smallest_vertex: "<<smallest_vertex<<", "<<decp_data[smallest_vertex]<<endl;
            //     // cout<<"diff: "<<d<<","<<"d: "<<d<<endl;
            // }
            if(decp_data[index]>decp_data[largest_index] or(decp_data[index]==decp_data[largest_index] and index>largest_index)){
                de_direction_as[index] = -1;
            }
            if(d>=1e-16){
                
                if (decp_data[index]<=decp_data[largest_index]){
                    if(abs(input_data[largest_index]-decp_data[index]+d)){
                        // if(largest_index==66783){cout<<"在这里17"<<endl;}
                        decp_data[largest_index] = decp_data[index]-d;
                    }
                }
                
            
                
            }
            
            else{
                if(decp_data[index]<=decp_data[largest_index]){
                    // if(index==78){
                    //         cout<<"在这里1"<<endl;
                    //     }
                    decp_data[index] = input_data[index] + bound;
                }
                    
            }

            // if(index==15885 and count_f_max==7){
            //     cout<<"改变后"<<endl;
            //     cout<<"index: "<<index<<", "<<decp_data[index]<<endl;
            //     cout<<"next_vertex: "<<largest_index<<","<<decp_data[largest_index]<<endl;
            //     // cout<<"smallest_vertex: "<<smallest_vertex<<", "<<decp_data[smallest_vertex]<<endl;
            //     cout<<"diff: "<<d<<","<<"d: "<<d<<endl;
            // }
            
        }
        
        
    
    }
    
    else if (direction != 0 && index_f<count_f_min && lowgradientindices[all_min[index_f]]==0){
        int index = all_min[index_f];
        if (or_mini[index]!=-1){
            // find_direction2(1,index);
            int next_vertex= from_direction_to_index1(index,or_mini[index]);
            
            double diff = (bound - (input_data[next_vertex]-decp_data[index]))/2.0;
            double d =  (bound+input_data[index]-decp_data[index])/2.0;
            // double d1 =  (bound-(input_data[next_vertex]-decp_data[next_vertex]))/2.0;
            
            double d1 = (decp_data[next_vertex]-input_data[next_vertex]+bound)/2.0;
            if(decp_data[index]>decp_data[next_vertex] or (decp_data[index]==decp_data[next_vertex] and index>next_vertex)){
                de_direction_ds[index]=or_mini[index];
                return;
            }

            // if(index == 6595 and count_f_min==5){
            //     cout<<"下降："<<endl;
            //     cout<<"next: "<<next_vertex<<", "<<decp_data[next_vertex]<<endl;
            //     cout<<"index: "<<index<<", "<<decp_data[index]<<endl;
            //     cout<<"daxiaoguanxi: "<<decp_data[index]-decp_data[next_vertex]<<endl;
            //     cout<<"diff: "<<diff<<endl;
            //     cout<<"d: "<<d<<endl;
            //     cout<<"d1: "<<d1<<endl;
            // }
            
            if(diff>=1e-16 or d>=1e-16 or d1>=1e-16){
                
                if(decp_data[index]==decp_data[next_vertex]){
                    
                      
                    
                        while(abs(input_data[next_vertex]-decp_data[index]-diff)>bound and diff>=2e-16){
                            diff/=2;
                        }
                        
                        if(abs(input_data[next_vertex]-decp_data[index]+diff)<=bound and diff>=1e-16){
                            // if(index==344033 and count_f_min==2){cout<<"在这里22"<<d<<endl;}
                            decp_data[next_vertex]= decp_data[index]-diff;
                        }
                        else if(d1>=1e-16){
                            // if(index==344033 and count_f_min==2){cout<<"在这里23"<<d<<endl;}
                            decp_data[next_vertex]-=d1;
                        }
                        else if(d>=1e-16){
                            // if(index==344033 and count_f_min==2){cout<<"在这里24"<<d<<endl;}
                            decp_data[index]+=d;
                        }

                    
                    
                }
                else{
                    if(decp_data[index]<=decp_data[next_vertex]){
                        
                            while(abs(input_data[next_vertex]-decp_data[index]+diff)>bound and diff >= 2e-16){
                                    diff/=2;
                            }
                            
                            
                            if (abs(input_data[next_vertex]-decp_data[index]+diff)<=bound and decp_data[index]<=decp_data[next_vertex] and diff>=1e-16){
                                // while(abs(input_data[next_vertex]-decp_data[index]+diff)<=bound and diff<1e-17){
                                //     diff*=2;
                                // }
                                // if(index==270808 and count_f_min==1){cout<<"在这里2！"<< endl;}
                                while(abs(input_data[next_vertex]-decp_data[index]+diff)<bound and diff <= 1e-17){
                                    diff*=2;
                                }
                                if(abs(input_data[next_vertex]-decp_data[index]+diff)<=bound){
                                    decp_data[next_vertex] = decp_data[index]-diff;
                                }
                                // if(index == 6595 and count_f_min==5){
                                //     cout<<"在这里1！"<< diff <<", "<<index<<", "<<decp_data[index]<<","<<input_data[index]<<","<<input_data[next_vertex]<<endl;

                                // }
                                // if(next_vertex==66783){cout<<"在这里13"<<endl;}
                                // decp_data[next_vertex] = decp_data[index]-diff;
                                // if(index==89797){
                                //         cout<<"在这里2"<<diff<<", "<<d<<endl;
                                // }

                                // decp_data[index]+=d;
                            }
                            // else if(abs(input_data[index]-decp_data[index]-d)<=bound and decp_data[index]<=decp_data[next_vertex] and d>0){
                            //     if(index==135569){cout<<"在这里23"<<endl;}
                            //     decp_data[index]+=d;
                            // }
                            else if(abs(input_data[next_vertex]-decp_data[next_vertex]+d1)<=bound and decp_data[index]<=decp_data[next_vertex] and d1>=1e-16){
                                while(abs(input_data[next_vertex]-decp_data[next_vertex]+d1)<bound and d1<=1e-17){
                                    d1*=2;
                                }
                                // if(count_f_min<=12){cout<<"在这里2！"<<abs(input_data[next_vertex]-decp_data[next_vertex]+d1)<<"," <<d1<< endl;}
                                if(abs(input_data[next_vertex]-decp_data[next_vertex]+d1)<=bound and d1>=1e-16){
                                    decp_data[next_vertex]-=d1;
                                }
                                // else{
                                //     decp_data[index] += d;
                                // }
                                // else{
                                // decp_data[next_vertex] = input_data[next_vertex] - bound;}
                                
                            }
                            else{
                                decp_data[next_vertex] = input_data[next_vertex] - bound;
                                // if(index == 6595 and count_f_min==5){cout<<"在这里3！"<<abs(input_data[next_vertex]-bound-decp_data[next_vertex])<< endl;}
                            }
                            
                            
                        
                        
                };

                }
                
                

                
            }

            else{
                
                if(decp_data[index]<decp_data[next_vertex]){
                    // if(next_vertex==339928 and wrong_maxi_cp.size()==84){
                    //     cout<<"np下降："<<endl;
                    //     cout<<"next: "<<next_vertex<<", "<<decp_data[next_vertex]<<endl;
                    //     cout<<"index: "<<index<<", "<<decp_data[index]<<endl;
                    //     cout<<"daxiaoguanxi: "<<decp_data[index]-decp_data[next_vertex]<<endl;
                    //     cout<<"diff: "<<diff<<endl;
                    //     cout<<"d: "<<d<<endl;
                
                    //     }
                        
                        // if(abs(input_data[index]-(decp_data[next_vertex]))<=bound and abs(input_data[next_vertex]-decp_data[index])<=bound){
                        //     double t = decp_data[index];
                        //     decp_data[index] = decp_data[next_vertex];
                        //     if(next_vertex==66783){cout<<"在这里14"<<endl;}
                        //     decp_data[next_vertex] = t;
                            
                        // }
                        if(abs(input_data[next_vertex]-decp_data[index])<bound){
                            double t = (decp_data[index]-(input_data[index]-bound))/2.0;
                            // if(index==949999){cout<<"在这里24"<<endl;}
                            // decp_data[index] = decp_data[next_vertex];
                            // if(next_vertex==66783){cout<<"在这里14"<<endl;}
                            decp_data[next_vertex] = decp_data[index]-t;
                            
                        }
                        else{
                            // if(index==949999){cout<<"在这里29"<<endl;}
                            decp_data[index] = input_data[index] + bound;
                        }
                }
                
                else if(decp_data[index]==decp_data[next_vertex]){
                    double d = (bound - (input_data[index]-decp_data[index]))/64.0;
                    // while(abs(input_data[index]-decp_data[index]-d)>bound and d>=2e-16){
                    //         d/=2;
                    // }
                    // if(index==949999){
                    //     cout<<"在这里99 "<<d<<endl;
                    // }   
                    // double d = 1e-16;
                    if(abs(input_data[index]-decp_data[index]-d)<=bound){
                        decp_data[index]+=d;
                    }
                    else if(abs(input_data[next_vertex]-decp_data[next_vertex]+d)<=bound){
                        // if(next_vertex==66783){cout<<"在这里13"<<endl;}
                        decp_data[next_vertex]-=d;
                    }
                }
            }
            

            
            
            
        // if(index == 6595 and count_f_min==5){
        //         cout<<"下降后："<<endl;
        //         cout<<"next: "<<next_vertex<<", "<<decp_data[next_vertex]<<endl;
        //         cout<<"index: "<<index<<", "<<decp_data[index]<<endl;
        //         cout<<"daxiaoguanxi: "<<decp_data[index]-decp_data[next_vertex]<<endl;
        //         cout<<"diff: "<<diff<<endl;
        //         cout<<"d: "<<d<<endl;
        //         cout<<"d1: "<<d1<<endl;
        //         cout<<input_data[index]<<","<<input_data[next_vertex]<<endl;
        //     }
            
        
        }
    
        else{
            // find_direction2(0,index);
            int largest_index = from_direction_to_index1(index,de_direction_ds[index]);
            double diff = (bound-(input_data[index]-decp_data[index]))/2.0;
            double d = (bound-(input_data[largest_index]-decp_data[index]))/2.0;
            // if(count_f_min==84){
            //     cout<<"np下降："<<endl;
            //     cout<<"next: "<<largest_index<<", "<<decp_data[largest_index]<<endl;
            //     cout<<"index: "<<index<<", "<<decp_data[index]<<endl;
            //     cout<<"daxiaoguanxi: "<<decp_data[index]-decp_data[largest_index]<<endl;
            //     cout<<"diff: "<<diff<<endl;
            //     cout<<"d: "<<d<<endl;
                
            // }
            if(decp_data[index]<decp_data[largest_index] or (decp_data[index]==decp_data[largest_index] and index<largest_index)){
                de_direction_ds[index] = -1;
                return;
            }
            
            if (diff>=1e-16){
                if (decp_data[index]>=decp_data[largest_index]){
                    while(abs(input_data[index]-decp_data[index]+diff)>bound and diff>=2e-16){
                        diff/=2;
                    }
                    
                    
                    if(abs(input_data[index]-decp_data[index]+diff)<=bound){
                        // if(index==999973){
                        //     cout<<"在这里2！"<<endl;
                        // }
                        
                        decp_data[index] -= diff;
                    }
                    
                    
                }                    
            }
            
                    
            else{
                if (decp_data[index]>=decp_data[largest_index]){
                    
                    // if(index==66783){cout<<"在这里15"<<endl;}
                    decp_data[index] = input_data[index] - bound;
                }   
    
            }


               
        }

        
    }    
    return;
}
__global__ void fixpath11(int direction){
    int index_f = blockIdx.x * blockDim.x + threadIdx.x;
    if(direction == 0 && index_f<count_p_max && lowgradientindices[all_p_max[index_f]]==0){
        int index = all_p_max[index_f];
        int cur = index;
        while (or_maxi[cur] == de_direction_as[cur]){
            int next_vertex =  from_direction_to_index1(cur,de_direction_as[cur]);
            
            if(de_direction_as[cur]==-1 && next_vertex == cur){
                cur = -1;
                break;
            }
            if(next_vertex == cur){
                cur = next_vertex;
                break;
            };
            
            cur = next_vertex;
        }

        int start_vertex = cur;
        
        
        if (start_vertex==-1) return;
        else{
            
            int false_index= from_direction_to_index1(cur,de_direction_as[cur]);
            int true_index= from_direction_to_index1(cur, or_maxi[cur]);
            if(false_index==true_index) return;
            // 对的
            double diff = (input_data[true_index]-bound-decp_data[false_index])/2.0;
            // double diff = (bound-(input_data[true_index]-decp_data[false_index]))/2.0;
            // 对的
            double d = (decp_data[false_index]-input_data[false_index]+bound)/2.0;
            // double d = (bound-(input_data[false_index]-decp_data[false_index]))/2.0;
            // diff是用来给true_index增加的
            // d是用来给false_index见效的
            // double diff = (input_data[true_index]-bound_data[true_index]-decp_data[false_index])/2.0;
            // double d = (input_data[false_index]-bound_data[false_index]-decp_data[false_index])/2.0;
            // if(wrong_index_as.size()<=50){
            // // pre=1;
            //     cout<<index<<", "<<decp_data[index]<<"," <<input_data[index]<<endl;
            //     cout<<start_vertex<<", "<<decp_data[start_vertex]<<"," <<input_data[start_vertex]<<endl;
            //     cout<<true_index<<", "<<decp_data[true_index]<<"," <<input_data[true_index]<<endl;
            //     cout<<false_index<<", "<<decp_data[false_index]<<"," <<input_data[false_index]<<endl;
            //     cout<<diff<<endl;
            //     cout<<d<<endl;
            // }
            if(decp_data[false_index]<decp_data[true_index]){
                de_direction_as[cur]=or_maxi[cur];
            //     if(wrong_maxi_cp.size()==1 and wrong_min_cp.size()==0){
            //     cout<<de_direction_as[64582]<<endl;
            // }
                return;
            }
            double threshold = -DBL_MAX;;
            int smallest_vertex = false_index;
            
            for(int j=0;j<12;j++){
                int i = adjacency[12*false_index+j];
                if(i==-1) continue;
                if(input_data[i]<input_data[false_index] and input_data[i]>threshold and i!=false_index){
                    smallest_vertex = i;
                    threshold = input_data[i];
                }
            }
            
            threshold = decp_data[smallest_vertex];

            double threshold1 = DBL_MAX;;
            int smallest_vertex1 = true_index;
            
            for(int j=0;j<12;j++){
                int i = adjacency[12*true_index+j];
                if(i==-1) continue;
                if(input_data[i]>input_data[true_index] and input_data[i]<threshold1 and i!=true_index){
                    smallest_vertex1 = i;
                    threshold = input_data[i];
                }
            }
            
            threshold1 = decp_data[smallest_vertex1];

            if (diff>=1e-16 or d>=1e-16){
                if (decp_data[false_index]>=decp_data[true_index]){

                    
                    // diff = (bound-abs(input_data[true_index]-decp_data[false_index]))/2.0;
                    while(abs(input_data[false_index]-decp_data[false_index] + d)>bound and d>2e-16){
                                d/=2;
                    }
                    
                    
                    if(decp_data[false_index]>threshold and threshold<decp_data[true_index]){
                            
                            while(decp_data[false_index] - d < threshold and d>=2e-16)
                            {
                                d/=2;
                            }
                            
                            
                    }
                    else if(threshold>=decp_data[true_index]){
                        
                        
                        double diff2 = (bound-(input_data[smallest_vertex1]-decp_data[smallest_vertex1]))/2;
                        
                        if(diff2>1e-16){
                            while(abs(input_data[smallest_vertex]-decp_data[smallest_vertex]+diff2)>bound and diff2>=2e-16 and decp_data[smallest_vertex]-diff2>decp_data[true_index]){
                                
                                diff2/=2;
                            }
                            
                            if(abs(input_data[smallest_vertex]-decp_data[smallest_vertex]+diff2)<=bound){
                                decp_data[smallest_vertex]-=diff2;
                                // if(index==97) cout<<"处理97的时候: "<<decp_data[next_vertex]<<", "<<decp_data[index]<<endl;
                            }
                            
                            
                        }
                        
                    }
                    while(abs(input_data[true_index]-(decp_data[false_index] + diff))>bound and diff>2e-16){
                                diff/=2;
                    }
                    if(decp_data[true_index]<=threshold and threshold>=decp_data[false_index]){
                            
                            while(decp_data[false_index] + diff > threshold and diff>=2e-16)
                            {
                                diff/=2;
                            }
                            
                            
                    }
                    // else if(threshold<=decp_data[false_index]){
                        
                        
                    //     double diff2 = (bound-(input_data[smallest_vertex1]-decp_data[smallest_vertex1]))/2;
                        
                    //     if(diff2>1e-16){
                    //         while(abs(input_data[smallest_vertex]-decp_data[smallest_vertex]-diff2)>bound and diff2>=2e-16 and decp_data[smallest_vertex]+diff2<decp_data[false_index]){
                                
                    //             diff2/=2;
                    //         }
                            
                    //         if(abs(input_data[smallest_vertex]-decp_data[smallest_vertex]-diff2)<=bound){
                    //             decp_data[smallest_vertex]+=diff2;
                    //             // if(index==97) cout<<"处理97的时候: "<<decp_data[next_vertex]<<", "<<decp_data[index]<<endl;
                    //         }
                            
                            
                    //     }
                        
                    // }
                    if (abs(input_data[true_index]-(decp_data[false_index] + diff))<=bound and decp_data[false_index]>=decp_data[true_index]){
                        decp_data[true_index] = decp_data[false_index] + diff;
                    }
                    if (abs(input_data[false_index]-decp_data[false_index] + d)<=bound){
                        
                        decp_data[false_index] -=d;
                    }
                    
                    
                        
                }

                else{
                    de_direction_as[cur] = or_maxi[cur];
                }
                    
            }
            
            else{
                //对的
                double diff = (bound-(input_data[false_index]-decp_data[false_index]))/2.0;
                // if(wrong_index_as.size()==2){
                //     cout<<diff<<endl;
                //     cout<<false_index<<endl;
                // }
                if (decp_data[false_index]>=decp_data[true_index]){
                    if(abs(input_data[false_index]-((decp_data[false_index]+input_data[true_index]-bound)/2.0))<=bound){
                        decp_data[false_index] = (decp_data[false_index]+input_data[true_index]-bound)/2.0;
                    }
                        
                    else{
                        decp_data[false_index] = input_data[false_index] - bound;
                    }
                    
                }
                else{
                    de_direction_as[cur] = or_maxi[cur];
                };        
            }
            
        }
    }

    else if(direction != 0 && index_f<count_p_min && lowgradientindices[all_p_min[index_f]]==0){
        int index = all_p_min[index_f];
        int cur = index;
        
        
        while (or_mini[cur] == de_direction_ds[cur]){
            
            int next_vertex = from_direction_to_index1(cur,de_direction_ds[cur]);
            
            // if(de_direction_ds[cur]==-1 && next_vertex == cur){
            //     if(wrong_index_ds.size()==4){
            //         cout<<cur<<", "<<index <<", "<<de_direction_ds[cur]<<", "<<or_mini[cur]<<endl;
            //     }
            //     cur = -1;
            //     break;
            // }
            if (next_vertex == cur){
                cur = next_vertex;
                break;
            }
            cur = next_vertex;

            // if (cur == -1) break;
                
        }
    
        int start_vertex = cur;
        // if(wrong_index_ds.size()==4){
        //     cout<<"修复的时候变成了:" <<endl;
        //     cout<<start_vertex<<", "<<de_direction_ds[start_vertex]<<", "<<or_direction_ds[start_vertex]<<endl;
        // }
        if (start_vertex==-1) return;
        
        else{
            
            int false_index= from_direction_to_index1(cur,de_direction_ds[cur]);
            int true_index= from_direction_to_index1(cur, or_mini[cur]);
            if(false_index==true_index) return;

            // double diff = (input_data[true_index]+bound-decp_data[false_index])/2.0;
            double diff = (bound-(input_data[true_index]-decp_data[false_index]))/2.0;
            // double d = (input_data[false_index]bound-decp_data[false_index])/1000.0;
            // double d = (input_data[false_index]+bound-decp_data[false_index])/2.0;
            // double diff = (input_data[true_index]-bound-decp_data[false_index])/2.0;
            // // double diff = (bound-(input_data[true_index]-decp_data[false_index]))/2.0;
            // double d = (input_data[false_index]-bound-decp_data[false_index])/2.0;
            double d = (bound-(input_data[false_index]-decp_data[false_index]))/2.0;
            // diff是用来给true_index增加的
            // d是用来给false_index见效的
            // if(wrong_index_as.size()<=10){
            //     cout<<index<<", "<<decp_data[index]<<"," <<input_data[index]<<endl;
            //     cout<<start_vertex<<", "<<decp_data[start_vertex]<<"," <<input_data[start_vertex]<<endl;
            //     cout<<false_index<<", "<<decp_data[false_index]<<"," <<input_data[false_index]<<endl;   
            //     cout<<true_index<<", "<<decp_data[true_index]<<"," <<input_data[true_index]<<endl;                     
            // }
            if(decp_data[false_index]>decp_data[true_index]){
                de_direction_ds[cur]=or_mini[cur];
                return;
            }
            
            if(diff>=1e-16 or d>=1e-16){
                if(decp_data[false_index]<=decp_data[true_index]){
                    
                    // else{
                        
                        // diff = (bound-abs(input_data[true_index]-decp_data[false_index]))/2.0;
                        while(abs(input_data[false_index]-decp_data[false_index] - d)>bound and d>=2e-17){
                            d/=2;
                        }
                        while(abs(input_data[true_index]-(decp_data[false_index] - diff))>bound and diff>=2e-17){
                                    diff/=2;
                        }
                        if(abs(input_data[true_index]-(decp_data[false_index] - diff))<=bound and decp_data[false_index]<=decp_data[true_index]){
                            // decp_data[false_index] = decp_data[true_index] + diff;
                            decp_data[true_index] = decp_data[false_index] - diff;
                        }
                        if(abs(input_data[false_index]-decp_data[false_index] - d)<=bound){
                            decp_data[false_index] += d;
                        }
                        
                        
                        

                        // diff = (bound-abs(input_data[false_index]-decp_data[true_index]))/2.0;
                        
                        
                        
                        // diff = (bound-abs(input_data[true_index]-decp_data[false_index]))/2.0;
                        if (decp_data[false_index]==decp_data[true_index]){
                            if(abs(input_data[false_index]-decp_data[false_index] - d)<=bound){
                        
                                decp_data[false_index] += d;
                        }
                       
                    }
                    // }
                    
                }
            
                else{
                    de_direction_ds[cur] = or_mini[cur];
                }
            }

            else{
                
                double diff = (bound-(input_data[false_index]-decp_data[false_index]))/2.0;
                
                if(decp_data[false_index]<=decp_data[true_index]){
                    if(abs(input_data[false_index]-((decp_data[true_index]+input_data[true_index]+bound)/2.0))<=bound){
                        decp_data[false_index] =  (decp_data[true_index]+input_data[true_index]+bound)/2.0;
                    }
                    else{
                        decp_data[false_index] = input_data[false_index] + bound;
                    }
                    while(abs(input_data[false_index]-(decp_data[false_index] + diff))>bound and diff>=2e-17){
                        diff/=2;
                    }
                    if (decp_data[false_index]==decp_data[true_index]){
                        double diff = (bound-(input_data[false_index]-decp_data[false_index]))/2.0;
                        decp_data[false_index]+=diff;
                    }
                
                }
            
                else{
                    de_direction_ds[cur] = or_mini[cur];
                }
            }
        }
    }

    return;
};
__global__ void addKernel(int* globalVar) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    // printf("hello word from thefind gpu!\n");
    if(i<=20){
        atomicAdd(globalVar, 1);
    }
    
}


// void init_or_data(std::vector<int> *a, std::vector<int> *b, std::vector<int> *c, std::vector<int> *d, std::vector<double> *input_data1, std::vector<double> *decp_data1, int num){
    
//     int* temp;
    
//     int* temp1;
//     double* temp3;
//     int* tempd;
//     int* tempd1;
//     double* temp5;

//     cudaMalloc(&temp, num * sizeof(int));
//     cudaMalloc(&temp1, num * sizeof(int));
//     cudaMalloc(&tempd, num * sizeof(int));
//     cudaMalloc(&tempd1, num * sizeof(int));
//     cudaMalloc(&temp3, num * sizeof(double));
//     cudaMalloc((void**)&temp5, num * sizeof(double));
    



//     cudaMemcpy(temp, a->data(), num * sizeof(int), cudaMemcpyHostToDevice);
//     cudaMemcpy(temp1, b->data(), num * sizeof(int), cudaMemcpyHostToDevice);
//     cudaMemcpy(tempd, c->data(), num * sizeof(int), cudaMemcpyHostToDevice);
//     cudaMemcpy(tempd1, d->data(), num * sizeof(int), cudaMemcpyHostToDevice);
//     cudaMemcpy(temp3, input_data1->data(), num * sizeof(double), cudaMemcpyHostToDevice);
//     cudaMemcpy(temp5, decp_data1->data(), num * sizeof(double), cudaMemcpyHostToDevice);

//     cudaMemcpyToSymbol(or_maxi, &temp, sizeof(int*));
//     cudaMemcpyToSymbol(or_mini, &temp1, sizeof(int*));
//     cudaMemcpyToSymbol(de_direction_as, &tempd, sizeof(int*));
//     cudaMemcpyToSymbol(de_direction_ds, &tempd1, sizeof(int*));
//     cudaMemcpyToSymbol(input_data, &temp3, sizeof(double*));
//     cudaMemcpyToSymbol(decp_data, &temp5, sizeof(double*));
//     cudaDeviceSynchronize();
    

//     dim3 blockSize(1000);
    
//     dim3 gridSize((num + blockSize.x - 1) / blockSize.x);f
    
//     int* tempDevicePtr = nullptr;
//     size_t arraySize = num*6; // 确定所需的大小
//     cudaError_t cudaStatus = cudaMalloc(&tempDevicePtr, arraySize * sizeof(int));
    
//     cudaStatus = cudaMemcpyToSymbol(adjacency, &tempDevicePtr, sizeof(tempDevicePtr));
    
//     computeAdjacency<<<gridSize, blockSize>>>(num,100,100,6);

//     cudaDeviceSynchronize();
    


    
//     iscriticle<<<gridSize,blockSize>>>(num);

    
//     cudaDeviceSynchronize();

    
    
//     int host_count_f_max;
//     cudaMemcpyFromSymbol(&host_count_f_max, count_f_max, sizeof(int), 0, cudaMemcpyDeviceToHost);
    
//     int host_count_f_min;
//     cudaMemcpyFromSymbol(&host_count_f_min, count_f_min, sizeof(int), 0, cudaMemcpyDeviceToHost);
//     // cout<<host_count_f_max<<endl;
//     while(host_count_f_max>0 or host_count_f_min>0){
        
//         // cout<<host_count_f_max<<", "<<host_count_f_min<<endl;
//         dim3 blockSize1(1000);
//         dim3 gridSize1((host_count_f_max + blockSize1.x - 1) / blockSize1.x);
        
//         fix_maxi_critical1<<<gridSize1, blockSize1>>>(0);
//         cudaDeviceSynchronize();

//         dim3 blocknum(1000);
//         dim3 gridnum((host_count_f_min + blocknum.x - 1) / blocknum.x);
        
        
//         fix_maxi_critical1<<<gridnum, blocknum>>>(1);
//         cudaDeviceSynchronize();
//         // 重新检查错误cp个数
//         int initialValue = 0;
//         cudaStatus = cudaMemcpyToSymbol(count_f_max, &initialValue, sizeof(int));
//         if (cudaStatus != cudaSuccess) {
//             std::cerr << "cudaMemcpyToSymbol failed1: " << cudaGetErrorString(cudaStatus) << std::endl;
//         }
//         // int initialValue = 0;
//         cudaStatus = cudaMemcpyToSymbol(count_f_min, &initialValue, sizeof(int));
       
//         if (cudaStatus != cudaSuccess) {
//             std::cerr << "cudaMemcpyToSymbol failed2: " << cudaGetErrorString(cudaStatus) << std::endl;
//         }

//         iscriticle<<<gridSize, blockSize>>>(num);
        
//         cudaMemcpyFromSymbol(&host_count_f_max, count_f_max, sizeof(int), 0, cudaMemcpyDeviceToHost);
        
//         cudaMemcpyFromSymbol(&host_count_f_min, count_f_min, sizeof(int), 0, cudaMemcpyDeviceToHost);
//         // cout<<host_count_f_max<<", "<<host_count_f_min<<endl;
//         cudaDeviceSynchronize();
        
        
//     }
    
    
//     cudaStatus = cudaMemcpy(decp_data1->data(), temp5, num * sizeof(double), cudaMemcpyDeviceToHost);

//     if (cudaStatus != cudaSuccess) {
//             std::cerr << "cudaMemcpyToSymbol failed3: " << cudaGetErrorString(cudaStatus) << std::endl;
//     }
//     cudaDeviceSynchronize();
    
    
//     cudaFree(temp);
//     cudaFree(temp1);
//     cudaFree(temp3);
//     cudaFree(temp5);
    
    
    

    
//     return;
    
// }
__global__ void initializeWithIndex(int size, int type=0) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int *label;
    if (index < size) {
        int *direction_ds;
        int *direction_as;
        if(type==0){
            direction_ds = de_direction_ds;
            direction_as = de_direction_as;
            label = dec_label;
        }
        else{
            
            direction_ds = or_mini;
            direction_as = or_maxi;
            label = or_label;
        }

        if(direction_ds[index]!=-1){
            label[index*2] = index;
        }
        else{
            
            label[index*2] = -1;
            // if(type==0){
            //     printf("%d\n",index);
            // }
        }

        if(direction_as[index]!=-1){
            label[index*2+1] = index;
        }
        else{
            label[index*2+1] = -1;
        }
    }
}
void init_inputdata(std::vector<int> *a,std::vector<int> *b,std::vector<int> *c,std::vector<int> *d,std::vector<double> *input_data1,std::vector<double> *decp_data1,int width1, int height1, int depth1, std::vector<int> *low,double bound1,double range,double br,std::string compressor_id,float &datatransfer,float &finddirection,float &getfcp, float &fixtime_cp, float &mappath_path,float &getfpath,float &fixtime_path){
    int* temp;
    int *un_sign_as;
    int* or_l;
    int* dec_l;
    
    hipMalloc((void**)&un_sign_as, sizeof(int));
    hipMemset(un_sign_as, 0, sizeof(int));

    int *un_sign_ds;
    hipMalloc((void**)&un_sign_ds, sizeof(int));
    hipMemset(un_sign_ds, 0, sizeof(int));
    int* temp1;
    int* d_data;
    
    

    double* temp3;
    double* temp4;
    
    int num1 = width1*height1*depth1;
    // float datatransfer = 0.0;
    hipMalloc(&or_l, num1 * 2  * sizeof(int));
    hipMalloc(&dec_l, num1 * 2 * sizeof(int));
    
    hipMemcpyToSymbol(HIP_SYMBOL(or_label), &or_l, sizeof(int*));
    hipMemcpyToSymbol(HIP_SYMBOL(dec_label), &dec_l, sizeof(int*));
    int h_un_sign_ds = num1;
    int h_un_sign_as = num1;
    float elapsedTime;
    // float find_direciton = 0.0;
    // float getfcp = 0.0;
    int initialValue = 0;
    cout<<num1<<endl;
    

    // cout<<num1<<endl;
    // size_t size = num1 * sizeof(int);
    

    hipError_t cudaStatus= hipMemcpyToSymbol(HIP_SYMBOL(width), &width1, sizeof(int), 0, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
            std::cerr << "hipMemcpyToSymbol failed101: " << hipGetErrorString(cudaStatus) << std::endl;
        }
    hipMemcpyToSymbol(HIP_SYMBOL(height), &height1, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(depth), &depth1, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(num), &num1, sizeof(int), 0, hipMemcpyHostToDevice);
    cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(bound), &bound1, sizeof(double), 0, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
            std::cerr << "hipMemcpyToSymbol failed91: " << hipGetErrorString(cudaStatus) << std::endl;
        }
    
    hipMalloc(&temp, num1 * sizeof(int));
    hipMalloc(&temp1, num1 * sizeof(int));
    cudaStatus =hipMalloc(&temp3, num1  * sizeof(double));
    if (cudaStatus != hipSuccess) {
            std::cerr << "hipMemcpyToSymbol failed89: " << hipGetErrorString(cudaStatus) << std::endl;
        }
    hipMalloc(&temp4, num1  * sizeof(double));
    hipMalloc(&d_data, num1 * sizeof(int));
    
    hipEvent_t start, stop;

    hipEventCreate(&start);

    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    

    cudaStatus = hipMemcpy(temp3, input_data1->data(), num1 * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
            std::cerr << "hipMemcpyToSymbol failed89: " << hipGetErrorString(cudaStatus) << std::endl;
        }
    cudaStatus = hipMemcpy(temp4, decp_data1->data(), num1 * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
            std::cerr << "hipMemcpyToSymbol failed17: " << hipGetErrorString(cudaStatus) << std::endl;
        }
    cudaStatus = hipMemcpy(d_data, low->data(), num1 * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
            std::cerr << "hipMemcpyToSymbol failed27: " << hipGetErrorString(cudaStatus) << std::endl;
        }
    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    
    // 计算这次迭代的时间并加到总时间上
    hipEventElapsedTime(&elapsedTime, start, stop);
    datatransfer+=elapsedTime;
    
    int *d_temp;  // 用于在主机端临时存储设备内存地址
    size_t size = num1 * sizeof(int);

    // 为设备端数组分配内存
    hipMalloc(&d_temp, size);

    // 将设备端内存地址复制到设备端全局指针
    
    hipEventRecord(start, 0);
    hipMemcpyToSymbol(HIP_SYMBOL(all_max), &d_temp, sizeof(int*));
    hipMemcpyToSymbol(HIP_SYMBOL(lowgradientindices), &d_data, sizeof(int*));
    
    int *d_temp1;  // 用于在主机端临时存储设备内存地址
    size_t size1 = num1 * sizeof(int);

    // 为设备端数组分配内存
    hipMalloc(&d_temp1, size1);

    // 将设备端内存地址复制到设备端全局指针
    hipMemcpyToSymbol(HIP_SYMBOL(all_min), &d_temp1, sizeof(int*));

    int *d_temp2;  // 用于在主机端临时存储设备内存地址
    size_t size4 = num1 * sizeof(int);
    // 为设备端数组分配内存
    hipMalloc(&d_temp2, size4);

    // 将设备端内存地址复制到设备端全局指针
    hipMemcpyToSymbol(HIP_SYMBOL(de_direction_as), &d_temp2, sizeof(int*));

    int *p_temp;  // 用于在主机端临时存储设备内存地址
    // size_t size1 = num1 * sizeof(int);

    // 为设备端数组分配内存
    hipMalloc(&p_temp, size1);

    // 将设备端内存地址复制到设备端全局指针
    hipMemcpyToSymbol(HIP_SYMBOL(all_p_min), &p_temp, sizeof(int*));

    int *p_temp1;  // 用于在主机端临时存储设备内存地址
    // size_t size1 = num1 * sizeof(int);

    // 为设备端数组分配内存
    hipMalloc(&p_temp1, size1);

    // 将设备端内存地址复制到设备端全局指针
    hipMemcpyToSymbol(HIP_SYMBOL(all_p_max), &p_temp1, sizeof(int*));

    int *d_temp3;  // 用于在主机端临时存储设备内存地址
    size_t size3 = num1 * sizeof(int);

    // 为设备端数组分配内存
    hipMalloc(&d_temp3, size3);

    // 将设备端内存地址复制到设备端全局指针
    cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(de_direction_ds), &d_temp3, sizeof(int*));
    if (cudaStatus != hipSuccess) {
            std::cerr << "hipMemcpyToSymbol failed87: " << hipGetErrorString(cudaStatus) << std::endl;
        }
    cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(or_maxi), &temp, sizeof(int*));
    if (cudaStatus != hipSuccess) {
            std::cerr << "hipMemcpyToSymbol failed83: " << hipGetErrorString(cudaStatus) << std::endl;
        }
    cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(or_mini), &temp1, sizeof(int*));
    if (cudaStatus != hipSuccess) {
            std::cerr << "hipMemcpyToSymbol failed84: " << hipGetErrorString(cudaStatus) << std::endl;
        }
    hipMemcpyToSymbol(HIP_SYMBOL(input_data), &temp3, sizeof(double*));
    hipMemcpyToSymbol(HIP_SYMBOL(decp_data), &temp4, sizeof(double*));
    
    
    dim3 blockSize(1024);
    
    dim3 gridSize((num1 + blockSize.x - 1) / blockSize.x);
    
    int* tempDevicePtr = nullptr;
    size_t arraySize = num1*12; // 确定所需的大小
    cudaStatus = hipMalloc(&tempDevicePtr, arraySize * sizeof(int));
    
    cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(adjacency), &tempDevicePtr, sizeof(tempDevicePtr));
    if (cudaStatus != hipSuccess) {
            std::cerr << "hipMemcpyToSymbol failed81: " << hipGetErrorString(cudaStatus) << std::endl;
        }
    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // 计算这次迭代的时间并加到总时间上
    hipEventElapsedTime(&elapsedTime, start, stop);
    datatransfer+=elapsedTime;
    hipEventRecord(start, 0);
    computeAdjacency<<<gridSize, blockSize>>>();
    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // 计算这次迭代的时间并加到总时间上
    hipEventElapsedTime(&elapsedTime, start, stop);
    cout<<"comupte_adjacency: "<<elapsedTime<<endl;
    // cout<<"出发"<<endl;
    hipEventRecord(start, 0);
    // for(int i =0;i<1000;i++){
    find_direction<<<gridSize, blockSize>>>(1);
    
    //     cout<<"1次finddirection: "<<elapsedTime<<endl;
    // }
    
    
    // cout<<"1000次finddirection: "<<elapsedTime<<endl;
    find_direction<<<gridSize, blockSize>>>();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    finddirection+=elapsedTime;

    hipEventRecord(start, 0);
    iscriticle<<<gridSize, blockSize>>>();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    // cout<<"出发"<<endl;
    hipDeviceSynchronize();
    getfcp+=elapsedTime;
    int host_count_f_max;
    cudaStatus = hipMemcpyFromSymbol(&host_count_f_max, HIP_SYMBOL(count_f_max), sizeof(int), 0, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipMemcpyToSymbol failed11: " << hipGetErrorString(cudaStatus) << std::endl;
    }
    int host_count_f_min;
    cudaStatus = hipMemcpyFromSymbol(&host_count_f_min, HIP_SYMBOL(count_f_min), sizeof(int), 0, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
            std::cerr << "hipMemcpyToSymbol failed12: " << hipGetErrorString(cudaStatus) << std::endl;
    }

    while(false){
        
            // cout<<host_count_f_max<<", "<<host_count_f_min<<endl;

            // cpite+=1;
            dim3 blockSize1(1024);
            dim3 gridSize1((host_count_f_max + blockSize1.x - 1) / blockSize1.x);
            // cudaEventRecord(start, 0);
            hipEventRecord(start, 0);
            fix_maxi_critical1<<<gridSize1, blockSize1>>>(0);
            
            // cudaDeviceSynchronize();

            dim3 blocknum(1024);
            dim3 gridnum((host_count_f_min + blocknum.x - 1) / blocknum.x);
            
            
            fix_maxi_critical1<<<gridnum, blocknum>>>(1);
            // cout<<"wanc"<<endl;
            hipDeviceSynchronize();
            
            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);

            // 计算这次迭代的时间并加到总时间上
            hipEventElapsedTime(&elapsedTime, start, stop);
            fixtime_cp+=elapsedTime;
            // fixtime_cp+=elapsedTime;
            // 重新检查错误cp个数
            
            cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(count_f_max), &initialValue, sizeof(int));
            // if (cudaStatus != cudaSuccess) {
            //     std::cerr << "cudaMemcpyToSymbol failed4: " << cudaGetErrorString(cudaStatus) << std::endl;
            // }
            // int initialValue = 0;
            cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(count_f_min), &initialValue, sizeof(int));
            


            // if (cudaStatus != cudaSuccess) {
            //     std::cerr << "cudaMemcpyToSymbol failed5: " << cudaGetErrorString(cudaStatus) << std::endl;
            // }
            
            // std::cout << "Average Time Per Iteration = " << elapsedTime << " ms" << std::endl;
            hipEventRecord(start, 0);

            iscriticle<<<gridSize, blockSize>>>();
            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);

            // 计算这次迭代的时间并加到总时间上
            hipEventElapsedTime(&elapsedTime, start, stop);
            getfcp+=elapsedTime;

            hipEventRecord(start, 0);
            find_direction<<<gridSize,blockSize>>>();
            
            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&elapsedTime, start, stop);
            finddirection+=elapsedTime;
            // 计算这次迭代的时间并加到总时间上
            
            
            hipMemcpyFromSymbol(&host_count_f_max, HIP_SYMBOL(count_f_max), sizeof(int), 0, hipMemcpyDeviceToHost);
            
            hipMemcpyFromSymbol(&host_count_f_min, HIP_SYMBOL(count_f_min), sizeof(int), 0, hipMemcpyDeviceToHost);
            // cout<<host_count_f_max<<", "<<host_count_f_min<<endl;
            hipDeviceSynchronize();
            
            // exit(0);
        }

    initializeWithIndex<<<gridSize, blockSize>>>(num1,0);
    initializeWithIndex<<<gridSize, blockSize>>>(num1,1);
    hipEventRecord(start, 0);
    while(h_un_sign_as>0 or h_un_sign_ds>0){
        
        int zero = 0;
        int zero1 = 0;

        // cout<<"找path"<<h_un_sign_as<<", "<<h_un_sign_ds<<endl;
        hipMemcpy(un_sign_as, &zero, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(un_sign_ds, &zero1, sizeof(int), hipMemcpyHostToDevice);
        getlabel<<<gridSize, blockSize>>>(un_sign_as,un_sign_ds,0);
        
        hipMemcpy(&h_un_sign_as, un_sign_as, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&h_un_sign_ds, un_sign_ds, sizeof(int), hipMemcpyDeviceToHost);
        
        
    }   
    
    h_un_sign_as = num1;
    h_un_sign_ds = num1;
    while(h_un_sign_as>0 or h_un_sign_ds>0){
        
        int zero = 0;
        int zero1 = 0;

        // cout<<"1找path"<<h_un_sign_as<<", "<<h_un_sign_ds<<endl;
        hipMemcpy(un_sign_as, &zero, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(un_sign_ds, &zero1, sizeof(int), hipMemcpyHostToDevice);
        getlabel<<<gridSize, blockSize>>>(un_sign_as,un_sign_ds,1);
        
        hipMemcpy(&h_un_sign_as, un_sign_as, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&h_un_sign_ds, un_sign_ds, sizeof(int), hipMemcpyDeviceToHost);
        // exit(0);
        
        
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    mappath_path+=elapsedTime;
    hipMemcpyToSymbol(HIP_SYMBOL(count_p_max), &initialValue, sizeof(int));
    
    hipMemcpyToSymbol(HIP_SYMBOL(count_p_min), &initialValue, sizeof(int));

    hipEventRecord(start, 0);
    get_wrong_index_path1<<<gridSize, blockSize>>>();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    getfpath+=elapsedTime;
    // int host_count;
    // cudaMemcpyFromSymbol(&host_count, cnt, sizeof(int), 0, cudaMemcpyDeviceToHost);
    // cout<<1-static_cast<double>(host_count)/num1<<endl;
    // std::ofstream outFilep("/pscratch/sd/y/yuxiaoli/result/S3D_rate_distortion.txt", std::ios::app);
    // // 检查文件是否成功打开
    // if (!outFilep) {
    //     std::cerr << "Unable to open file for writing." << std::endl;
    //     return; // 返回错误码
    // }
    // finddirection:0, getfcp:1,  mappath2, fixcp:3
    
    // outFilep << std::to_string(number_of_thread)<<":" << std::endl;
    // outFilep << "compressor_id: "<<compressor_id << std::endl;
    // outFilep << std::setprecision(17)<<"bound: "<<bound1 << std::endl;
    // outFilep << std::setprecision(17)<<"range: "<<range << std::endl;
    // outFilep << std::setprecision(17)<<"br: "<<br << std::endl;
    // outFilep << std::setprecision(17)<<"right_labeled_ratio: "<<1-static_cast<double>(host_count)/num1<<endl;
    // outFilep << "\n"<< std::endl;
    // exit(0);
    int host_count_p_max;
    
    cudaStatus = hipMemcpyFromSymbol(&host_count_p_max, HIP_SYMBOL(count_p_max), sizeof(int), 0, hipMemcpyDeviceToHost);
    
    int host_count_p_min;
    cudaStatus = hipMemcpyFromSymbol(&host_count_p_min, HIP_SYMBOL(count_p_min), sizeof(int), 0, hipMemcpyDeviceToHost);
    
    while(host_count_p_min>0 or host_count_p_max>0){
        cout<<host_count_p_max<<", "<<host_count_p_min<<endl;
        dim3 blockSize2(1);
        dim3 gridSize2((host_count_p_max + blockSize2.x - 1) / blockSize2.x);
        hipEventRecord(start, 0);
        fixpath11<<<gridSize2, blockSize2>>>(0);
        hipDeviceSynchronize();

        dim3 blockSize3(1);
        dim3 gridSize3((host_count_p_min + blockSize3.x - 1) / blockSize3.x);
        fixpath11<<<gridSize3, blockSize3>>>(1);
        hipDeviceSynchronize();
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsedTime, start, stop);
        fixtime_path+=elapsedTime;

        hipEventRecord(start, 0);
        find_direction<<<gridSize, blockSize>>>(0);

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsedTime, start, stop);
        finddirection+=elapsedTime;

        cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(count_f_max), &initialValue, sizeof(int));
        
        cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(count_f_min), &initialValue, sizeof(int));
            

        hipEventRecord(start, 0);

        iscriticle<<<gridSize, blockSize>>>();

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsedTime, start, stop);
        getfcp+=elapsedTime;

        hipMemcpyFromSymbol(&host_count_f_max, HIP_SYMBOL(count_f_max), sizeof(int), 0, hipMemcpyDeviceToHost);
        
        hipMemcpyFromSymbol(&host_count_f_min, HIP_SYMBOL(count_f_min), sizeof(int), 0, hipMemcpyDeviceToHost);

        while(host_count_f_max>0 or host_count_f_min>0){
        
            // cout<<host_count_f_max<<", "<<host_count_f_min<<endl;

            
            dim3 blockSize1(1024);
            dim3 gridSize1((host_count_f_max + blockSize1.x - 1) / blockSize1.x);
            // cudaEventRecord(start, 0);
            hipEventRecord(start, 0);
            fix_maxi_critical1<<<gridSize1, blockSize1>>>(0);
            
            // cudaDeviceSynchronize();

            dim3 blocknum(1024);
            dim3 gridnum((host_count_f_min + blocknum.x - 1) / blocknum.x);
            
            
            fix_maxi_critical1<<<gridnum, blocknum>>>(1);
            // cout<<"wanc"<<endl;
            hipDeviceSynchronize();
            
            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);

            // 计算这次迭代的时间并加到总时间上
            hipEventElapsedTime(&elapsedTime, start, stop);
            fixtime_cp+=elapsedTime;
            // 重新检查错误cp个数
            
            cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(count_f_max), &initialValue, sizeof(int));
            // if (cudaStatus != cudaSuccess) {
            //     std::cerr << "cudaMemcpyToSymbol failed4: " << cudaGetErrorString(cudaStatus) << std::endl;
            // }
            // int initialValue = 0;
            cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(count_f_min), &initialValue, sizeof(int));
            


            // if (cudaStatus != cudaSuccess) {
            //     std::cerr << "cudaMemcpyToSymbol failed5: " << cudaGetErrorString(cudaStatus) << std::endl;
            // }
            
            // std::cout << "Average Time Per Iteration = " << elapsedTime << " ms" << std::endl;
            hipEventRecord(start, 0);

            iscriticle<<<gridSize, blockSize>>>();
            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);

            // 计算这次迭代的时间并加到总时间上
            hipEventElapsedTime(&elapsedTime, start, stop);
            getfcp+=elapsedTime;

            hipEventRecord(start, 0);
            find_direction<<<gridSize,blockSize>>>();
            
            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&elapsedTime, start, stop);
            finddirection+=elapsedTime;
            // 计算这次迭代的时间并加到总时间上
            
            
            hipMemcpyFromSymbol(&host_count_f_max, HIP_SYMBOL(count_f_max), sizeof(int), 0, hipMemcpyDeviceToHost);
            
            hipMemcpyFromSymbol(&host_count_f_min, HIP_SYMBOL(count_f_min), sizeof(int), 0, hipMemcpyDeviceToHost);
            // cout<<host_count_f_max<<", "<<host_count_f_min<<endl;
            hipDeviceSynchronize();
            
            // exit(0);
        }

        initializeWithIndex<<<gridSize, blockSize>>>(num1,0);
        h_un_sign_as = num1;
        h_un_sign_ds = num1;
        hipEventRecord(start, 0);
        while(h_un_sign_as>0 or h_un_sign_ds>0){
        
            int zero = 0;
            int zero1 = 0;

            // cout<<"找path"<<h_un_sign_as<<", "<<h_un_sign_ds<<endl;
            hipMemcpy(un_sign_as, &zero, sizeof(int), hipMemcpyHostToDevice);
            hipMemcpy(un_sign_ds, &zero1, sizeof(int), hipMemcpyHostToDevice);
            getlabel<<<gridSize, blockSize>>>(un_sign_as,un_sign_ds,0);
            
            hipMemcpy(&h_un_sign_as, un_sign_as, sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(&h_un_sign_ds, un_sign_ds, sizeof(int), hipMemcpyDeviceToHost);
            // exit(0);
            // cout<<"找path1:"<<h_un_sign_as<<", "<<h_un_sign_ds<<endl;
            
            
        } 
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsedTime, start, stop);
        mappath_path+=elapsedTime;
        
        hipMemcpyToSymbol(HIP_SYMBOL(count_p_max), &initialValue, sizeof(int));
        hipMemcpyToSymbol(HIP_SYMBOL(count_p_min), &initialValue, sizeof(int));

        hipEventRecord(start, 0);
        get_wrong_index_path1<<<gridSize, blockSize>>>();
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsedTime, start, stop);
        getfpath+=elapsedTime;

        cudaStatus = hipMemcpyFromSymbol(&host_count_p_max, HIP_SYMBOL(count_p_max), sizeof(int), 0, hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            std::cerr << "hipMemcpyToSymbol failed11: " << hipGetErrorString(cudaStatus) << std::endl;
        }
        cudaStatus = hipMemcpyFromSymbol(&host_count_p_min, HIP_SYMBOL(count_p_min), sizeof(int), 0, hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
                std::cerr << "hipMemcpyToSymbol failed12: " << hipGetErrorString(cudaStatus) << std::endl;
        }

    }
    // cout<<host_count_p_max<<"," <<host_count_p_min<<endl;
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // 计算这次迭代的时间并加到总时间上
    hipEventElapsedTime(&elapsedTime, start, stop);
    finddirection+=elapsedTime;
    hipEventRecord(start, 0);
    hipMemcpy(a->data(), temp, num1 * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(b->data(), temp1, num1 * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(c->data(), d_temp2, num1 * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(d->data(), d_temp3, num1 * sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // 计算这次迭代的时间并加到总时间上
    hipEventElapsedTime(&elapsedTime, start, stop);
    datatransfer+=elapsedTime;
    
    cout<<"data_transfer:"<<datatransfer<<endl;
    cout<<"find_dierction: "<<find_direction<<endl;
    // cudaFree(temp);
    // cudaFree(temp1);
    // cudaFree(temp3);
    // cudaFree(tempDevicePtr);
    
    return;
}
__global__ void copyDeviceVarToDeviceMem(int *deviceMem,int *deviceMem1) {
    if (threadIdx.x == 0) {  // 只在一个线程上执行
        *deviceMem = *de_direction_as;
        *deviceMem1 = *de_direction_ds;
    }
}


void fix_process(std::vector<int> *c,std::vector<int> *d,std::vector<double> *decp_data1,float &datatransfer, float &finddirection, float &getfcp, float &fixtime_cp, int &cpite){
    auto total_start2 = std::chrono::high_resolution_clock::now();
    int num1;
    hipMemcpyFromSymbol(&num1, HIP_SYMBOL(num), sizeof(int), 0, hipMemcpyDeviceToHost);
    
    double* temp5;
    float elapsedTime;
    
    hipEvent_t start, stop;
    
    hipEventCreate(&start);
    
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    
    hipError_t cudaStatus = hipMalloc((void**)&temp5, num1 * sizeof(double));
    
    cudaStatus = hipMemcpy(temp5, decp_data1->data(), num1 * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
            std::cerr << "hipMemcpyToSymbol failed7: " << hipGetErrorString(cudaStatus) << std::endl;
    }
    
    
    cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(decp_data), &temp5, sizeof(double*));
    if (cudaStatus != hipSuccess) {
            std::cerr << "hipMemcpyToSymbol failed73: " << hipGetErrorString(cudaStatus) << std::endl;
    }
   
    
    
    
    

    hipDeviceSynchronize();
    

    
    
    int* hostArray;
    cudaStatus = hipMalloc((void**)&hostArray, num1 * sizeof(int));
    if (cudaStatus != hipSuccess) {
            std::cerr << "hipMemcpyToSymbol failed70: " << hipGetErrorString(cudaStatus) << std::endl;
        }
    
    // 从设备内存复制数据到主机内存
    hipMemcpyToSymbol(HIP_SYMBOL(de_direction_as), &hostArray, sizeof(int*));
    
    int* hostArray1;

    
    cudaStatus = hipMalloc((void**)&hostArray1, num1 * sizeof(int));
    if (cudaStatus != hipSuccess) {
            std::cerr << "hipMemcpyToSymbol failed71: " << hipGetErrorString(cudaStatus) << std::endl;
        }
    cudaStatus =  hipMemcpyToSymbol(HIP_SYMBOL(de_direction_ds), &hostArray1, sizeof(int*));
    if (cudaStatus != hipSuccess) {
            std::cerr << "hipMemcpyToSymbol failed72: " << hipGetErrorString(cudaStatus) << std::endl;
    }
    
    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    
    hipEventElapsedTime(&elapsedTime, start, stop);
    datatransfer+=elapsedTime;

    dim3 blockSize(1024);
    dim3 gridSize((num1 + blockSize.x - 1) / blockSize.x);
    hipEventRecord(start, 0);

    find_direction<<<gridSize,blockSize>>>();
    
    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    
    hipEventElapsedTime(&elapsedTime, start, stop);
    // cout<<"1000次finddirection:"<<elapsedTime<<endl;
    
    finddirection+=elapsedTime;

    hipEventRecord(start, 0);
    
    iscriticle<<<gridSize,blockSize>>>();
    
    
    
    hipDeviceSynchronize();

    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    
    hipEventElapsedTime(&elapsedTime, start, stop);
    // cout<<"1000cigetfcp: "<<elapsedTime;
    getfcp+=elapsedTime;
    
    
    int host_count_f_max;
    cudaStatus = hipMemcpyFromSymbol(&host_count_f_max, HIP_SYMBOL(count_f_max), sizeof(int), 0, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipMemcpyToSymbol failed11: " << hipGetErrorString(cudaStatus) << std::endl;
    }
    int host_count_f_min;
    cudaStatus = hipMemcpyFromSymbol(&host_count_f_min, HIP_SYMBOL(count_f_min), sizeof(int), 0, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
            std::cerr << "hipMemcpyToSymbol failed12: " << hipGetErrorString(cudaStatus) << std::endl;
    }
    // cout<<host_count_f_max<<", "<<host_count_f_min<<num1<<endl;
    // return;
    
    
    // cout<<"wrong: "<<(host_count_f_max+host_count_f_min)/num1<<endl;

    while(host_count_f_max>0 or host_count_f_min>0){
        
        // cout<<host_count_f_max<<", "<<host_count_f_min<<endl;

        cpite+=1;
        dim3 blockSize1(1024);
        dim3 gridSize1((host_count_f_max + blockSize1.x - 1) / blockSize1.x);
        // cudaEventRecord(start, 0);
        hipEventRecord(start, 0);
        fix_maxi_critical1<<<gridSize1, blockSize1>>>(0);
        
        // cudaDeviceSynchronize();

        dim3 blocknum(1024);
        dim3 gridnum((host_count_f_min + blocknum.x - 1) / blocknum.x);
        
        
        fix_maxi_critical1<<<gridnum, blocknum>>>(1);
        // cout<<"wanc"<<endl;
        hipDeviceSynchronize();
        
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);

        // 计算这次迭代的时间并加到总时间上
        hipEventElapsedTime(&elapsedTime, start, stop);
        fixtime_cp+=elapsedTime;
        // 重新检查错误cp个数
        int initialValue = 0;
        cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(count_f_max), &initialValue, sizeof(int));
        // if (cudaStatus != cudaSuccess) {
        //     std::cerr << "cudaMemcpyToSymbol failed4: " << cudaGetErrorString(cudaStatus) << std::endl;
        // }
        // int initialValue = 0;
        cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(count_f_min), &initialValue, sizeof(int));

        // if (cudaStatus != cudaSuccess) {
         //     std::cerr << "cudaMemcpyToSymbol failed5: " << cudaGetErrorString(cudaStatus) << std::endl;
        // }
        
        // std::cout << "Average Time Per Iteration = " << elapsedTime << " ms" << std::endl;
        hipEventRecord(start, 0);

        iscriticle<<<gridSize, blockSize>>>();
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);

        // 计算这次迭代的时间并加到总时间上
        hipEventElapsedTime(&elapsedTime, start, stop);
        getfcp+=elapsedTime;

        hipEventRecord(start, 0);
        find_direction<<<gridSize,blockSize>>>();
        
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsedTime, start, stop);
        finddirection+=elapsedTime;
        // 计算这次迭代的时间并加到总时间上
        
        
        hipMemcpyFromSymbol(&host_count_f_max, HIP_SYMBOL(count_f_max), sizeof(int), 0, hipMemcpyDeviceToHost);
        
        hipMemcpyFromSymbol(&host_count_f_min, HIP_SYMBOL(count_f_min), sizeof(int), 0, hipMemcpyDeviceToHost);
        // cout<<host_count_f_max<<", "<<host_count_f_min<<endl;
        hipDeviceSynchronize();
        
        // exit(0);
    }
    // cudaEventRecord(stop, 0);
    // cudaEventSynchronize(stop);
    
    hipEventRecord(start, 0);
    find_direction<<<gridSize,blockSize>>>();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    // finddirection+=elapsedTime;
    // cudaEventElapsedTime(&wholeTime, start1, stop);
    // cout<<"["<<totalElapsedTime/wholeTime<<", "<<totalElapsedTime_fcp/wholeTime<<", "<<totalElapsedTime_fd/wholeTime<<"],"<<endl;;
    // start2 = std::chrono::high_resolution_clock::now();
    hipEventRecord(start, 0);
    cudaStatus = hipMemcpy(decp_data1->data(), temp5, num1 * sizeof(double), hipMemcpyDeviceToHost);
    

    


    

    
    // cudaMemcpy(hostArray1, de_direction_ds, num1 * sizeof(int), cudaMemcpyDeviceToHost);
    hipMemcpy(c->data(), hostArray, num1 * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(d->data(), hostArray1, num1 * sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    datatransfer+=elapsedTime;
    
    hipDeviceSynchronize();
    
    // delete[] hostArray;
    // delete[] hostArray1;
    // delete[] temp5;
    hipFree(temp5);
    hipFree(hostArray);
    hipFree(hostArray1);
    // cudaFree(num1);
    
    
    // printf("%f, ",time/duration2.count());
    

    return;
    
}

__global__ void copyDeviceToArray(int* hostArray,int* hostArray1) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < num) {
        
        hostArray[index] = de_direction_as[index];
        
        hostArray1[index] = de_direction_ds[index];
    }
    
}




void mappath1(std::vector<int> *label, std::vector<int> *direction_as, std::vector<int> *direction_ds, float &finddirection, float &mappath_path, float &datatransfer,int type=0){
    int num1;
    hipMemcpyFromSymbol(&num1, HIP_SYMBOL(num), sizeof(int), 0, hipMemcpyDeviceToHost);
    
    int *un_sign_as;
    hipMalloc((void**)&un_sign_as, sizeof(int));
    hipMemset(un_sign_as, 0, sizeof(int));

    int *un_sign_ds;
    hipMalloc((void**)&un_sign_ds, sizeof(int));
    hipMemset(un_sign_ds, 0, sizeof(int));

    
    
    
    dim3 blockSize1(1024);
    dim3 gridSize1((num1 + blockSize1.x - 1) / blockSize1.x);

    float elapsedTime;
    
    hipEvent_t start, stop;

    hipEventCreate(&start);
    
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    
    int* label_temp;
    hipError_t cudaStatus = hipMalloc((void**)&label_temp, num1*2 * sizeof(int));
    if (cudaStatus != hipSuccess) {
            std::cerr << "hipMemcpyToSymbol failed60: " << hipGetErrorString(cudaStatus) << std::endl;
    }
    
    
    
    int h_un_sign_as = num1;
    int h_un_sign_ds = num1;
    // int *un_sign_as = 0;
    // int *un_sign_ds = 0;
    int* hostArray;
    cudaStatus = hipMalloc((void**)&hostArray, num1 * sizeof(int));
    // cout<<num1<<"大小"<<endl;
    // cudaMemcpy(decp_data1->data(), temp5, num1 * sizeof(double), cudaMemcpyDeviceToHost);
    
    cudaStatus = hipMemcpy(hostArray,direction_as->data(), num1 * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
            std::cerr << "hipMemcpyToSymbol failed76: " << hipGetErrorString(cudaStatus) << std::endl;
    }

    int* hostArray1;
    cudaStatus = hipMalloc((void**)&hostArray1, num1 * sizeof(int));
    cudaStatus = hipMemcpy(hostArray1,direction_ds->data(),  num1 * sizeof(int), hipMemcpyHostToDevice);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    datatransfer+=elapsedTime;

    if (cudaStatus != hipSuccess) {
        std::cerr << "hipMemcpyToSymbol failed78: " << hipGetErrorString(cudaStatus) << std::endl;
    }
    if(type==0){
        // cout<<"集哪里"<<endl;
        
        // 从设备内存复制数据到主机内存
        hipEventRecord(start, 0);
        hipMemcpyToSymbol(HIP_SYMBOL(de_direction_as), &hostArray, sizeof(int*));
        
        
        cudaStatus =  hipMemcpyToSymbol(HIP_SYMBOL(de_direction_ds), &hostArray1, sizeof(int*));
        if (cudaStatus != hipSuccess) {
                std::cerr << "hipMemcpyToSymbol failed72: " << hipGetErrorString(cudaStatus) << std::endl;
        }
        
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsedTime, start, stop);
        datatransfer+=elapsedTime;
        
    }
    hipEventRecord(start, 0);
    // for(int i=0;i<1000;i++){
    initializeWithIndex<<<gridSize1, blockSize1>>>(num1,type);
    hipDeviceSynchronize();
    
    // h_un_sign_as = num1;
    while(h_un_sign_as>0 or h_un_sign_ds>0){
        
        int zero = 0;
        int zero1 = 0;

        // cout<<"找path"<<h_un_sign_as<<", "<<h_un_sign_ds<<endl;
        hipMemcpy(un_sign_as, &zero, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(un_sign_ds, &zero1, sizeof(int), hipMemcpyHostToDevice);
        getlabel<<<gridSize1,blockSize1>>>(un_sign_as,un_sign_ds,type);
        
        hipMemcpy(&h_un_sign_as, un_sign_as, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&h_un_sign_ds, un_sign_ds, sizeof(int), hipMemcpyDeviceToHost);
        // exit(0);
        
        
    }   
        


    //     cudaDeviceSynchronize();
    // }
    hipDeviceSynchronize();
    

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    // cout<<"1000cimappath:"<<elapsedTime<<endl;
    mappath_path+=elapsedTime;

    hipEventRecord(start, 0);
    cudaStatus = hipMemcpy(label->data(), label_temp, num1 *2 * sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    datatransfer+=elapsedTime;
    if (cudaStatus != hipSuccess) {
            std::cerr << "hipMemcpyToSymbol failed61: " << hipGetErrorString(cudaStatus) << std::endl;
    }
    if(type==0){
        hipFree(label_temp);
        
    }
    
    hipFree(hostArray1);
    hipFree(hostArray);
    
    
    return;
};
