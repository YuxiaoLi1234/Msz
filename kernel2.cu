#include <iostream>
#include <float.h> 
#include <hipblas.h>

#include <fstream>
#include <sstream>
#include <vector>
#include <cstdlib>
#include <stdio.h>
#include <parallel/algorithm>  
#include <unordered_map>
#include <random>
#include <iostream>
#include <cstring> 
#include <chrono> 
#include <hip/hip_runtime.h>
#include <string>
#include <omp.h>
#include <unordered_set>
#include <set>
#include <map>
#include <algorithm>
#include <numeric>
#include <utility>
#include <iomanip>
#include <chrono>

using std::count;
using std::cout;
using std::endl;

// nvcc -c kernel2.cu -o kernel2.o
// CUDA核函数，执行向量加法
// __device__ std::vector<float> decp_data;
__device__ float* decp_data ;
__device__ float* decp_data_copy ;
__device__ int directions1[36] =  {0,1,0,0,-1,0,1,0,0,-1,0,0,-1,1,0,1,-1,0,0,0, -1,  0,-1, 1, 0,0, 1,  0,1, -1,  -1,0, 1,   1, 0,-1};
__device__ int width;
__device__ int height;
__device__ int depth;
__device__ int num;
__device__ int* adjacency;
__device__ int* all_max; 
__device__ int* all_min;
__device__ int* unsigned_n;
__device__ int count_max;
__device__ int count_min;

__device__ int count_f_max;
__device__ int count_f_min;
__device__ int* maxi;
__device__ int* mini;
__device__ float bound;
__device__ int* or_maxi;
__device__ int* or_mini;
__device__ int* lowgradientindices;
__device__ float* input_data;
__device__ int* de_direction_as;
__device__ int* de_direction_ds;
__device__ int maxNeighbors = 12;

__device__ int direction_to_index_mapping[12][3] = {{0,1,0},{0,-1,0},{1,0,0},{-1,0,0},{-1,1,0},{1,-1,0},{0,0, -1},  {0,-1, 1}, {0,0, 1},  {0,1, -1},  {-1,0, 1},   {1, 0,-1}};   


__device__ int getDirection(int x, int y, int z){
    
    for (int i = 0; i < 12; ++i) {
        if (direction_to_index_mapping[i][0] == x && direction_to_index_mapping[i][1] == y && direction_to_index_mapping[i][2] == z) {
            return i+1;  
        }
    }
    return -1;  

// 26302898,3378820
// 27930227,32438238
}
__device__ int from_direction_to_index1(int cur, int direc){
    
    if (direc==-1) return cur;
    int row = cur % height;
    int rank1 = (cur / height) % width;
    int depth1 = cur/(width * height);
    // printf("%d %d\n", row, rank1);
    if (direc >= 1 && direc <= 12) {
        int delta_row = direction_to_index_mapping[direc-1][0];
        int delta_col = direction_to_index_mapping[direc-1][1];
        int delta_dep = direction_to_index_mapping[direc-1][2];
        
        
        int next_row = row + delta_row;
        int next_col = rank1 + delta_col;
        int next_dep = depth1 + delta_dep;
        // printf("%d \n", next_row * width + next_col);
        // return next_row * width + next_col + next_dep* (height * width);
        return next_row + next_col * height + next_dep* (height * width);
    }
    else {
        return -1;
    }
    // return 0;
};

__device__ void find_direction2 (int type, int index){
    float *data;
    int *direction_as;
    int *direction_ds;
    if(type==0){
        data = decp_data;
        direction_as = de_direction_as;
        direction_ds = de_direction_ds;
    }
    else{
        data = input_data;
        direction_as = or_maxi;
        direction_ds = or_mini;
    }
    
    float mini = 0;
    
    
    // std::vector<int> indexs = adjacency[index];
    int largetst_index = index;
    
    
        
    for(int j =0;j<12;++j){
        int i = adjacency[index*12+j];
        
        if(i==-1){
            break;
        }
        if(lowgradientindices[i]==1){
            continue;
        }
        if((data[i]>data[largetst_index] or (data[i]==data[largetst_index] and i>largetst_index))){
            mini = data[i]-data[index];
            
            largetst_index = i;
            // }
            
        };
    };
    int row_l = (largetst_index / (height)) % width;
    int row_i = (index / (height)) % width;
    
    int col_diff = row_l - row_i;
    int row_diff = (largetst_index % height) - (index % height);

    int dep_diff = (largetst_index /(width * height)) - (index /(width * height));
    direction_as[index] = getDirection(row_diff, col_diff,dep_diff);
    // if(index==8058 and type==0){
    //     printf("%d %d \n" ,direction_ds[index],or_mini[index]);
    // }
    
    

    mini = 0;
    largetst_index = index;
    for(int j =0;j<12;++j){
        int i = adjacency[index*12+j];
        
        if(i==-1){
            break;
        }
        if(lowgradientindices[i]==1){
            continue;
        }
        // if(i==8186 and index==8058 and type==0){
        //     printf("%.20f %.20f\n",data[i]-data[index],data[8057]-data[index]);
        //     // cout<<data[i]<<", "<<data[index]<<", "<<data[8057]<<endl;
        // }
        if((data[i]<data[largetst_index] or (data[i]==data[largetst_index] and i<largetst_index))){
            mini = data[i]-data[index];
            
            largetst_index = i;

            
        };
    };
    
    row_l = (largetst_index / (height)) % width;
    row_i = (index / (height)) % width;
    
    col_diff = row_l - row_i;
    row_diff = (largetst_index % height) - (index % height);

    dep_diff = (largetst_index /(width * height)) - (index /(width * height));
    // row_l = (largetst_index % (height * width)) / width;
    // row_i = (index % (height * width)) / width;
    
    // row_diff = row_l - row_i;
    // col_diff = (largetst_index % width) - (index % width);

    // dep_diff = (largetst_index /(width * height)) - (index /(width * height));
    
    direction_ds[index] = getDirection(row_diff, col_diff,dep_diff);
    
    
    
}
__global__ void find_direction (int type=0){
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if(index>=num or lowgradientindices[index]==1){
        return;
    }
    
    float *data;
    int *direction_as;
    int *direction_ds;
    if(type==0){
        data = decp_data;
        direction_as = de_direction_as;
        direction_ds = de_direction_ds;
    }
    else{
        data = input_data;
        direction_as = or_maxi;
        direction_ds = or_mini;
    }
    
    float mini = 0;
    
    
    // std::vector<int> indexs = adjacency[index];
    int largetst_index = index;
    
    
        
    for(int j =0;j<12;++j){
        int i = adjacency[index*12+j];
        
        if(i==-1){
            break;
        }
        if(lowgradientindices[i]==1){
            continue;
        }
        if((data[i]>data[largetst_index] or (data[i]==data[largetst_index] and i>largetst_index))){
            mini = data[i]-data[index];
            
            largetst_index = i;
            // }
            
        };
    };
    // int row_l = (largetst_index % (height * width)) / width;
    // int row_i = (index % (height * width)) / width;
    
    // int row_diff = row_l - row_i;
    // int col_diff = (largetst_index % width) - (index % width);

    // int dep_diff = (largetst_index /(width * height)) - (index /(width * height));
    int row_l = (largetst_index / (height)) % width;
    int row_i = (index / (height)) % width;
    
    int col_diff = row_l - row_i;
    int row_diff = (largetst_index % height) - (index % height);

    int dep_diff = (largetst_index /(width * height)) - (index /(width * height));
    direction_as[index] = getDirection(row_diff, col_diff,dep_diff);
    
    // if(index==24654784 and type==0){
        
    //     printf("值：");
    //     printf("%d %d %d\n",row_diff, col_diff,dep_diff);
    //     printf("%d %d \n", largetst_index % 750, index % 750);
    //     // printf("%f %f \n" ,decp_data[index],input_data[index]);
    //     // for(int i=0;i<12;i++){
    //     //     int j = adjacency[index*12+i];
    //     //     if(j==-1){
    //     //         break;
    //     //     }
    //     //     printf("%f %f \n" ,decp_data[j],input_data[j]);
    //     // }
        
    // }
    
    

    mini = 0;
    largetst_index = index;
    for(int j =0;j<12;++j){
        int i = adjacency[index*12+j];
        
        if(i==-1){
            break;
        }
        if(lowgradientindices[i]==1){
            continue;
        }
        // if(i==8186 and index==8058 and type==0){
        //     printf("%.20f %.20f\n",data[i]-data[index],data[8057]-data[index]);
        //     // cout<<data[i]<<", "<<data[index]<<", "<<data[8057]<<endl;
        // }
        if((data[i]<data[largetst_index] or (data[i]==data[largetst_index] and i<largetst_index))){
            mini = data[i]-data[index];
            
            largetst_index = i;

            
        };
    };
    
    
    // row_l = (largetst_index % (height * width)) / width;
    // row_i = (index % (height * width)) / width;
    
    // row_diff = row_l - row_i;
    // col_diff = (largetst_index % width) - (index % width);

    // dep_diff = (largetst_index /(width * height)) - (index /(width * height));
    row_l = (largetst_index / (height)) % width;
    row_i = (index / (height)) % width;
    
    col_diff = row_l - row_i;
    row_diff = (largetst_index % height) - (index % height);

    dep_diff = (largetst_index /(width * height)) - (index /(width * height));
    direction_ds[index] = getDirection(row_diff, col_diff,dep_diff);
    
    
    
    
    
    return;

};
__global__ void checkElementKernel(int* array, int size, int target, bool* result) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < size) {
        if (array[idx] == target) {
            *result = true;
        }
    }
}

__global__ void iscriticle(){
        
        int i = threadIdx.x + blockIdx.x * blockDim.x;
        
        if(i>=num or lowgradientindices[i]==1){
            
            return;
        }
        
        bool is_maxima = true;
        bool is_minima = true;
        
        for (int index=0;index<12;index++) {
            int j = adjacency[i*12+index];
            if(j==-1){
                break;
            }
            if(lowgradientindices[j]==1){
                continue;
            }
            
                
            if (decp_data[j] > decp_data[i]) {
                
                is_maxima = false;
                
                break;
            }
            else if(decp_data[j] == decp_data[i] and j>i){
                is_maxima = false;
                break;
            }
        }
        for (int index=0;index< 12;index++) {
            int j = adjacency[i*12+index];
            if(j==-1){
                break;
            }
            if(lowgradientindices[j]==1){
                    continue;
            }
            
            if (decp_data[j] < decp_data[i]) {
                is_minima = false;
                break;
            }
            else if(decp_data[j] == decp_data[i] and j<i){
                is_minima = false;
                break;
            }
        }
        
        
        if((is_maxima && or_maxi[i]!=-1) or (!is_maxima && or_maxi[i]==-1)){
            int idx_fp_max = atomicAdd(&count_f_max, 1);
            // if(i==6345199){
            //     printf("%d %d \n",is_maxima,or_maxi[i]);
            // }
            all_max[idx_fp_max] = i;
            
        }
        
        else if ((is_minima && or_mini[i]!=-1) or (!is_minima && or_mini[i]==-1)) {
            int idx_fp_min = atomicAdd(&count_f_min, 1);// in one instruction
            
            all_min[idx_fp_min] = i;
            
        } 
        
       
        
}

__global__ void freeDeviceMemory() {
    // 释放 decp_data 指向的内存
    if (decp_data != nullptr) {
        delete[] decp_data;
        decp_data = nullptr;  // 避免野指针
    }
} 
__global__ void freeDeviceMemory1() {
    // 释放 decp_data 指向的内存
    if (de_direction_as != nullptr) {
        delete[] de_direction_as;
        de_direction_as = nullptr;  // 避免野指针
    }
    if (de_direction_ds != nullptr) {
        delete[] de_direction_ds;
        de_direction_ds = nullptr;  // 避免野指针
    }
}
__global__ void computeAdjacency() {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i < num and lowgradientindices[i]==0) {
        
        int y = (i / (height)) % width; // Get the x coordinate
        int x = i % height; // Get the y coordinate
        int z = i / (width * height);
        int neighborIdx = 0;
        
        for (int d = 0; d < 12; d++) {
            
            int dirX = directions1[d * 3];     
            int dirY = directions1[d * 3 + 1]; 
            int dirZ = directions1[d * 3 + 2]; 
            int newX = x + dirX;
            int newY = y + dirY;
            int newZ = z + dirZ;
            int r = newX + newY * height + newZ* (height * width); // Calculate the index of the adjacent vertex
            // if(lowgradientindices[r]==1){
            //     continue;
            // }
            if (newX >= 0 && newX < height && newY >= 0 && newY < width && r < width*height*depth && newZ<depth && newZ>=0 && lowgradientindices[r]==0) {
                
                adjacency[i * maxNeighbors + neighborIdx] = r;
                neighborIdx++;

            }
        }

        // Fill the remaining slots with -1 or another placeholder value
        
        for (int j = neighborIdx; j < maxNeighbors; ++j) {
            adjacency[i * maxNeighbors + j] = -1;
        }
    }
}

__global__ void allocateDeviceMemory() {
    if (threadIdx.x == 0) { // 仅在一个线程上执行
        // printf("%d %d \n", threadIdx.x,num );
        all_max = new int[num];
        
        all_min = new int[num];
    }
    return;
}



// __global__ void fix_maxi_critical1(int direction){
//     int index_f = blockIdx.x * blockDim.x + threadIdx.x;
    
//     if (direction == 0 && index_f<count_f_max && lowgradientindices[all_max[index_f]]==0){
        
//         int index = all_max[index_f];
        
//         if (or_maxi[index]!=-1){
            
//             int next_vertex = from_direction_to_index1(index,or_maxi[index]);
            
//             int smallest_vertex = next_vertex;
//             float threshold = -DBL_MAX;
            
            
//             for(int j=0;j<12;j++){
//                 int i = adjacency[index*12+j];
//                 if(lowgradientindices[i]==1){
//                     continue;
//                 }
//                 if(i==-1){
//                     break;
//                 }
//                 if(input_data[i]<input_data[index] and input_data[i]>threshold and i!=next_vertex){
//                     smallest_vertex = i;
//                     threshold = input_data[i];
//                 }
//             }
            
//             threshold = decp_data[smallest_vertex];
//             // float diff = (bound - (input_data[next_vertex]-decp_data[index]))/2.0;
//             float d = (bound - (input_data[index]-decp_data[index]))/2.0;
            
//             if(decp_data[index]<decp_data[next_vertex] or (decp_data[index]==decp_data[next_vertex] and index<next_vertex)){
                
//                 de_direction_as[index]=or_maxi[index];
                
//                 return;
//             }
            
//             if(d>=1e-6){
                
//                 if(decp_data[index]==decp_data[next_vertex])
//                     {
                        
//                         while(abs(input_data[index]-decp_data[index]+d)>bound and d>=2e-16){
//                             d/=2;
//                         }
//                         if (abs(input_data[index]-decp_data[index]+d)<=bound){
//                             decp_data[index] -= d;
//                         }

                    
//                     }
//                 else{
//                     if(decp_data[index]>=decp_data[next_vertex]){
                        
//                         while(abs(input_data[index]-decp_data[index]+d)>bound and d>=2e-16){
//                                 d/=2;
//                         }
                        
//                         if(decp_data[index]>=threshold and threshold<=decp_data[next_vertex]){
                            
//                             while(decp_data[index] - d < threshold and d>=2e-16)
//                             {
//                                 d/=2;
//                             }
                            
                            
//                         }
//                         else if(threshold>decp_data[next_vertex]){
                            
                            
//                             float diff2 = (bound-(input_data[smallest_vertex]-decp_data[smallest_vertex]))/2;
                            
//                             if(diff2>1e-6){
//                                 while(abs(input_data[smallest_vertex]-decp_data[smallest_vertex]+diff2)>bound and diff2>=2e-16 and decp_data[smallest_vertex]-diff2>decp_data[next_vertex]){
                                    
//                                     diff2/=2;
//                                 }
                                
//                                 if(abs(input_data[smallest_vertex]-decp_data[smallest_vertex]+diff2)<=bound){
//                                     decp_data[smallest_vertex]-=diff2;
//                                     // if(index==97) cout<<"处理97的时候: "<<decp_data[next_vertex]<<", "<<decp_data[index]<<endl;
//                                 }
                                
                                
//                             }
                            
//                         }

//                         if(abs(input_data[index]-(decp_data[index]-d))<=bound and decp_data[index]>=decp_data[next_vertex]){
//                             decp_data[index] -= d;
                            
//                         }
                        
                        
                   
//                 };
//                      }

                 
            
                
//             }
//             else{
                
//                 if(decp_data[index]>=decp_data[next_vertex]){
//                     if(abs(input_data[index]-(input_data[next_vertex] -bound+ decp_data[index])/2.0)<=bound){
//                         decp_data[index] = (input_data[next_vertex] -bound + decp_data[index])/2.0;
//                     }
//                     else{
                        
//                         decp_data[index] = input_data[index] - bound;
//                     }
                    
//                 }
                
//             }
            
            
        
//         }
//         else{
//             // printf("%d \n",or_maxi[index]);
//             int largest_index = from_direction_to_index1(index,de_direction_as[index]);
//             // float diff = (bound-(input_data[index]-decp_data[index]))/2.0;
//             float d = (bound-(input_data[largest_index]-decp_data[index]))/2.0;
//             if(decp_data[index]>decp_data[largest_index] or(decp_data[index]==decp_data[largest_index] and index>largest_index)){
//                 de_direction_as[index] = -1;
//             }
//             if(d>=1e-6){
                
//                 if (decp_data[index]<=decp_data[largest_index]){
//                     if(abs(input_data[largest_index]-decp_data[index]+d)){
//                         decp_data[largest_index] = decp_data[index]-d;
//                     }
//                 }
                
            
                
//             }
            
//             else{
//                 if(decp_data[index]<=decp_data[largest_index]){
//                     decp_data[index] = input_data[index] + bound;
//                 }
                    
//             }
            
//         }
        
        
    
//     }
    
//     else if(direction == 1 && index_f<count_f_min && lowgradientindices[all_min[index_f]]==0){
//         int index = all_min[index_f];
//         if (or_mini[index]!=-1){
//             int next_vertex= from_direction_to_index1(index,or_mini[index]);
            
//             float diff = (bound - (input_data[next_vertex]-decp_data[index]))/2.0;
//             float d =  (bound-(input_data[index]-decp_data[index]))/2.0;
            
            
//             if(decp_data[index]>decp_data[next_vertex] or (decp_data[index]==decp_data[next_vertex] and index>next_vertex)){
//                 de_direction_ds[index]=or_mini[index];
//                 return;
//             }
            
//             if(diff>=1e-6 or d>=1e-6){
//                 if(decp_data[index]==decp_data[next_vertex]){
                    
                    
//                         while(abs(input_data[next_vertex]-decp_data[index]-d)>bound and d>=2e-16){
//                             d/=2;
//                         }
                        
//                         if(abs(input_data[index]-decp_data[index]-d)<=bound){
//                             decp_data[index]+=d;
//                         }
                    
                    
                    
                    
//                 }
//                 else{
//                     if(decp_data[index]<=decp_data[next_vertex]){
                        
//                             while(abs(input_data[next_vertex]-decp_data[index]+diff)>bound and diff >= 2e-16){
//                                     diff/=2;
//                             }
                            
//                             if (abs(input_data[next_vertex]-decp_data[index]+d)<=bound and decp_data[index]<=decp_data[next_vertex]){
//                                 decp_data[next_vertex] = decp_data[index]-diff;
//                             }
                            
                            
                        
                        
//                 };

//                 }
                
                

                
//             }

//             else{
                
//                 if(decp_data[index]<=decp_data[next_vertex]){
//                     if(abs(input_data[index]-(input_data[next_vertex] + bound + decp_data[index])/2.0)<=bound){
//                         decp_data[index] = (input_data[next_vertex] + bound + decp_data[index])/2.0;
//                     }
//                     else{
//                         decp_data[index] = input_data[index] + bound;
//                     }
//                 }
//             }
            

            
            
            

            
        
//         }
    
//         else{
            
//             int largest_index = from_direction_to_index1(index,de_direction_ds[index]);
//             float diff = (bound-(input_data[index]-decp_data[index]))/2.0;
//             // float d = (bound-(input_data[largest_index]-decp_data[index]))/2.0;
            
//             if(decp_data[index]<decp_data[largest_index] or (decp_data[index]==decp_data[largest_index] and index<largest_index)){
//                 de_direction_ds[index] = -1;
//                 return;
//             }
            
//             if (diff>=1e-6){
//                 if (decp_data[index]>=decp_data[largest_index]){
//                     while(abs(input_data[index]-decp_data[index]+diff)>bound and diff>=2e-16){
//                         diff/=2;
//                     }
                    
                    
//                     if(abs(input_data[index]-decp_data[index]+diff)<=bound){
//                         decp_data[index] -= diff;
//                     }
                    
                    
//                 }                    
//             }
            
                    
//             else{
//                 if (decp_data[index]>=decp_data[largest_index]){
//                     decp_data[index] = input_data[index] - bound;
//                 }   
    
//             }


               
//         }

        
//     }    
//     return;
// };
__global__ void fix_maxi_critical1(int direction){
    int index_f = blockIdx.x * blockDim.x + threadIdx.x;
    
    
        
    
    if (direction == 0 && index_f<count_f_max && lowgradientindices[all_max[index_f]]==0){
        
        int index = all_max[index_f];
      
	// printf("%d\n",index);
        if (or_maxi[index]!=-1){
            // printf("%d\n",index);
            // find_direction2(1,index);
            int next_vertex = from_direction_to_index1(index,or_maxi[index]);
            
            int smallest_vertex = next_vertex;
            float threshold = -DBL_MAX;
            
            
            for(int j=0;j<12;j++){
                int i = adjacency[index*12+j];
                if(i==-1){
                    break;
                }
                if(lowgradientindices[i]==1){
                    continue;
                }
                
                if(input_data[i]<input_data[index] and input_data[i]>threshold and i!=next_vertex){
                    smallest_vertex = i;
                    threshold = input_data[i];
                }
            }
            
            threshold = decp_data[smallest_vertex];
            // 对的
            // d是把index还要降低
            // 如果是tthresh的话，那它的下限就是：input_data[index]-(abs(inaput_data[index]-decp_data[index]))
            // 之前的
            // float diff = (bound - (input_data[next_vertex]-decp_data[index]))/2.0;
            // float d = (decp_data[index] - input_data[index] + bound )/2.0;
            // // float d = (decp_data[index]-(input_data[index]-(abs(input_data[index]-decp_data_copy[index]))))/2.0;
            // float d1 = ((input_data[next_vertex] + bound) - decp_data[next_vertex])/2.0;
            // float diff1 = (bound - (input_data[next_vertex]-decp_data[index]))/2.0;
            float diff = (bound - (input_data[next_vertex]-decp_data[index]))/2.0;
            float d = (decp_data[index] - input_data[index] + bound )/2.0;
            float d1 = ((input_data[next_vertex] + bound) - decp_data[next_vertex])/2.0;
            float diff1 = (bound - (input_data[next_vertex]-decp_data[index]))/2.0;
            // float diff = ((input_data[next_vertex]-decp_data_copy[next_vertex]) - (input_data[next_vertex]-decp_data[index]))/2.0;
            // float d = (decp_data[index] - input_data[index] + (input_data[index]-decp_data_copy[index]))/2.0;
            // float d1 = ((input_data[next_vertex] + (input_data[next_vertex]-decp_data_copy[next_vertex])) - decp_data[next_vertex])/2.0;
            // float diff1 = ((input_data[next_vertex]-decp_data_copy[next_vertex]) - (input_data[next_vertex]-decp_data[index]))/2.0;
            if(count_f_max==3 and count_f_min==0 and index==123716372){
                 printf("改变前");
                 printf("%d, %f\n", index, decp_data[index]);
                 printf("%d %f\n", next_vertex, decp_data[next_vertex]);
                 printf("%f %f \n",diff, d);
                 printf("%f %f \n",decp_data[index], decp_data[index]-d);
            //     // cout<<"index: "<<index<<", "<<decp_data[index]<<endl;
            //     // cout<<"next_vertex: "<<next_vertex<<","<<decp_data[next_vertex]<<endl;
            //     // cout<<"smallest_vertex: "<<smallest_vertex<<", "<<decp_data[smallest_vertex]<<endl;
            //     // cout<<"diff: "<<d<<","<<"d: "<<d<<endl;
             }
            
            if(decp_data[index]<decp_data[next_vertex] or (decp_data[index]==decp_data[next_vertex] and index<next_vertex)){
                
                de_direction_as[index]=or_maxi[index];
            
                return;
            }
            
            if(d>=1e-6){
                
                if(decp_data[index]==decp_data[next_vertex])
                    {
                        
                        while(abs(input_data[index]-decp_data[index]+d)>bound and d>=2e-6){
                            d/=2;
                        }
                        if (abs(input_data[index]-decp_data[index]+d)<=bound){
                            decp_data[index] -= d;
                        }
                    }
                else{
                    if(decp_data[index]>=decp_data[next_vertex]){
                        
                        while(abs(input_data[index]-decp_data[index]+d)>bound and d>=2e-6){
                                d/=2;
                        }
                        
                        if(decp_data[index]>=threshold and threshold<=decp_data[next_vertex]){
                            
                            while(decp_data[index] - d < threshold and d>=2e-6)
                            {
                                d/=2;
                            }
                            
                            
                        }
                        // else if(threshold>decp_data[next_vertex]){
                            
                            
                        //     float diff2 = (bound-(input_data[smallest_vertex]-decp_data[smallest_vertex]))/64;
                            
                        //     if(diff2>=1e-6){
                        //         while(abs(input_data[smallest_vertex]-decp_data[smallest_vertex]+diff2)>bound and diff2>=2e-16 and decp_data[smallest_vertex]-diff2>decp_data[next_vertex]){
                                    
                        //             diff2/=2;
                        //         }
                                
                        //         if(abs(input_data[smallest_vertex]-decp_data[smallest_vertex]+diff2)<=bound){
                        //             if(smallest_vertex==66783){cout<<"在这里11."<<endl;}
                        //             decp_data[smallest_vertex]-=diff2;
                        //             // if(index==97) cout<<"处理97的时候: "<<decp_data[next_vertex]<<", "<<decp_data[index]<<endl;
                        //         }
                                
                                
                        //     }
                            
                        // }

                        if(abs(input_data[index]-(decp_data[index]-d))<=bound and decp_data[index]>=decp_data[next_vertex] and d>=1e-6){
                            // if(index==1620477){
                            //     // cout<<"next_vertex: "<<decp_data[next_vertex]<<endl;
                            //     // cout<<"smallest_vertex: "<<decp_data[smallest_vertex]<<endl;
                            //     cout<<"before index: "<<decp_data[index]<<endl;
                                
                            // }
                            
                            decp_data[index] -= d;
                            
                            
                                            
                        }
                        // else if(abs(input_data[next_vertex]-(decp_data[next_vertex]+d1))<=bound and decp_data[index]>=decp_data[next_vertex] and d1>0){
                        //     // if(index==1620477){
                        //     //     // cout<<"next_vertex: "<<decp_data[next_vertex]<<endl;
                        //     //     // cout<<"smallest_vertex: "<<decp_data[smallest_vertex]<<endl;
                        //     //     cout<<"before index: "<<decp_data[index]<<endl;
                                
                        //     // }
                            
                        //     decp_data[next_vertex] += d1;
                            
                            
                                            
                        // }
                        
                        
                   
                };
                     }
            if(count_f_max==3 and count_f_min==0 and index==123716372){
                 printf("改变后");
                 printf("%d, %f\n", index, decp_data[index]);
                 printf("%d %f\n", next_vertex, decp_data[next_vertex]);
                 printf("%f %f \n",diff, d);
                 printf("%f %f \n",decp_data[index], decp_data[index]-d);
            //     // cout<<"index: "<<index<<", "<<decp_data[index]<<endl;
            //     // cout<<"next_vertex: "<<next_vertex<<","<<decp_data[next_vertex]<<endl;
            //     // cout<<"smallest_vertex: "<<smallest_vertex<<", "<<decp_data[smallest_vertex]<<endl;
            //     // cout<<"diff: "<<d<<","<<"d: "<<d<<endl;
             }
                 
            
                
            }
            else{
                
                if(decp_data[index]>decp_data[next_vertex]){
                    
                    if(abs(input_data[index]-decp_data[next_vertex])<bound){
                            
                            float t = (decp_data[next_vertex]-(input_data[index]-bound))/2.0;
                            decp_data[index] = decp_data[next_vertex] - t;
                            // decp_data[next_vertex] = t;
                        }
                    else{
                        decp_data[index] = input_data[index] - bound;
                        
                    }
                    
                }
                else if(decp_data[index]==decp_data[next_vertex]){
                    // float bound1 = abs(input_data[index]-decp_data[index]);
                    //
                    float d = (bound - (input_data[index]-decp_data[index]))/2.0;
                    // while(abs(input_data[index]-decp_data[index]-d)>bound and d>=2e-16){
                    //         d/=2;
                    // }
                    // if(index==157569){
                    //     cout<<"在这时候d: "<<d<<endl;
                    // }   
                    // float d = 1e-6;
                    if(abs(input_data[index]-decp_data[index]+d)<=bound){
                        
                        decp_data[index]-=d;
                    }
                    else if(abs(input_data[next_vertex]-decp_data[next_vertex]-d)<=bound){
                        // if(next_vertex==78){cout<<"在这里21"<<endl;}
                        decp_data[next_vertex]+=d;
                    }
                }
                
            }
            
            
        
        }
        else{
            // if(index==25026 and count_f_max<=770){
            //     cout<<"在这里"<<endl;
            // }
            // find_direction2(0,index);
            int largest_index = from_direction_to_index1(index,de_direction_as[index]);
            // 对的
            float diff = (bound-(input_data[index]-decp_data[index]))/2.0;
            float d = (bound-(input_data[largest_index]-decp_data[index]))/2.0;
            // float diff = (input_data[index]-decp_data[index])/2.0;
            // float d = (input_data[largest_index]-decp_data[index])/2.0;
            // float d1 = ((input_data[next_vertex] + (input_data[next_vertex]-decp_data_copy[next_vertex])) - decp_data[next_vertex])/2.0;
            // float diff1 = ((input_data[next_vertex]-decp_data_copy[next_vertex]) - (input_data[next_vertex]-decp_data[index]))/2.0;
            // if(index==25026 and count_f_max<=770){
            //     cout<<"改变前"<<endl;
            //     cout<<"index: "<<index<<", "<<decp_data[index]<<endl;
            //     cout<<"next_vertex: "<<largest_index<<","<<decp_data[largest_index]<<endl;
            //     // cout<<"smallest_vertex: "<<smallest_vertex<<", "<<decp_data[smallest_vertex]<<endl;
            //     cout<<"diff: "<<d<<","<<"d: "<<d<<endl;
            //     cout<<or_direction_as[25026]<<de_direction_as[25026]<<endl;
            // }
            
            if(decp_data[index]>decp_data[largest_index] or(decp_data[index]==decp_data[largest_index] and index>largest_index)){
                de_direction_as[index] = -1;
            }
            if(d>=1e-6){
                
                if (decp_data[index]<=decp_data[largest_index]){
                    if(abs(input_data[largest_index]-decp_data[index]+d)){
                        // if(largest_index==66783){cout<<"在这里17"<<endl;}
                        decp_data[largest_index] = decp_data[index]-d;
                    }
                }
                
            
                
            }
            
            else{
                if(decp_data[index]<=decp_data[largest_index]){
                    // if(index==78){
                    //         cout<<"在这里1"<<endl;
                    //     }
                    decp_data[index] = input_data[index] + bound;
                }
                    
            }

            // if(index==15885 and count_f_max==7){
            //     cout<<"改变后"<<endl;
            //     cout<<"index: "<<index<<", "<<decp_data[index]<<endl;
            //     cout<<"next_vertex: "<<largest_index<<","<<decp_data[largest_index]<<endl;
            //     // cout<<"smallest_vertex: "<<smallest_vertex<<", "<<decp_data[smallest_vertex]<<endl;
            //     cout<<"diff: "<<d<<","<<"d: "<<d<<endl;
            // }
            
        }
        
        
    
    }
    
    else if (direction != 0 && index_f<count_f_min && lowgradientindices[all_min[index_f]]==0){
        int index = all_min[index_f];
        if (or_mini[index]!=-1){
            // find_direction2(1,index);
            int next_vertex= from_direction_to_index1(index,or_mini[index]);
            
            float diff = (bound - (input_data[next_vertex]-decp_data[index]))/2.0;
            float d =  (bound+input_data[index]-decp_data[index])/2.0;
            // float d1 =  (bound-(input_data[next_vertex]-decp_data[next_vertex]))/2.0;
            
            float d1 = (decp_data[next_vertex]-input_data[next_vertex]+bound)/2.0;
            if(decp_data[index]>decp_data[next_vertex] or (decp_data[index]==decp_data[next_vertex] and index>next_vertex)){
                de_direction_ds[index]=or_mini[index];
                return;
            }

            // if(index == 6595 and count_f_min==5){
            //     cout<<"下降："<<endl;
            //     cout<<"next: "<<next_vertex<<", "<<decp_data[next_vertex]<<endl;
            //     cout<<"index: "<<index<<", "<<decp_data[index]<<endl;
            //     cout<<"daxiaoguanxi: "<<decp_data[index]-decp_data[next_vertex]<<endl;
            //     cout<<"diff: "<<diff<<endl;
            //     cout<<"d: "<<d<<endl;
            //     cout<<"d1: "<<d1<<endl;
            // }
            // if(count_f_max==0 and count_f_min==4){
            //      printf("改变前");
            //      printf("%d, %f\n", index, decp_data[index]);
            //      printf("%d %f\n", next_vertex, decp_data[next_vertex]);
            //      printf("%f %f \n",diff, d);
            //      printf("%f %f \n",decp_data[index], decp_data[index]-d);
            // //     // cout<<"index: "<<index<<", "<<decp_data[index]<<endl;
            // //     // cout<<"next_vertex: "<<next_vertex<<","<<decp_data[next_vertex]<<endl;
            // //     // cout<<"smallest_vertex: "<<smallest_vertex<<", "<<decp_data[smallest_vertex]<<endl;
            // //     // cout<<"diff: "<<d<<","<<"d: "<<d<<endl;
            //  }
            
            if(diff>=1e-6 or d>=1e-6 or d1>=1e-6){
                
                if(decp_data[index]==decp_data[next_vertex]){
                    
                      
                    
                        while(abs(input_data[next_vertex]-decp_data[index]-diff)>bound and diff>=2e-6){
                            diff/=2;
                        }
                        
                        if(abs(input_data[next_vertex]-decp_data[index]+diff)<=bound and diff>=1e-6){
                            // if(index==344033 and count_f_min==2){cout<<"在这里22"<<d<<endl;}
                            decp_data[next_vertex]= decp_data[index]-diff;
                        }
                        else if(d1>=1e-6){
                            // if(index==344033 and count_f_min==2){cout<<"在这里23"<<d<<endl;}
                            decp_data[next_vertex]-=d1;
                        }
                        else if(d>=1e-6){
                            // if(index==344033 and count_f_min==2){cout<<"在这里24"<<d<<endl;}
                            decp_data[index]+=d;
                        }

                    
                    
                }
                else{
                    if(decp_data[index]<=decp_data[next_vertex]){
                        
                            while(abs(input_data[next_vertex]-decp_data[index]+diff)>bound and diff >= 2e-6){
                                    diff/=2;
                            }
                            
                            
                            if (abs(input_data[next_vertex]-decp_data[index]+diff)<=bound and decp_data[index]<=decp_data[next_vertex] and diff>=1e-6){
                                // while(abs(input_data[next_vertex]-decp_data[index]+diff)<=bound and diff<1e-7){
                                //     diff*=2;
                                // }
                                // if(index==270808 and count_f_min==1){cout<<"在这里2！"<< endl;}
                                while(abs(input_data[next_vertex]-decp_data[index]+diff)<bound and diff <= 1e-6){
                                    diff*=2;
                                }
                                if(abs(input_data[next_vertex]-decp_data[index]+diff)<=bound){
                                    decp_data[next_vertex] = decp_data[index]-diff;
                                }
                                // if(index == 6595 and count_f_min==5){
                                //     cout<<"在这里1！"<< diff <<", "<<index<<", "<<decp_data[index]<<","<<input_data[index]<<","<<input_data[next_vertex]<<endl;

                                // }
                                // if(next_vertex==66783){cout<<"在这里13"<<endl;}
                                // decp_data[next_vertex] = decp_data[index]-diff;
                                // if(index==89797){
                                //         cout<<"在这里2"<<diff<<", "<<d<<endl;
                                // }

                                // decp_data[index]+=d;
                            }
                            // else if(abs(input_data[index]-decp_data[index]-d)<=bound and decp_data[index]<=decp_data[next_vertex] and d>0){
                            //     if(index==135569){cout<<"在这里23"<<endl;}
                            //     decp_data[index]+=d;
                            // }
                            else if(abs(input_data[next_vertex]-decp_data[next_vertex]+d1)<=bound and decp_data[index]<=decp_data[next_vertex] and d1>=1e-6){
                                while(abs(input_data[next_vertex]-decp_data[next_vertex]+d1)<bound and d1<=1e-6){
                                    d1*=2;
                                }
                                // if(count_f_min<=12){cout<<"在这里2！"<<abs(input_data[next_vertex]-decp_data[next_vertex]+d1)<<"," <<d1<< endl;}
                                if(abs(input_data[next_vertex]-decp_data[next_vertex]+d1)<=bound and d1>=1e-6){
                                    decp_data[next_vertex]-=d1;
                                }
                                // else{
                                //     decp_data[index] += d;
                                // }
                                // else{
                                // decp_data[next_vertex] = input_data[next_vertex] - bound;}
                                
                            }
                            else{
                                decp_data[next_vertex] = input_data[next_vertex] - bound;
                                // if(index == 6595 and count_f_min==5){cout<<"在这里3！"<<abs(input_data[next_vertex]-bound-decp_data[next_vertex])<< endl;}
                            }
                            
                            
                        
                        
                };

                }
                
                

                
            }

            else{
                
                if(decp_data[index]<decp_data[next_vertex]){
                    // if(next_vertex==339928 and wrong_maxi_cp.size()==84){
                    //     cout<<"np下降："<<endl;
                    //     cout<<"next: "<<next_vertex<<", "<<decp_data[next_vertex]<<endl;
                    //     cout<<"index: "<<index<<", "<<decp_data[index]<<endl;
                    //     cout<<"daxiaoguanxi: "<<decp_data[index]-decp_data[next_vertex]<<endl;
                    //     cout<<"diff: "<<diff<<endl;
                    //     cout<<"d: "<<d<<endl;
                
                    //     }
                        
                        // if(abs(input_data[index]-(decp_data[next_vertex]))<=bound and abs(input_data[next_vertex]-decp_data[index])<=bound){
                        //     float t = decp_data[index];
                        //     decp_data[index] = decp_data[next_vertex];
                        //     if(next_vertex==66783){cout<<"在这里14"<<endl;}
                        //     decp_data[next_vertex] = t;
                            
                        // }
                        if(abs(input_data[next_vertex]-decp_data[index])<bound){
                            float t = (decp_data[index]-(input_data[index]-bound))/2.0;
                            // if(index==949999){cout<<"在这里24"<<endl;}
                            // decp_data[index] = decp_data[next_vertex];
                            // if(next_vertex==66783){cout<<"在这里14"<<endl;}
                            decp_data[next_vertex] = decp_data[index]-t;
                            
                        }
                        else{
                            // if(index==949999){cout<<"在这里29"<<endl;}
                            decp_data[index] = input_data[index] + bound;
                        }
                }
                
                else if(decp_data[index]==decp_data[next_vertex]){
                    float d = (bound - (input_data[index]-decp_data[index]))/2.0;
                    // while(abs(input_data[index]-decp_data[index]-d)>bound and d>=2e-16){
                    //         d/=2;
                    // }
                    // if(index==949999){
                    //     cout<<"在这里99 "<<d<<endl;
                    // }   
                    // float d = 1e-6;
                    if(abs(input_data[index]-decp_data[index]-d)<=bound){
                        decp_data[index]+=d;
                    }
                    else if(abs(input_data[next_vertex]-decp_data[next_vertex]+d)<=bound){
                        // if(next_vertex==66783){cout<<"在这里13"<<endl;}
                        decp_data[next_vertex]-=d;
                    }
                }
            }
            

            
            
            
        // if(index == 6595 and count_f_min==5){
        //         cout<<"下降后："<<endl;
        //         cout<<"next: "<<next_vertex<<", "<<decp_data[next_vertex]<<endl;
        //         cout<<"index: "<<index<<", "<<decp_data[index]<<endl;
        //         cout<<"daxiaoguanxi: "<<decp_data[index]-decp_data[next_vertex]<<endl;
        //         cout<<"diff: "<<diff<<endl;
        //         cout<<"d: "<<d<<endl;
        //         cout<<"d1: "<<d1<<endl;
        //         cout<<input_data[index]<<","<<input_data[next_vertex]<<endl;
        //     }
            
        
        }
    
        else{
            // find_direction2(0,index);
            int largest_index = from_direction_to_index1(index,de_direction_ds[index]);
            float diff = (bound-(input_data[index]-decp_data[index]))/2.0;
            float d = (bound-(input_data[largest_index]-decp_data[index]))/2.0;
            // if(count_f_min==84){
            //     cout<<"np下降："<<endl;
            //     cout<<"next: "<<largest_index<<", "<<decp_data[largest_index]<<endl;
            //     cout<<"index: "<<index<<", "<<decp_data[index]<<endl;
            //     cout<<"daxiaoguanxi: "<<decp_data[index]-decp_data[largest_index]<<endl;
            //     cout<<"diff: "<<diff<<endl;
            //     cout<<"d: "<<d<<endl;
                
            // }
            if(count_f_max==0 and count_f_min==4 and index==6655977){
                 printf("改变前");
                 printf("%d, %f\n", index, decp_data[index]);
                 printf("%d %f\n", largest_index, decp_data[largest_index]);
                 printf("%f %f \n",diff, d);
                 printf("%f %f \n",decp_data[index], decp_data[index]-d);
                 printf("%f %f \n",input_data[index], input_data[largest_index]);
            //     // cout<<"index: "<<index<<", "<<decp_data[index]<<endl;
            //     // cout<<"next_vertex: "<<next_vertex<<","<<decp_data[next_vertex]<<endl;
            //     // cout<<"smallest_vertex: "<<smallest_vertex<<", "<<decp_data[smallest_vertex]<<endl;
            //     // cout<<"diff: "<<d<<","<<"d: "<<d<<endl;
             }
            if(decp_data[index]<decp_data[largest_index] or (decp_data[index]==decp_data[largest_index] and index<largest_index)){
                de_direction_ds[index] = -1;
                return;
            }
            
            if (diff>=1e-6){
                if (decp_data[index]>=decp_data[largest_index]){
                    while(abs(input_data[index]-decp_data[index]+diff)>bound and diff>=2e-6){
                        diff/=2;
                    }
                    
                    
                    if(abs(input_data[index]-decp_data[index]+diff)<=bound){
                        // if(index==999973){
                        //     cout<<"在这里2！"<<endl;
                        // }
                        
                        decp_data[index] -= diff;
                    }
                    
                    
                }                    
            }
            
                    
            else{
                if (decp_data[index]>=decp_data[largest_index]){
                    
                    // if(index==66783){cout<<"在这里15"<<endl;}
                    decp_data[index] = input_data[index] - bound;
                    if(count_f_max==0 and count_f_min==4 and index==6655977){
                        printf("改变后");
                        // printf("%d, %f\n", index, decp_data[index]);
                        printf("%d %f\n", largest_index, decp_data[largest_index]);
                        
                        // printf("%f %f \n",diff, d);
                        // printf("%f %f \n",decp_data[index], decp_data[index]-d);
                    //     // cout<<"index: "<<index<<", "<<decp_data[index]<<endl;
                    //     // cout<<"next_vertex: "<<next_vertex<<","<<decp_data[next_vertex]<<endl;
                    //     // cout<<"smallest_vertex: "<<smallest_vertex<<", "<<decp_data[smallest_vertex]<<endl;
                    //     // cout<<"diff: "<<d<<","<<"d: "<<d<<endl;
                    }
                }   
    
            }


               
        }

        
    }    
    return;
}

__global__ void addKernel(int* globalVar) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    // printf("hello word from thefind gpu!\n");
    if(i<=20){
        atomicAdd(globalVar, 1);
    }
    
}


// void init_or_data(std::vector<int> *a, std::vector<int> *b, std::vector<int> *c, std::vector<int> *d, std::vector<float> *input_data1, std::vector<float> *decp_data1, int num){
    
//     int* temp;
    
//     int* temp1;
//     float* temp3;
//     int* tempd;
//     int* tempd1;
//     float* temp5;

//     cudaMalloc(&temp, num * sizeof(int));
//     cudaMalloc(&temp1, num * sizeof(int));
//     cudaMalloc(&tempd, num * sizeof(int));
//     cudaMalloc(&tempd1, num * sizeof(int));
//     cudaMalloc(&temp3, num * sizeof(float));
//     cudaMalloc((void**)&temp5, num * sizeof(float));
    



//     cudaMemcpy(temp, a->data(), num * sizeof(int), cudaMemcpyHostToDevice);
//     cudaMemcpy(temp1, b->data(), num * sizeof(int), cudaMemcpyHostToDevice);
//     cudaMemcpy(tempd, c->data(), num * sizeof(int), cudaMemcpyHostToDevice);
//     cudaMemcpy(tempd1, d->data(), num * sizeof(int), cudaMemcpyHostToDevice);
//     cudaMemcpy(temp3, input_data1->data(), num * sizeof(float), cudaMemcpyHostToDevice);
//     cudaMemcpy(temp5, decp_data1->data(), num * sizeof(float), cudaMemcpyHostToDevice);

//     cudaMemcpyToSymbol(or_maxi, &temp, sizeof(int*));
//     cudaMemcpyToSymbol(or_mini, &temp1, sizeof(int*));
//     cudaMemcpyToSymbol(de_direction_as, &tempd, sizeof(int*));
//     cudaMemcpyToSymbol(de_direction_ds, &tempd1, sizeof(int*));
//     cudaMemcpyToSymbol(input_data, &temp3, sizeof(float*));
//     cudaMemcpyToSymbol(decp_data, &temp5, sizeof(float*));
//     cudaDeviceSynchronize();
    

//     dim3 blockSize(1000);
    
//     dim3 gridSize((num + blockSize.x - 1) / blockSize.x);f
    
//     int* tempDevicePtr = nullptr;
//     size_t arraySize = num*6; // 确定所需的大小
//     cudaError_t cudaStatus = cudaMalloc(&tempDevicePtr, arraySize * sizeof(int));
    
//     cudaStatus = cudaMemcpyToSymbol(adjacency, &tempDevicePtr, sizeof(tempDevicePtr));
    
//     computeAdjacency<<<gridSize, blockSize>>>(num,100,100,6);

//     cudaDeviceSynchronize();
    


    
//     iscriticle<<<gridSize,blockSize>>>(num);

    
//     cudaDeviceSynchronize();

    
    
//     int host_count_f_max;
//     cudaMemcpyFromSymbol(&host_count_f_max, count_f_max, sizeof(int), 0, cudaMemcpyDeviceToHost);
    
//     int host_count_f_min;
//     cudaMemcpyFromSymbol(&host_count_f_min, count_f_min, sizeof(int), 0, cudaMemcpyDeviceToHost);
//     // cout<<host_count_f_max<<endl;
//     while(host_count_f_max>0 or host_count_f_min>0){
        
//         // cout<<host_count_f_max<<", "<<host_count_f_min<<endl;
//         dim3 blockSize1(1000);
//         dim3 gridSize1((host_count_f_max + blockSize1.x - 1) / blockSize1.x);
        
//         fix_maxi_critical1<<<gridSize1, blockSize1>>>(0);
//         cudaDeviceSynchronize();

//         dim3 blocknum(1000);
//         dim3 gridnum((host_count_f_min + blocknum.x - 1) / blocknum.x);
        
        
//         fix_maxi_critical1<<<gridnum, blocknum>>>(1);
//         cudaDeviceSynchronize();
//         // 重新检查错误cp个数
//         int initialValue = 0;
//         cudaStatus = cudaMemcpyToSymbol(count_f_max, &initialValue, sizeof(int));
//         if (cudaStatus != cudaSuccess) {
//             std::cerr << "cudaMemcpyToSymbol failed1: " << cudaGetErrorString(cudaStatus) << std::endl;
//         }
//         // int initialValue = 0;
//         cudaStatus = cudaMemcpyToSymbol(count_f_min, &initialValue, sizeof(int));
       
//         if (cudaStatus != cudaSuccess) {
//             std::cerr << "cudaMemcpyToSymbol failed2: " << cudaGetErrorString(cudaStatus) << std::endl;
//         }

//         iscriticle<<<gridSize, blockSize>>>(num);
        
//         cudaMemcpyFromSymbol(&host_count_f_max, count_f_max, sizeof(int), 0, cudaMemcpyDeviceToHost);
        
//         cudaMemcpyFromSymbol(&host_count_f_min, count_f_min, sizeof(int), 0, cudaMemcpyDeviceToHost);
//         // cout<<host_count_f_max<<", "<<host_count_f_min<<endl;
//         cudaDeviceSynchronize();
        
        
//     }
    
    
//     cudaStatus = cudaMemcpy(decp_data1->data(), temp5, num * sizeof(float), cudaMemcpyDeviceToHost);

//     if (cudaStatus != cudaSuccess) {
//             std::cerr << "cudaMemcpyToSymbol failed3: " << cudaGetErrorString(cudaStatus) << std::endl;
//     }
//     cudaDeviceSynchronize();
    
    
//     cudaFree(temp);
//     cudaFree(temp1);
//     cudaFree(temp3);
//     cudaFree(temp5);
    
    
    

    
//     return;
    
// }

void init_inputdata(std::vector<int> *a,std::vector<int> *b,std::vector<int> *c,std::vector<int> *d,std::vector<float> *input_data1,std::vector<float> *decp_data1,int width1, int height1, int depth1, std::vector<int> *low,float bound1){
    int* temp;
    
    int* temp1;
    int* d_data;
    
    

    float* temp3;
    float* temp4;
    
    int num1 = width1*height1*depth1;


    

    // cout<<num1<<endl;
    // size_t size = num1 * sizeof(int);
    hipMemcpyToSymbol(HIP_SYMBOL(width), &width1, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(height), &height1, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(depth), &depth1, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(num), &num1, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(bound), &bound1, sizeof(float), 0, hipMemcpyHostToDevice);

    hipMalloc(&temp, num1 * sizeof(int));
    hipMalloc(&temp1, num1 * sizeof(int));
    hipMalloc(&temp3, num1 * 2 * sizeof(float));
    hipMalloc(&temp4, num1 * 2 * sizeof(float));
    hipMalloc(&d_data, num1 * sizeof(int));
    
    hipMemcpy(temp3, input_data1->data(), num1 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(temp4, decp_data1->data(), num1 * sizeof(float), hipMemcpyHostToDevice);
    
    hipMemcpy(d_data, low->data(), num1 * sizeof(int), hipMemcpyHostToDevice);

    int *d_temp;  // 用于在主机端临时存储设备内存地址
    size_t size = num1 * sizeof(int);

    // 为设备端数组分配内存
    hipMalloc(&d_temp, size);

    // 将设备端内存地址复制到设备端全局指针
    hipMemcpyToSymbol(HIP_SYMBOL(all_max), &d_temp, sizeof(int*));
    hipMemcpyToSymbol(HIP_SYMBOL(lowgradientindices), &d_data, sizeof(int*));
    int *d_temp1;  // 用于在主机端临时存储设备内存地址
    size_t size1 = num1 * sizeof(int);

    // 为设备端数组分配内存
    hipMalloc(&d_temp1, size1);

    // 将设备端内存地址复制到设备端全局指针
    hipMemcpyToSymbol(HIP_SYMBOL(all_min), &d_temp1, sizeof(int*));

    int *d_temp2;  // 用于在主机端临时存储设备内存地址
    size_t size4 = num1 * sizeof(int);
    // 为设备端数组分配内存
    hipMalloc(&d_temp2, size4);

    // 将设备端内存地址复制到设备端全局指针
    hipMemcpyToSymbol(HIP_SYMBOL(de_direction_as), &d_temp2, sizeof(int*));

    int *d_temp3;  // 用于在主机端临时存储设备内存地址
    size_t size3 = num1 * sizeof(int);

    // 为设备端数组分配内存
    hipMalloc(&d_temp3, size3);

    // 将设备端内存地址复制到设备端全局指针
    hipMemcpyToSymbol(HIP_SYMBOL(de_direction_ds), &d_temp3, sizeof(int*));
    
    hipError_t cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(or_maxi), &temp, sizeof(int*));
    if (cudaStatus != hipSuccess) {
            std::cerr << "hipMemcpyToSymbol failed80: " << hipGetErrorString(cudaStatus) << std::endl;
        }
    cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(or_mini), &temp1, sizeof(int*));
    if (cudaStatus != hipSuccess) {
            std::cerr << "hipMemcpyToSymbol failed80: " << hipGetErrorString(cudaStatus) << std::endl;
        }
    hipMemcpyToSymbol(HIP_SYMBOL(input_data), &temp3, sizeof(float*));
    hipMemcpyToSymbol(HIP_SYMBOL(decp_data), &temp4, sizeof(float*));
    hipDeviceSynchronize();
    

    dim3 blockSize(1024);
    
    dim3 gridSize((num1 + blockSize.x - 1) / blockSize.x);
    
    int* tempDevicePtr = nullptr;
    size_t arraySize = num1*12; // 确定所需的大小
    cudaStatus = hipMalloc(&tempDevicePtr, arraySize * sizeof(int));
    
    cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(adjacency), &tempDevicePtr, sizeof(tempDevicePtr));
    if (cudaStatus != hipSuccess) {
            std::cerr << "hipMemcpyToSymbol failed80: " << hipGetErrorString(cudaStatus) << std::endl;
        }

    computeAdjacency<<<gridSize, blockSize>>>();
    // cout<<"出发"<<endl;
    find_direction<<<gridSize, blockSize>>>(1);
    find_direction<<<gridSize, blockSize>>>();
    // cout<<"出发"<<endl;
    hipDeviceSynchronize();
    
    hipMemcpy(a->data(), temp, num1 * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(b->data(), temp1, num1 * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(c->data(), d_temp2, num1 * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(d->data(), d_temp3, num1 * sizeof(int), hipMemcpyDeviceToHost);
    // cudaFree(temp);
    // cudaFree(temp1);
    // cudaFree(temp3);
    // cudaFree(tempDevicePtr);
    
    return;
}
__global__ void copyDeviceVarToDeviceMem(int *deviceMem,int *deviceMem1) {
    if (threadIdx.x == 0) {  // 只在一个线程上执行
        *deviceMem = *de_direction_as;
        *deviceMem1 = *de_direction_ds;
    }
}
__global__ void getlabel(int *label, int *un_sign_ds, int *un_sign_as, int type=0){
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int *direction_as;
    int *direction_ds;
    // if(type==0){
    //     printf("%d\n", i*2+1);
    // }
    if(i>=num or lowgradientindices[i]==1){
        return;
    }
    if(type==0){
        direction_as = de_direction_as;
        direction_ds = de_direction_ds;
    }
    else{
        direction_as = or_maxi;
        direction_ds = or_mini;
    }
    
    int cur = label[i*2+1];
    
    int next_vertex;
    
    if (cur!=-1 and direction_as[cur]!=-1){
        // printf("%d\n", cur);
        int direc = direction_as[cur];
        
        next_vertex = from_direction_to_index1(cur, direc);
        // switch (direc) {
        //     case 1:
        //         next_vertex = (row)*width + (rank1-1);
        //         break;
        //     case 2:
        //         next_vertex = (row-1)*width + (rank1);
        //         break;
        //     case 3:
        //         next_vertex = (row-1)*width + (rank1+1);
        //         break;
        //     case 4:
        //         next_vertex = (row)*width + (rank1+1);
        //         break;
        //     case 5:
        //         next_vertex = (row+1)*width + (rank1);
        //         break;
        //     case 6:
        //         next_vertex = (row+1)*width + (rank1-1);
        //         break;
        // };

        
        // 检查他的下一步是不是extrema以及他的下一步是否已经converge
        
        if (direction_as[next_vertex] != -1 and direction_as[label[next_vertex*2+1]] != -1){
            // if(i==7072){
            //     printf("%d %d %d %d %d\n",next_vertex);
            // }
            *un_sign_as+=1;
        }
        
        if(direction_as[i]!=-1){

            if(label[next_vertex*2+1]!=-1 and direction_as[label[next_vertex*2+1]]==-1){
                label[i*2+1] = label[next_vertex*2+1];
            }
            else{

                label[i*2+1] = next_vertex;
            }
            
            
        }
        else{
            label[i*2+1] = -1;
        };
        // if(i==66590){
        //     printf("%d %d %d %d %d\n",next_vertex,de_direction_as[next_vertex],de_direction_as[label[next_vertex*2+1]],label[next_vertex*2+1],label[i*2+1]);
        // }
        
    }

    
    
    cur = label[i*2];
    int next_vertex1;
    
    
    if (cur!=-1 and direction_ds[cur]!=-1){
        
        int direc = direction_ds[cur];
            
        // int row = (cur% (height * width)) / width;
        // int rank1 = cur%width;
        // int depth1 = cur/(width * height);
        
        next_vertex1 = from_direction_to_index1(cur, direc);

        
        // 下一步不是extrema或者下一步没有converge，检查下一步的label是不是extrema
        if (direction_ds[next_vertex1] != -1 and direction_ds[label[next_vertex1*2]] != -1){
            
            *un_sign_ds+=1;
            // printf("%d \n",de_direction_ds[label[next_vertex1*2]]);
            // printf("%d \n",i);
        }
        // 更新当前的点。
        // 检查当前点是不是extrema
        if(direction_ds[i]!=-1){
            // 如果不是extrema，如果下一步已经收敛，那就把标签替换为下一步的标签。
            // 首先要检查下一步是不是extrema
            if(label[next_vertex1*2]!=-1 and direction_ds[label[next_vertex1*2]]==-1){
                label[i*2] = label[next_vertex1*2];
            }
            else{
                label[i*2] = next_vertex1;
            }
            
            
        }
        else{
            label[i*2] = -1;
        };
        
        
    }

    

}

void fix_process(std::vector<int> *c,std::vector<int> *d,std::vector<float> *decp_data1){
    auto total_start2 = std::chrono::high_resolution_clock::now();
    int num1;
    hipMemcpyFromSymbol(&num1, HIP_SYMBOL(num), sizeof(int), 0, hipMemcpyDeviceToHost);
    
    float* temp5;
    float time = 0.0;
    hipError_t cudaStatus = hipMalloc((void**)&temp5, num1 * sizeof(float));
    auto start2 = std::chrono::high_resolution_clock::now();
    cudaStatus = hipMemcpy(temp5, decp_data1->data(), num1 * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
            std::cerr << "hipMemcpyToSymbol failed7: " << hipGetErrorString(cudaStatus) << std::endl;
    }
    hipDeviceSynchronize();
    auto end2 = std::chrono::high_resolution_clock::now();
    auto duration2 = std::chrono::duration_cast<std::chrono::microseconds>(end2 - start2);
    time+=duration2.count();
    cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(decp_data), &temp5, sizeof(float*));
    if (cudaStatus != hipSuccess) {
            std::cerr << "hipMemcpyToSymbol failed73: " << hipGetErrorString(cudaStatus) << std::endl;
    }
   
    
    
    
    

    hipDeviceSynchronize();
    

    
    
    int* hostArray;
    cudaStatus = hipMalloc((void**)&hostArray, num1 * sizeof(int));
    if (cudaStatus != hipSuccess) {
            std::cerr << "hipMemcpyToSymbol failed70: " << hipGetErrorString(cudaStatus) << std::endl;
        }
    
    // 从设备内存复制数据到主机内存
    hipMemcpyToSymbol(HIP_SYMBOL(de_direction_as), &hostArray, sizeof(int*));
    
    int* hostArray1;

    
    cudaStatus = hipMalloc((void**)&hostArray1, num1 * sizeof(int));
    if (cudaStatus != hipSuccess) {
            std::cerr << "hipMemcpyToSymbol failed71: " << hipGetErrorString(cudaStatus) << std::endl;
        }
    cudaStatus =  hipMemcpyToSymbol(HIP_SYMBOL(de_direction_ds), &hostArray1, sizeof(int*));
    if (cudaStatus != hipSuccess) {
            std::cerr << "hipMemcpyToSymbol failed72: " << hipGetErrorString(cudaStatus) << std::endl;
    }
    
    hipDeviceSynchronize();
    

    dim3 blockSize(1024);
    
    dim3 gridSize((num1 + blockSize.x - 1) / blockSize.x);
    find_direction<<<gridSize,blockSize>>>();
    
    iscriticle<<<gridSize,blockSize>>>();
    // return;
    
    hipDeviceSynchronize();

    
    
    int host_count_f_max;
    cudaStatus = hipMemcpyFromSymbol(&host_count_f_max, HIP_SYMBOL(count_f_max), sizeof(int), 0, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipMemcpyToSymbol failed11: " << hipGetErrorString(cudaStatus) << std::endl;
    }
    int host_count_f_min;
    cudaStatus = hipMemcpyFromSymbol(&host_count_f_min, HIP_SYMBOL(count_f_min), sizeof(int), 0, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
            std::cerr << "hipMemcpyToSymbol failed12: " << hipGetErrorString(cudaStatus) << std::endl;
    }
    // cout<<host_count_f_max<<", "<<host_count_f_min<<num1<<endl;
    // return;
    float elapsedTime;
    float wholeTime;
    float totalElapsedTime = 0.0;
    float totalElapsedTime_fcp = 0.0;
    float totalElapsedTime_fd = 0.0;
    hipEvent_t start, stop;
    hipEvent_t start1;
    hipEventCreate(&start);
    hipEventCreate(&start1);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    hipEventRecord(start1, 0);
    // cout<<"wrong: "<<(host_count_f_max+host_count_f_min)/num1<<endl;
    while(host_count_f_max>0 or host_count_f_min>0){
        
        cout<<host_count_f_max<<", "<<host_count_f_min<<endl;

        
        dim3 blockSize1(1024);
        dim3 gridSize1((host_count_f_max + blockSize1.x - 1) / blockSize1.x);
        // cudaEventRecord(start, 0);
        fix_maxi_critical1<<<gridSize1, blockSize1>>>(0);
        // cudaDeviceSynchronize();
        // cudaDeviceSynchronize();

        dim3 blocknum(1024);
        dim3 gridnum((host_count_f_min + blocknum.x - 1) / blocknum.x);
        
        
        fix_maxi_critical1<<<gridnum, blocknum>>>(1);
        // cout<<"wanc"<<endl;
        hipDeviceSynchronize();
        
        // 重新检查错误cp个数
        int initialValue = 0;
        cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(count_f_max), &initialValue, sizeof(int));
        // if (cudaStatus != cudaSuccess) {
        //     std::cerr << "cudaMemcpyToSymbol failed4: " << cudaGetErrorString(cudaStatus) << std::endl;
        // }
        // int initialValue = 0;
        cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(count_f_min), &initialValue, sizeof(int));

        // if (cudaStatus != cudaSuccess) {
        //     std::cerr << "cudaMemcpyToSymbol failed5: " << cudaGetErrorString(cudaStatus) << std::endl;
        // }
        // cudaEventRecord(stop, 0);
        // cudaEventSynchronize(stop);

        // 计算这次迭代的时间并加到总时间上
        // cudaEventElapsedTime(&elapsedTime, start, stop);
        // totalElapsedTime+=elapsedTime;
        // std::cout << "Average Time Per Iteration = " << elapsedTime << " ms" << std::endl;
        // cudaEventRecord(start, 0);

        iscriticle<<<gridSize, blockSize>>>();
        // cudaEventRecord(stop, 0);
        // cudaEventSynchronize(stop);

        // 计算这次迭代的时间并加到总时间上
        // cudaEventElapsedTime(&elapsedTime, start, stop);
        // totalElapsedTime_fcp+=elapsedTime;

        // cudaEventRecord(start, 0);
        find_direction<<<gridSize,blockSize>>>();
        
        // cudaEventRecord(stop, 0);
        // cudaEventSynchronize(stop);
        // cudaEventElapsedTime(&elapsedTime, start, stop);
        // totalElapsedTime_fd+=elapsedTime;
        // 计算这次迭代的时间并加到总时间上
        
        
        hipMemcpyFromSymbol(&host_count_f_max, HIP_SYMBOL(count_f_max), sizeof(int), 0, hipMemcpyDeviceToHost);
        
        hipMemcpyFromSymbol(&host_count_f_min, HIP_SYMBOL(count_f_min), sizeof(int), 0, hipMemcpyDeviceToHost);
        // cout<<host_count_f_max<<", "<<host_count_f_min<<endl;
        hipDeviceSynchronize();
        
        // exit(0);
    }
    // cudaEventRecord(stop, 0);
    // cudaEventSynchronize(stop);
    
    // cudaEventRecord(start, 0);
    find_direction<<<gridSize,blockSize>>>();
    // cudaEventRecord(stop, 0);
    // cudaEventSynchronize(stop);
    // cudaEventElapsedTime(&elapsedTime, start, stop);
    // totalElapsedTime_fd+=elapsedTime;
    // cudaEventElapsedTime(&wholeTime, start1, stop);
    // cout<<"["<<totalElapsedTime/wholeTime<<", "<<totalElapsedTime_fcp/wholeTime<<", "<<totalElapsedTime_fd/wholeTime<<"],"<<endl;;
    start2 = std::chrono::high_resolution_clock::now();
    cudaStatus = hipMemcpy(decp_data1->data(), temp5, num1 * sizeof(float), hipMemcpyDeviceToHost);

    
    hipDeviceSynchronize();

    


    

    // 从设备内存复制数据到主机内存
    // cudaMemcpy(hostArray1, de_direction_ds, num1 * sizeof(int), cudaMemcpyDeviceToHost);
    hipMemcpy(c->data(), hostArray, num1 * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(d->data(), hostArray1, num1 * sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    end2 = std::chrono::high_resolution_clock::now();

    // Calculate duration
    duration2 = std::chrono::duration_cast<std::chrono::microseconds>(end2 - start2);
    time+=duration2.count();
    // delete[] hostArray;
    // delete[] hostArray1;
    // delete[] temp5;
    hipFree(temp5);
    hipFree(hostArray);
    hipFree(hostArray1);
    // cudaFree(num1);
    
    end2 = std::chrono::high_resolution_clock::now();
    duration2 = std::chrono::duration_cast<std::chrono::microseconds>(end2 - total_start2);
    // printf("%f, ",time/duration2.count());
    

    return;
    
}

__global__ void copyDeviceToArray(int* hostArray,int* hostArray1) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < num) {
        
        hostArray[index] = de_direction_as[index];
        
        hostArray1[index] = de_direction_ds[index];
    }
    
}


__global__ void initializeWithIndex(int* label, int size, int type=0) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size) {
        int *direction_ds;
        int *direction_as;
        if(type==0){
            direction_ds = de_direction_ds;
            direction_as = de_direction_as;
        }
        else{
            
            direction_ds = or_mini;
            direction_as = or_maxi;
        
        }

        if(direction_ds[index]!=-1){
            label[index*2] = index;
        }
        else{
            label[index*2] = -1;
        }

        if(direction_as[index]!=-1){
            label[index*2+1] = index;
        }
        else{
            label[index*2+1] = -1;
        }
    }
}

void mappath1(std::vector<int> *label, std::vector<int> *direction_as, std::vector<int> *direction_ds, int type=0){
    int num1;
    hipMemcpyFromSymbol(&num1, HIP_SYMBOL(num), sizeof(int), 0, hipMemcpyDeviceToHost);
    
    int *un_sign_as;
    hipMalloc((void**)&un_sign_as, sizeof(int));
    hipMemset(un_sign_as, 0, sizeof(int));

    int *un_sign_ds;
    hipMalloc((void**)&un_sign_ds, sizeof(int));
    hipMemset(un_sign_ds, 0, sizeof(int));

    
    
    
    dim3 blockSize1(1024);
    dim3 gridSize1((num1 + blockSize1.x - 1) / blockSize1.x);

    int* label_temp;
    hipError_t cudaStatus = hipMalloc((void**)&label_temp, num1*2 * sizeof(int));
    if (cudaStatus != hipSuccess) {
            std::cerr << "hipMemcpyToSymbol failed60: " << hipGetErrorString(cudaStatus) << std::endl;
    }
    
    
    
    int h_un_sign_as = num1;
    int h_un_sign_ds = num1;
    // int *un_sign_as = 0;
    // int *un_sign_ds = 0;
    int* hostArray;
    cudaStatus = hipMalloc((void**)&hostArray, num1 * sizeof(int));
    // cout<<num1<<"大小"<<endl;
    // cudaMemcpy(decp_data1->data(), temp5, num1 * sizeof(float), cudaMemcpyDeviceToHost);
    
    cudaStatus = hipMemcpy(hostArray,direction_as->data(), num1 * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
            std::cerr << "hipMemcpyToSymbol failed76: " << hipGetErrorString(cudaStatus) << std::endl;
    }

    int* hostArray1;
    cudaStatus = hipMalloc((void**)&hostArray1, num1 * sizeof(int));
    cudaStatus = hipMemcpy(hostArray1,direction_ds->data(),  num1 * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipMemcpyToSymbol failed78: " << hipGetErrorString(cudaStatus) << std::endl;
    }
    if(type==0){
        // cout<<"集哪里"<<endl;
        
        // 从设备内存复制数据到主机内存
        
        hipMemcpyToSymbol(HIP_SYMBOL(de_direction_as), &hostArray, sizeof(int*));
        
        
        cudaStatus =  hipMemcpyToSymbol(HIP_SYMBOL(de_direction_ds), &hostArray1, sizeof(int*));
        if (cudaStatus != hipSuccess) {
                std::cerr << "hipMemcpyToSymbol failed72: " << hipGetErrorString(cudaStatus) << std::endl;
        }
        
        
    }
    // else{
    //     cudaMemcpyToSymbol(or_maxi, &hostArray, sizeof(int*));
    //     cudaMemcpyToSymbol(or_mini, &hostArray1, sizeof(int*));
    // }
    hipDeviceSynchronize();
    initializeWithIndex<<<gridSize1, blockSize1>>>(label_temp, num1,type);
    hipDeviceSynchronize();
    while(h_un_sign_as>0 or h_un_sign_ds>0){
        // cout<<h_un_sign_as<<","<<h_un_sign_ds<<endl;
        int zero = 0;
        int zero1 = 0;
        hipMemcpy(un_sign_as, &zero, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(un_sign_ds, &zero1, sizeof(int), hipMemcpyHostToDevice);
        getlabel<<<gridSize1,blockSize1>>>(label_temp,un_sign_as,un_sign_ds, type);
        
        hipMemcpy(&h_un_sign_as, un_sign_as, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&h_un_sign_ds, un_sign_ds, sizeof(int), hipMemcpyDeviceToHost);
        // exit(0);
    }   

    hipDeviceSynchronize();

    
    cudaStatus = hipMemcpy(label->data(), label_temp, num1 *2 * sizeof(int), hipMemcpyDeviceToHost);
    
    if (cudaStatus != hipSuccess) {
            std::cerr << "hipMemcpyToSymbol failed61: " << hipGetErrorString(cudaStatus) << std::endl;
    }
    if(type==0){
        hipFree(label_temp);
    }
    
    hipFree(hostArray1);
    hipFree(hostArray);
    return;
};
