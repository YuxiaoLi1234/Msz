#include <iostream>
#include <float.h> 
#include <hipblas.h>

#include <fstream>
#include <sstream>
#include <vector>
#include <cstdlib>
#include <stdio.h>
#include <parallel/algorithm>  
#include <unordered_map>
#include <random>

#include <hip/hip_runtime.h>
#include <string>
#include <omp.h>
#include <unordered_set>
#include <set>
#include <map>
#include <algorithm>
#include <numeric>
#include <utility>
#include <iomanip>
#include <chrono>

using std::count;
using std::cout;
using std::endl;
// nvcc -c kernel2d.cu -o kernel2d.o
// CUDA核函数，执行向量加法
// __device__ std::vector<double> decp_data;
__device__ double* decp_data ;
__device__ int directions1[12] =  {0, -1, -1, 0, -1, 1, 0, 1, 1, 0, 1, -1};
__device__ int width;
__device__ int height;
__device__ int num;
__device__ int* adjacency;
__device__ int* all_max; 
__device__ int* all_min;
__device__ int* unsigned_n;
__device__ int count_max;
__device__ int count_min;
__device__ int count_f_max;
__device__ int count_f_min;

__device__ int* maxi;
__device__ int* mini;
__device__ double bound;
__device__ int* or_maxi;
__device__ int* or_mini;
__device__ int* lowgradientindices;
__device__ double* input_data;
__device__ int* de_direction_as;
__device__ int* de_direction_ds;
__device__ int maxNeighbors = 6;
__device__ int direction_to_index_mapping[6][2] = {
    
    {0, -1},   
    {-1, 0},   
    {-1, 1},  
    {0, 1},    
    {1, 0},    
    {1, -1}   
};

__device__ int getDirection(int x, int y){
    
    for (int i = 0; i < 6; ++i) {
        if (direction_to_index_mapping[i][0] == x && direction_to_index_mapping[i][1] == y) {
            return i+1;  // 返回找到的位置
        }
    }
    return -1;  // 如果未找到，返回 -1


}
__device__ int from_direction_to_index1(int cur, int direc){
    
    if (direc==-1) return cur;
    int row = cur / height;
    int rank1 = cur % height;
    // printf("%d %d\n", row, rank1);
    if (direc > 0 && direc <= 6) {
        int delta_row = direction_to_index_mapping[direc-1][0];
        int delta_col = direction_to_index_mapping[direc-1][1];
        int next_row = row + delta_row;
        int next_col = rank1 + delta_col;
        // printf("%d \n", next_row * width + next_col);
        return next_row * height + next_col;
    }
    else {
        return -1;
    }
    // return 0;
};
__global__ void find_direction (int type=0){
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if(index>=num or lowgradientindices[index]==1){
        return;
    }
    double *data;
    int *direction_as;
    int *direction_ds;

    if(type==0){
        data = decp_data;
        direction_as = de_direction_as;
        direction_ds = de_direction_ds;
    }
    else{
        data = input_data;
        direction_as = or_maxi;
        direction_ds = or_mini;
    }
    
    double mini = 0;
    
    
    // std::vector<int> indexs = adjacency[index];
    int largetst_index = index;
    
    
        
    for(int j =0;j<6;++j){
        int i = adjacency[index*6+j];
        
        if(i==-1){
            continue;
        }
        if(lowgradientindices[i]==1){
            continue;
        }
        if((data[i]>data[largetst_index] or (data[i]==data[largetst_index] and i>largetst_index))){
            mini = data[i]-data[index];
            
            largetst_index = i;
            // }
            
        };
    };
    
    int row_l = largetst_index / height;
    int row_i = index / height;
    
    int row_diff = row_l - row_i;
    int col_diff = (largetst_index % height) - (index % height);
    
    direction_as[index] = getDirection(row_diff, col_diff);
    // if(index==55127 and type==1){
    //     printf("%d %d\n",direction_as[index],or_maxi[index]);
    //     printf("%d %.17f %.17f\n",largetst_index,decp_data[largetst_index],decp_data[index]);
    //     printf("%d %.17f %.17f\n",largetst_index,input_data[largetst_index],input_data[index]);
    // }
    // if(index==1949 and type==1){
    //     printf("%.17f\n",input_data[index]);
    //     for(int i=0;i<6;i++){
    //         int j = adjacency[index*6+i];
    //         if(j==-1) break;
    //         printf("%d, %.17f\n",j ,input_data[j]);
    //     }
    //     printf("%d %d %d\n",largetst_index,row_diff,col_diff);
    //     printf("找方向的时候： %d\n",direction_as[index]);
    // }

    mini = 0;
    largetst_index = index;
    for(int j =0;j<6;++j){
        int i = adjacency[index*6+j];
        
        if(i==-1){
            break;
        }
        if(lowgradientindices[i]==1){
            continue;
        }
        // if(i==8186 and index==8058 and type==0){
        //     printf("%.20f %.20f\n",data[i]-data[index],data[8057]-data[index]);
        //     // cout<<data[i]<<", "<<data[index]<<", "<<data[8057]<<endl;
        // }
        if((data[i]<data[largetst_index] or (data[i]==data[largetst_index] and i<largetst_index))){
            mini = data[i]-data[index];
            
            largetst_index = i;

            
        };
    };
    row_l = largetst_index / height;
    row_i = index / height;
    
    row_diff = row_l - row_i;
    col_diff = (largetst_index % height) - (index % height);
    
    direction_ds[index] = getDirection(row_diff, col_diff);
    
    
    
    
    
    return;

};
__global__ void checkElementKernel(int* array, int size, int target, bool* result) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < size) {
        if (array[idx] == target) {
            *result = true;
        }
    }
}

__global__ void iscriticle(){
        
        int i = threadIdx.x + blockIdx.x * blockDim.x;
        
        if(i>=num or lowgradientindices[i]==1){
            
            return;
        }
        
        bool is_maxima = true;
        bool is_minima = true;
        
        for (int index=0;index<6;index++) {
            int j = adjacency[i*6+index];
            
            if(j==-1){
                break;
            }
                
            if(lowgradientindices[j]==1){
                continue;
            }

            if (decp_data[j] > decp_data[i]) {
                is_maxima = false;
                break;
            }
            else if(decp_data[j] == decp_data[i] and j>i){
                is_maxima = false;
                break;
            }
        }
        for (int index=0;index< 6;index++) {
            int j = adjacency[i*6+index];
            if(j==-1){
                break;
            }
            if(lowgradientindices[j]==1){
                    continue;
            }
            if (decp_data[j] < decp_data[i]) {
                is_minima = false;
                break;
            }
            else if(decp_data[j] == decp_data[i] and j<i){
                is_minima = false;
                break;
            }
        }
        
        
        if((is_maxima && or_maxi[i]!=-1) or (!is_maxima && or_maxi[i]==-1)){
            int idx_fp_max = atomicAdd(&count_f_max, 1);
            // printf("%d \n", count_f_max);
            // if(i==1949){
            //     printf("%d %d\n", is_maxima, or_maxi[i]);
            // }
            all_max[idx_fp_max] = i;
            
        }
        
        else if ((is_minima && or_mini[i]!=-1) or (!is_minima && or_mini[i]==-1)) {
            int idx_fp_min = atomicAdd(&count_f_min, 1);// in one instruction
            
            all_min[idx_fp_min] = i;
            
        } 
        
       
        
}

__global__ void getcp(){
        
        int i = threadIdx.x + blockIdx.x * blockDim.x;
        
        if(i>=num){
            
            return;
        }
        
        bool is_maxima = true;
        bool is_minima = true;
        
        for (int index=0;index<6;index++) {
            int j = adjacency[i*6+index];
            
            if(j==-1){
                
                break;
            }
                
            if (input_data[j] > input_data[i]) {
                
                is_maxima = false;
                
                break;
            }
            else if(input_data[j] == input_data[i] and j>i){
                is_maxima = false;
                break;
            }
        }
        for (int index=0;index< 6;index++) {
            int j = adjacency[i*6+index];
            if(j==-1){
                break;
            }
            if (input_data[j] < input_data[i]) {
                is_minima = false;
                break;
            }
            else if(input_data[j] == input_data[i] and j<i){
                is_minima = false;
                break;
            }
        }
        
        if(is_maxima){
            int idx_fp_max = atomicAdd(&count_max, 1);
            // printf("%d \n", count_f_max);
            or_maxi[idx_fp_max] = i;

        }

        else if(is_minima){
            int idx_fp_min = atomicAdd(&count_min, 1);
            // printf("%d \n", count_f_max);
            or_mini[idx_fp_min] = i;

        }
        
       
        
}

__global__ void freeDeviceMemory() {
    // 释放 decp_data 指向的内存
    if (decp_data != nullptr) {
        delete[] decp_data;
        decp_data = nullptr;  // 避免野指针
    }
} 
__global__ void freeDeviceMemory1() {
    // 释放 decp_data 指向的内存
    if (de_direction_as != nullptr) {
        delete[] de_direction_as;
        de_direction_as = nullptr;  // 避免野指针
    }
    if (de_direction_ds != nullptr) {
        delete[] de_direction_ds;
        de_direction_ds = nullptr;  // 避免野指针
    }
}
__global__ void computeAdjacency() {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i < num and lowgradientindices[i]==0) {
        
        int x = i / height; // Get the x coordinate
        int y = i % height; // Get the y coordinate

        int neighborIdx = 0;
        
        for (int d = 0; d < 6; d++) {
            
            int dirX = directions1[d * 2];     
            int dirY = directions1[d * 2 + 1]; 
            
            int newX = x + dirX;
            int newY = y + dirY;
            int r = newX * height + newY; // Calculate the index of the adjacent vertex

            if (newX >= 0 && newX < width && newY >= 0 && newY < height && r < num && lowgradientindices[r]==0) {
                
                adjacency[i * maxNeighbors + neighborIdx] = r;
                neighborIdx++;

            }
        }

        // Fill the remaining slots with -1 or another placeholder value
        
        for (int j = neighborIdx; j < maxNeighbors; ++j) {
            adjacency[i * maxNeighbors + j] = -1;
        }
    }
}

__global__ void allocateDeviceMemory() {
    if (threadIdx.x == 0) { // 仅在一个线程上执行
        // printf("%d %d \n", threadIdx.x,num );
        all_max = new int[num];
        
        all_min = new int[num];
    }
    return;
}

__global__ void fix_maxi_critical1(int direction){
    int index_f = blockIdx.x * blockDim.x + threadIdx.x;
    
    
        
    
    if (direction == 0 && index_f<count_f_max && lowgradientindices[all_max[index_f]]==0){
        
        int index = all_max[index_f];
        // printf("%d\n",index);
        
        
        if (or_maxi[index]!=-1){
            // printf("%d\n",index);
            // find_direction2(1,index);
            int next_vertex = from_direction_to_index1(index,or_maxi[index]);
            
            int smallest_vertex = next_vertex;
            double threshold = -DBL_MAX;
            
            
            for(int j=0;j<6;j++){
                int i = adjacency[index*6+j];
                if(i==-1){
                    continue;
                }
                if(lowgradientindices[i]==1){
                    continue;
                }
                
                if(input_data[i]<input_data[index] and input_data[i]>threshold and i!=next_vertex){
                    smallest_vertex = i;
                    threshold = input_data[i];
                }
            }
            
            threshold = decp_data[smallest_vertex];
            // 对的
            // d是把index还要降低
            // 如果是tthresh的话，那它的下限就是：input_data[index]-(abs(inaput_data[index]-decp_data[index]))
            // 之前的
            // double diff = (bound - (input_data[next_vertex]-decp_data[index]))/2.0;
            // double d = (decp_data[index] - input_data[index] + bound )/2.0;
            // // double d = (decp_data[index]-(input_data[index]-(abs(input_data[index]-decp_data_copy[index]))))/2.0;
            // double d1 = ((input_data[next_vertex] + bound) - decp_data[next_vertex])/2.0;
            // double diff1 = (bound - (input_data[next_vertex]-decp_data[index]))/2.0;
            double diff = (bound - (input_data[next_vertex]-decp_data[index]))/2.0;
            double d = (decp_data[index] - input_data[index] + bound )/2.0;
            double d1 = ((input_data[next_vertex] + bound) - decp_data[next_vertex])/2.0;
            double diff1 = (bound - (input_data[next_vertex]-decp_data[index]))/2.0;
            // double diff = ((input_data[next_vertex]-decp_data_copy[next_vertex]) - (input_data[next_vertex]-decp_data[index]))/2.0;
            // double d = (decp_data[index] - input_data[index] + (input_data[index]-decp_data_copy[index]))/2.0;
            // double d1 = ((input_data[next_vertex] + (input_data[next_vertex]-decp_data_copy[next_vertex])) - decp_data[next_vertex])/2.0;
            // double diff1 = ((input_data[next_vertex]-decp_data_copy[next_vertex]) - (input_data[next_vertex]-decp_data[index]))/2.0;
            // if(count_f_max==1){
            //     printf("改变后");
            //     printf("%d, %.17lf\n", index, decp_data[index]);
            //     printf("%d %.17lf\n", next_vertex, decp_data[next_vertex]);
            //     printf("%.17lf %.17lf \n",d1, d);
            //     printf("%.17lf %.17lf \n",input_data[index], input_data[next_vertex]);
            //     // cout<<"index: "<<index<<", "<<decp_data[index]<<endl;
            //     // cout<<"next_vertex: "<<next_vertex<<","<<decp_data[next_vertex]<<endl;
            //     // cout<<"smallest_vertex: "<<smallest_vertex<<", "<<decp_data[smallest_vertex]<<endl;
            //     // cout<<"diff: "<<d<<","<<"d: "<<d<<endl;
            // }
            if(decp_data[index]<decp_data[next_vertex] or (decp_data[index]==decp_data[next_vertex] and index<next_vertex)){
                
                de_direction_as[index]=or_maxi[index];
            
                return;
            }
            
            if(d>=1e-16 ){
                
                if(decp_data[index]==decp_data[next_vertex])
                    {
                        
                        while(abs(input_data[index]-decp_data[index]+d)>bound and d>=2e-16){
                            d/=2;
                        }
                        if (abs(input_data[index]-decp_data[index]+d)<=bound){
                            decp_data[index] -= d;
                        }
                    }
                else{
                    if(decp_data[index]>=decp_data[next_vertex]){
                        
                        while(abs(input_data[index]-decp_data[index]+d)>bound and d>=2e-16){
                                d/=2;
                        }
                        
                        if(decp_data[index]>=threshold and threshold<=decp_data[next_vertex]){
                            
                            while(decp_data[index] - d < threshold and d>=2e-16)
                            {
                                d/=2;
                            }
                            
                            
                        }
                        // else if(threshold>decp_data[next_vertex]){
                            
                            
                        //     double diff2 = (bound-(input_data[smallest_vertex]-decp_data[smallest_vertex]))/64;
                            
                        //     if(diff2>=1e-16){
                        //         while(abs(input_data[smallest_vertex]-decp_data[smallest_vertex]+diff2)>bound and diff2>=2e-16 and decp_data[smallest_vertex]-diff2>decp_data[next_vertex]){
                                    
                        //             diff2/=2;
                        //         }
                                
                        //         if(abs(input_data[smallest_vertex]-decp_data[smallest_vertex]+diff2)<=bound){
                        //             if(smallest_vertex==66783){cout<<"在这里11."<<endl;}
                        //             decp_data[smallest_vertex]-=diff2;
                        //             // if(index==97) cout<<"处理97的时候: "<<decp_data[next_vertex]<<", "<<decp_data[index]<<endl;
                        //         }
                                
                                
                        //     }
                            
                        // }

                        if(abs(input_data[index]-(decp_data[index]-d))<=bound and decp_data[index]>=decp_data[next_vertex] and d>=1e-16){
                            // if(index==1620477){
                            //     // cout<<"next_vertex: "<<decp_data[next_vertex]<<endl;
                            //     // cout<<"smallest_vertex: "<<decp_data[smallest_vertex]<<endl;
                            //     cout<<"before index: "<<decp_data[index]<<endl;
                                
                            // }
                            
                            decp_data[index] -= d;
                            
                            
                                            
                        }
                        // else if(abs(input_data[next_vertex]-(decp_data[next_vertex]+d1))<=bound and decp_data[index]>=decp_data[next_vertex] and d1>0){
                        //     // if(index==1620477){
                        //     //     // cout<<"next_vertex: "<<decp_data[next_vertex]<<endl;
                        //     //     // cout<<"smallest_vertex: "<<decp_data[smallest_vertex]<<endl;
                        //     //     cout<<"before index: "<<decp_data[index]<<endl;
                                
                        //     // }
                            
                        //     decp_data[next_vertex] += d1;
                            
                            
                                            
                        // }
                        
                        // if(count_f_max==1){
                        //     printf("改变后dd");
                        //     printf("%d, %.17lf\n", index, decp_data[index]);
                        //     printf("%d %.17lf\n", next_vertex, decp_data[next_vertex]);
                        //     printf("%.17lf %.17lf \n",d1, d);
                        //     printf("%.17lf %.17lf \n",input_data[index], input_data[next_vertex]);
                        //     // cout<<"index: "<<index<<", "<<decp_data[index]<<endl;
                        //     // cout<<"next_vertex: "<<next_vertex<<","<<decp_data[next_vertex]<<endl;
                        //     // cout<<"smallest_vertex: "<<smallest_vertex<<", "<<decp_data[smallest_vertex]<<endl;
                        //     // cout<<"diff: "<<d<<","<<"d: "<<d<<endl;
                        // }
                   
                };
                     }
            
                 
            
                
            }
            else{
                
                if(decp_data[index]>decp_data[next_vertex]){
                    double t = (decp_data[next_vertex]-(input_data[index]-bound))/2.0;
                    if(abs(input_data[index]-decp_data[next_vertex]+t)<=bound and t>=1e-16){
                            
                            
                            decp_data[index] = decp_data[next_vertex] - t;
                            // decp_data[next_vertex] = t;
                        }
                    else{
                        
                        decp_data[index] = input_data[index] - bound;
                        
                    }
                    // if(count_f_max==1){
                    //         printf("改变后dd");
                    //         printf("%d, %.17lf, %.17lf\n", index, decp_data[index],input_data[index]-bound);
                    //         printf("%d %.17lf\n", next_vertex, decp_data[next_vertex]);
                    //         printf("%.17lf %.17lf \n",d1, d);
                    //         printf("%.17lf %.17lf \n",input_data[index], input_data[next_vertex]);
                    //         // cout<<"index: "<<index<<", "<<decp_data[index]<<endl;
                    //         // cout<<"next_vertex: "<<next_vertex<<","<<decp_data[next_vertex]<<endl;
                    //         // cout<<"smallest_vertex: "<<smallest_vertex<<", "<<decp_data[smallest_vertex]<<endl;
                    //         // cout<<"diff: "<<d<<","<<"d: "<<d<<endl;
                    //     }
                }
                else if(decp_data[index]==decp_data[next_vertex]){
                    // double bound1 = abs(input_data[index]-decp_data[index]);
                    //
                    double d = (bound - (input_data[index]-decp_data[index]))/2.0;
                    double d1 = (bound - (input_data[next_vertex]-decp_data[next_vertex]))/2.0;
                    // while(abs(input_data[index]-decp_data[index]-d)>bound and d>=2e-16){
                    //         d/=2;
                    // }
                    // if(index==157569){
                    //     cout<<"在这时候d: "<<d<<endl;
                    // }   
                    // double d = 1e-16;
                    if(abs(input_data[index]-decp_data[index]+d)<=bound){
                        
                        decp_data[index]-=d;
                    }
                    
                    else if(abs(input_data[next_vertex]-decp_data[next_vertex]-d1)<=bound){
                        // if(next_vertex==78){cout<<"在这里21"<<endl;}
                        decp_data[next_vertex]+=d1;
                    }
                }
                
            }
            
            
        
        }
        else{
            // if(index==25026 and count_f_max<=770){
            //     cout<<"在这里"<<endl;
            // }
            // find_direction2(0,index);
            int largest_index = from_direction_to_index1(index,de_direction_as[index]);
            // 对的
            double diff = (bound-(input_data[index]-decp_data[index]))/2.0;
            double d = (bound-(input_data[largest_index]-decp_data[index]))/2.0;
            // double diff = (input_data[index]-decp_data[index])/2.0;
            // double d = (input_data[largest_index]-decp_data[index])/2.0;
            // double d1 = ((input_data[next_vertex] + (input_data[next_vertex]-decp_data_copy[next_vertex])) - decp_data[next_vertex])/2.0;
            // double diff1 = ((input_data[next_vertex]-decp_data_copy[next_vertex]) - (input_data[next_vertex]-decp_data[index]))/2.0;
            // if(index==25026 and count_f_max<=770){
            //     cout<<"改变前"<<endl;
            //     cout<<"index: "<<index<<", "<<decp_data[index]<<endl;
            //     cout<<"next_vertex: "<<largest_index<<","<<decp_data[largest_index]<<endl;
            //     // cout<<"smallest_vertex: "<<smallest_vertex<<", "<<decp_data[smallest_vertex]<<endl;
            //     cout<<"diff: "<<d<<","<<"d: "<<d<<endl;
            //     cout<<or_direction_as[25026]<<de_direction_as[25026]<<endl;
            // }
            // if(count_f_max==1 and count_f_min==0){
            //     printf("fp改变后");
            //     printf("%d, %f\n", index, decp_data[index]);
            //     printf("%d %f\n", largest_index, decp_data[largest_index]);
            //     printf("%f %f \n",diff, d);
            //     printf("%.17lf %.17lf \n",input_data[index], input_data[largetst_index]);
            //     // cout<<"index: "<<index<<", "<<decp_data[index]<<endl;
            //     // cout<<"next_vertex: "<<next_vertex<<","<<decp_data[next_vertex]<<endl;
            //     // cout<<"smallest_vertex: "<<smallest_vertex<<", "<<decp_data[smallest_vertex]<<endl;
            //     // cout<<"diff: "<<d<<","<<"d: "<<d<<endl;
            // }
            // if(index==6345199){
            //     printf("改变后");
            //     printf("%d, %f\n", index, decp_data[index]);
            //     printf("%d %f\n",largest_index, decp_data[largest_index]);
            //     printf("%f %f \n",diff, d);
            //     printf("%d %d \n",de_direction_as[index],or_maxi[index]);
            //     // cout<<"index: "<<index<<", "<<decp_data[index]<<endl;
            //     // cout<<"next_vertex: "<<next_vertex<<","<<decp_data[next_vertex]<<endl;
            //     // cout<<"smallest_vertex: "<<smallest_vertex<<", "<<decp_data[smallest_vertex]<<endl;
            //     // cout<<"diff: "<<d<<","<<"d: "<<d<<endl;
            // }
            if(decp_data[index]>decp_data[largest_index] or(decp_data[index]==decp_data[largest_index] and index>largest_index)){
                de_direction_as[index] = -1;
            }
            if(d>=1e-16){
                
                if (decp_data[index]<=decp_data[largest_index]){
                    if(abs(input_data[largest_index]-decp_data[index]+d)){
                        // if(largest_index==66783){cout<<"在这里17"<<endl;}
                        decp_data[largest_index] = decp_data[index]-d;
                    }
                }
                
            
                
            }
            
            else{
                if(decp_data[index]<=decp_data[largest_index]){
                    // if(index==78){
                    //         cout<<"在这里1"<<endl;
                    //     }
                    decp_data[index] = input_data[index] + bound;
                }
                    
            }

            // if(index==15885 and count_f_max==7){
            //     cout<<"改变后"<<endl;
            //     cout<<"index: "<<index<<", "<<decp_data[index]<<endl;
            //     cout<<"next_vertex: "<<largest_index<<","<<decp_data[largest_index]<<endl;
            //     // cout<<"smallest_vertex: "<<smallest_vertex<<", "<<decp_data[smallest_vertex]<<endl;
            //     cout<<"diff: "<<d<<","<<"d: "<<d<<endl;
            // }
            
        }
        
        
    
    }
    
    else if (direction != 0 && index_f<count_f_min && lowgradientindices[all_min[index_f]]==0){
        int index = all_min[index_f];
        if (or_mini[index]!=-1){
            // find_direction2(1,index);
            int next_vertex= from_direction_to_index1(index,or_mini[index]);
            
            double diff = (bound - (input_data[next_vertex]-decp_data[index]))/2.0;
            double d =  (bound+input_data[index]-decp_data[index])/2.0;
            // double d1 =  (bound-(input_data[next_vertex]-decp_data[next_vertex]))/2.0;
            
            double d1 = (decp_data[next_vertex]-input_data[next_vertex]+bound)/2.0;
            if(decp_data[index]>decp_data[next_vertex] or (decp_data[index]==decp_data[next_vertex] and index>next_vertex)){
                de_direction_ds[index]=or_mini[index];
                return;
            }

            // if(index == 6595 and count_f_min==5){
            //     cout<<"下降："<<endl;
            //     cout<<"next: "<<next_vertex<<", "<<decp_data[next_vertex]<<endl;
            //     cout<<"index: "<<index<<", "<<decp_data[index]<<endl;
            //     cout<<"daxiaoguanxi: "<<decp_data[index]-decp_data[next_vertex]<<endl;
            //     cout<<"diff: "<<diff<<endl;
            //     cout<<"d: "<<d<<endl;
            //     cout<<"d1: "<<d1<<endl;
            // }
            
            if(diff>=1e-16){
                
                if(decp_data[index]==decp_data[next_vertex]){
                    
                      
                    
                        while(abs(input_data[next_vertex]-decp_data[index]-diff)>bound and diff>=2e-16){
                            diff/=2;
                        }
                        
                        if(abs(input_data[next_vertex]-decp_data[index]+diff)<=bound and diff>=1e-16){
                            // if(index==344033 and count_f_min==2){cout<<"在这里22"<<d<<endl;}
                            decp_data[next_vertex]= decp_data[index]-diff;
                        }
                        else if(d1>=1e-16){
                            // if(index==344033 and count_f_min==2){cout<<"在这里23"<<d<<endl;}
                            if(abs(input_data[next_vertex]-decp_data[next_vertex]+d1)<=bound){
                                decp_data[next_vertex]-=d1;
                            }
                            
                        }
                        else if(d>=1e-16){
                            // if(index==344033 and count_f_min==2){cout<<"在这里24"<<d<<endl;}
                            if(abs(input_data[index]-decp_data[index]-d)<=bound){
                            decp_data[index]+=d;}
                        }

                    
                    
                }
                else{
                    if(decp_data[index]<=decp_data[next_vertex]){
                        
                            while(abs(input_data[next_vertex]-decp_data[index]+diff)>bound and diff >= 2e-16){
                                    diff/=2;
                            }
                            
                            
                            if (abs(input_data[next_vertex]-decp_data[index]+diff)<=bound and decp_data[index]<=decp_data[next_vertex] and diff>=1e-16){
                                // while(abs(input_data[next_vertex]-decp_data[index]+diff)<=bound and diff<1e-17){
                                //     diff*=2;
                                // }
                                // if(index==270808 and count_f_min==1){cout<<"在这里2！"<< endl;}
                                while(abs(input_data[next_vertex]-decp_data[index]+diff)<bound and diff <= 1e-17){
                                    diff*=2;
                                }
                                if(abs(input_data[next_vertex]-decp_data[index]+diff)<=bound){
                                    decp_data[next_vertex] = decp_data[index]-diff;
                                }
                                // if(index == 6595 and count_f_min==5){
                                //     cout<<"在这里1！"<< diff <<", "<<index<<", "<<decp_data[index]<<","<<input_data[index]<<","<<input_data[next_vertex]<<endl;

                                // }
                                // if(next_vertex==66783){cout<<"在这里13"<<endl;}
                                // decp_data[next_vertex] = decp_data[index]-diff;
                                // if(index==89797){
                                //         cout<<"在这里2"<<diff<<", "<<d<<endl;
                                // }

                                // decp_data[index]+=d;
                            }
                            // else if(abs(input_data[index]-decp_data[index]-d)<=bound and decp_data[index]<=decp_data[next_vertex] and d>0){
                            //     if(index==135569){cout<<"在这里23"<<endl;}
                            //     decp_data[index]+=d;
                            // }
                            else if(abs(input_data[next_vertex]-decp_data[next_vertex]+d1)<=bound and decp_data[index]<=decp_data[next_vertex] and d1>=1e-16){
                                while(abs(input_data[next_vertex]-decp_data[next_vertex]+d1)<bound and d1<=1e-16){
                                    d1*=2;
                                }
                                // if(count_f_min<=12){cout<<"在这里2！"<<abs(input_data[next_vertex]-decp_data[next_vertex]+d1)<<"," <<d1<< endl;}
                                if(abs(input_data[next_vertex]-decp_data[next_vertex]+d1)<=bound and d1>=1e-16){
                                    decp_data[next_vertex]-=d1;
                                }
                                // else{
                                //     decp_data[index] += d;
                                // }
                                // else{
                                // decp_data[next_vertex] = input_data[next_vertex] - bound;}
                                
                            }
                            else{
                                decp_data[next_vertex] = input_data[next_vertex] - bound;
                                // if(index == 6595 and count_f_min==5){cout<<"在这里3！"<<abs(input_data[next_vertex]-bound-decp_data[next_vertex])<< endl;}
                            }
                            
                            
                        
                        
                };

                }
                
                

                
            }

            else{
                
                if(decp_data[index]<decp_data[next_vertex]){
                    // if(next_vertex==339928 and wrong_maxi_cp.size()==84){
                    //     cout<<"np下降："<<endl;
                    //     cout<<"next: "<<next_vertex<<", "<<decp_data[next_vertex]<<endl;
                    //     cout<<"index: "<<index<<", "<<decp_data[index]<<endl;
                    //     cout<<"daxiaoguanxi: "<<decp_data[index]-decp_data[next_vertex]<<endl;
                    //     cout<<"diff: "<<diff<<endl;
                    //     cout<<"d: "<<d<<endl;
                
                    //     }
                        
                        // if(abs(input_data[index]-(decp_data[next_vertex]))<=bound and abs(input_data[next_vertex]-decp_data[index])<=bound){
                        //     double t = decp_data[index];
                        //     decp_data[index] = decp_data[next_vertex];
                        //     if(next_vertex==66783){cout<<"在这里14"<<endl;}
                        //     decp_data[next_vertex] = t;
                            
                        // }
                        double t = (decp_data[index]-(input_data[index]-bound))/2.0;
                        if(abs(input_data[next_vertex]-decp_data[index]+t)<bound and t>=1e-16){
                            
                            // if(index==949999){cout<<"在这里24"<<endl;}
                            // decp_data[index] = decp_data[next_vertex];
                            // if(next_vertex==66783){cout<<"在这里14"<<endl;}
                            decp_data[next_vertex] = decp_data[index]-t;
                            
                        }
                        else{
                            // if(index==949999){cout<<"在这里29"<<endl;}
                            decp_data[index] = input_data[index] + bound;
                        }
                }
                
                else if(decp_data[index]==decp_data[next_vertex]){
                    double d = (bound - (input_data[index]-decp_data[index]))/2.0;
                    // while(abs(input_data[index]-decp_data[index]-d)>bound and d>=2e-16){
                    //         d/=2;
                    // }
                    // if(index==949999){
                    //     cout<<"在这里99 "<<d<<endl;
                    // }   
                    // double d = 1e-16;
                    if(abs(input_data[index]-decp_data[index]-d)<=bound){
                        decp_data[index]+=d;
                    }
                    else if(abs(input_data[next_vertex]-decp_data[next_vertex]+d)<=bound){
                        // if(next_vertex==66783){cout<<"在这里13"<<endl;}
                        decp_data[next_vertex]-=d;
                    }
                }
            }
            

            
            
            
        // if(count_f_min==1){
        //         printf("fp");
        //         printf("%d, %.17lf\n", index, decp_data[index]);
        //         printf("%d %.17lf\n", next_vertex, decp_data[next_vertex]);
        //         printf("%.17lf %.17lf \n",d1, d);
        //         printf("%.17lf %.17lf \n",input_data[index], input_data[next_vertex]);
        //         // cout<<"index: "<<index<<", "<<decp_data[index]<<endl;
        //         // cout<<"next_vertex: "<<next_vertex<<","<<decp_data[next_vertex]<<endl;
        //         // cout<<"smallest_vertex: "<<smallest_vertex<<", "<<decp_data[smallest_vertex]<<endl;
        //         // cout<<"diff: "<<d<<","<<"d: "<<d<<endl;
        //     }
            
        
        }
    
        else{
            // find_direction2(0,index);
            int largest_index = from_direction_to_index1(index,de_direction_ds[index]);
            double diff = (bound-(input_data[index]-decp_data[index]))/2.0;
            double d = (bound-(input_data[largest_index]-decp_data[index]))/2.0;
            // if(count_f_min==84){
            //     cout<<"np下降："<<endl;
            //     cout<<"next: "<<largest_index<<", "<<decp_data[largest_index]<<endl;
            //     cout<<"index: "<<index<<", "<<decp_data[index]<<endl;
            //     cout<<"daxiaoguanxi: "<<decp_data[index]-decp_data[largest_index]<<endl;
            //     cout<<"diff: "<<diff<<endl;
            //     cout<<"d: "<<d<<endl;
                
            // }
            // if(count_f_min==1){
            //     printf("fn\n");
            //     printf("%d, %.17lf\n", index, decp_data[index]);
            //     printf("%d %.17lf\n", largest_index, decp_data[largest_index]);
            //     printf("%.17lf %.17lf \n",d, diff);
            //     printf("%.17lf %.17lf \n",input_data[index], input_data[largest_index]);
            //     // cout<<"index: "<<index<<", "<<decp_data[index]<<endl;
            //     // cout<<"next_vertex: "<<next_vertex<<","<<decp_data[next_vertex]<<endl;
            //     // cout<<"smallest_vertex: "<<smallest_vertex<<", "<<decp_data[smallest_vertex]<<endl;
            //     // cout<<"diff: "<<d<<","<<"d: "<<d<<endl;
            // }
            if(decp_data[index]<decp_data[largest_index] or (decp_data[index]==decp_data[largest_index] and index<largest_index)){
                de_direction_ds[index] = -1;
                return;
            }
            
            if (diff>=1e-16){
                if (decp_data[index]>=decp_data[largest_index]){
                    while(abs(input_data[index]-decp_data[index]+diff)>bound and diff>=2e-16){
                        diff/=2;
                    }
                    
                    
                    if(abs(input_data[index]-decp_data[index]+diff)<=bound){
                        // if(index==999973){
                        //     cout<<"在这里2！"<<endl;
                        // }
                        
                        decp_data[index] -= diff;
                    }
                    
                    
                }                    
            }
            
                    
            else{
                if (decp_data[index]>=decp_data[largest_index]){
                    // printf("%.17lf",bound);
                    // if(index==66783){cout<<"在这里15"<<endl;}
                    decp_data[index] = input_data[index] - bound;
                }   
    
            }


               
        }

        
    }    
    return;
}



__global__ void fix_maxi_critical2(int direction){
    int index_f = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (direction == 0 && index_f<count_f_max){
        
        int index = all_max[index_f];
        
        if (or_maxi[index]!=-1){
            
            int next_vertex = from_direction_to_index1(index,or_maxi[index]);
            
            int smallest_vertex = next_vertex;
            double threshold = -DBL_MAX;
            
            
            for(int j=0;j<6;j++){
                int i = adjacency[index*6+j];
                if(i==-1){
                    break;
                }
                if(input_data[i]<input_data[index] and input_data[i]>threshold and i!=next_vertex){
                    smallest_vertex = i;
                    threshold = input_data[i];
                }
            }
            
            threshold = decp_data[smallest_vertex];
            // double diff = (bound - (input_data[next_vertex]-decp_data[index]))/2.0;
            double d = (bound - (input_data[index]-decp_data[index]))/2.0;
            
            if(decp_data[index]<decp_data[next_vertex] or (decp_data[index]==decp_data[next_vertex] and index<next_vertex)){
                
                de_direction_as[index]=or_maxi[index];
                
                return;
            }
            
            if(d>=1e-16){
                
                if(decp_data[index]==decp_data[next_vertex])
                    {
                        
                        while(abs(input_data[index]-decp_data[index]+d)>bound and d>=2e-16){
                            d/=2;
                        }
                        if (abs(input_data[index]-decp_data[index]+d)<=bound){
                            decp_data[index] -= d;
                        }

                    
                    }
                else{
                    if(decp_data[index]>=decp_data[next_vertex]){
                        
                        while(abs(input_data[index]-decp_data[index]+d)>bound and d>=2e-16){
                                d/=2;
                        }
                        
                        if(decp_data[index]>=threshold and threshold<=decp_data[next_vertex]){
                            
                            while(decp_data[index] - d < threshold and d>=2e-16)
                            {
                                d/=2;
                            }
                            
                            
                        }
                        else if(threshold>decp_data[next_vertex]){
                            
                            
                            double diff2 = (bound-(input_data[smallest_vertex]-decp_data[smallest_vertex]))/2;
                            
                            if(diff2>1e-16){
                                while(abs(input_data[smallest_vertex]-decp_data[smallest_vertex]+diff2)>bound and diff2>=2e-16 and decp_data[smallest_vertex]-diff2>decp_data[next_vertex]){
                                    
                                    diff2/=2;
                                }
                                
                                if(abs(input_data[smallest_vertex]-decp_data[smallest_vertex]+diff2)<=bound){
                                    decp_data[smallest_vertex]-=diff2;
                                    // if(index==97) cout<<"处理97的时候: "<<decp_data[next_vertex]<<", "<<decp_data[index]<<endl;
                                }
                                
                                
                            }
                            
                        }

                        if(abs(input_data[index]-(decp_data[index]-d))<=bound and decp_data[index]>=decp_data[next_vertex]){
                            decp_data[index] -= d;
                            
                        }
                        
                        
                   
                };
                     }

                 
            
                
            }
            else{
                
                if(decp_data[index]>=decp_data[next_vertex]){
                    if(abs(input_data[index]-(input_data[next_vertex] -bound+ decp_data[index])/2.0)<=bound){
                        decp_data[index] = (input_data[next_vertex] -bound + decp_data[index])/2.0;
                    }
                    else{
                        
                        decp_data[index] = input_data[index] - bound;
                    }
                    
                }
                
            }
            
            
        
        }
        else{
            // printf("%d \n",or_maxi[index]);
            int largest_index = from_direction_to_index1(index,de_direction_as[index]);
            // double diff = (bound-(input_data[index]-decp_data[index]))/2.0;
            double d = (bound-(input_data[largest_index]-decp_data[index]))/2.0;
            if(decp_data[index]>decp_data[largest_index] or(decp_data[index]==decp_data[largest_index] and index>largest_index)){
                de_direction_as[index] = -1;
            }
            if(d>=1e-16){
                
                if (decp_data[index]<=decp_data[largest_index]){
                    if(abs(input_data[largest_index]-decp_data[index]+d)){
                        decp_data[largest_index] = decp_data[index]-d;
                    }
                }
                
            
                
            }
            
            else{
                if(decp_data[index]<=decp_data[largest_index]){
                    decp_data[index] = input_data[index] + bound;
                }
                    
            }
            
        }
        
        
    
    }
    
    else if(direction == 1 && index_f<count_f_min){
        int index = all_min[index_f];
        if (or_mini[index]!=-1){
            int next_vertex= from_direction_to_index1(index,or_mini[index]);
            
            double diff = (bound - (input_data[next_vertex]-decp_data[index]))/2.0;
            double d =  (bound-(input_data[index]-decp_data[index]))/2.0;
            
            
            if(decp_data[index]>decp_data[next_vertex] or (decp_data[index]==decp_data[next_vertex] and index>next_vertex)){
                de_direction_ds[index]=or_mini[index];
                return;
            }
            
            if(diff>=1e-16 or d>=1e-16){
                if(decp_data[index]==decp_data[next_vertex]){
                    
                    
                        while(abs(input_data[next_vertex]-decp_data[index]-d)>bound and d>=2e-16){
                            d/=2;
                        }
                        
                        if(abs(input_data[index]-decp_data[index]-d)<=bound){
                            decp_data[index]+=d;
                        }
                    
                    
                    
                    
                }
                else{
                    if(decp_data[index]<=decp_data[next_vertex]){
                        
                            while(abs(input_data[next_vertex]-decp_data[index]+diff)>bound and diff >= 2e-16){
                                    diff/=2;
                            }
                            
                            if (abs(input_data[next_vertex]-decp_data[index]+d)<=bound and decp_data[index]<=decp_data[next_vertex]){
                                decp_data[next_vertex] = decp_data[index]-diff;
                            }
                            
                            
                        
                        
                };

                }
                
                

                
            }

            else{
                
                if(decp_data[index]<=decp_data[next_vertex]){
                    if(abs(input_data[index]-(input_data[next_vertex] + bound + decp_data[index])/2.0)<=bound){
                        decp_data[index] = (input_data[next_vertex] + bound + decp_data[index])/2.0;
                    }
                    else{
                        decp_data[index] = input_data[index] + bound;
                    }
                }
            }
            

            
            
            

            
        
        }
    
        else{
            
            int largest_index = from_direction_to_index1(index,de_direction_ds[index]);
            double diff = (bound-(input_data[index]-decp_data[index]))/2.0;
            // double d = (bound-(input_data[largest_index]-decp_data[index]))/2.0;
            
            if(decp_data[index]<decp_data[largest_index] or (decp_data[index]==decp_data[largest_index] and index<largest_index)){
                de_direction_ds[index] = -1;
                return;
            }
            
            if (diff>=1e-16){
                if (decp_data[index]>=decp_data[largest_index]){
                    while(abs(input_data[index]-decp_data[index]+diff)>bound and diff>=2e-16){
                        diff/=2;
                    }
                    
                    
                    if(abs(input_data[index]-decp_data[index]+diff)<=bound){
                        decp_data[index] -= diff;
                    }
                    
                    
                }                    
            }
            
                    
            else{
                if (decp_data[index]>=decp_data[largest_index]){
                    decp_data[index] = input_data[index] - bound;
                }   
    
            }


               
        }

        
    }    
    return;
};

__global__ void addKernel(int* globalVar) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    // printf("hello word from the gpu!\n");
    if(i<=20){
        atomicAdd(globalVar, 1);
    }
    
}



void init_inputdata(std::vector<int> *a,std::vector<int> *b,std::vector<int> *c,std::vector<int> *d,std::vector<double> *input_data1,std::vector<double> *decp_data1,int width1, int height1,  std::vector<int> *low,double bound1,float &datatransfer,float &finddirection){
    int* temp;
    
    int* temp1;
    int* d_data;
    
    

    double* temp3;
    double* temp4;
    
    int num1 = width1*height1;
    // float datatransfer = 0.0;
    float elapsedTime;
    // float find_direciton = 0.0;
    float getfcp = 0.0;
    cout<<num1<<endl;
    

    // cout<<num1<<endl;
    // size_t size = num1 * sizeof(int);
    

    hipError_t cudaStatus= hipMemcpyToSymbol(HIP_SYMBOL(width), &width1, sizeof(int), 0, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
            std::cerr << "hipMemcpyToSymbol failed101: " << hipGetErrorString(cudaStatus) << std::endl;
        }
    hipMemcpyToSymbol(HIP_SYMBOL(height), &height1, sizeof(int), 0, hipMemcpyHostToDevice);
    // cudaMemcpyToSymbol(depth, &depth1, sizeof(int), 0, cudaMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(num), &num1, sizeof(int), 0, hipMemcpyHostToDevice);
    cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(bound), &bound1, sizeof(double), 0, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
            std::cerr << "hipMemcpyToSymbol failed91: " << hipGetErrorString(cudaStatus) << std::endl;
        }
    
    hipMalloc(&temp, num1 * sizeof(int));
    hipMalloc(&temp1, num1 * sizeof(int));
    cudaStatus =hipMalloc(&temp3, num1  * sizeof(double));
    if (cudaStatus != hipSuccess) {
            std::cerr << "hipMemcpyToSymbol failed89: " << hipGetErrorString(cudaStatus) << std::endl;
        }
    hipMalloc(&temp4, num1  * sizeof(double));
    hipMalloc(&d_data, num1 * sizeof(int));
    
    hipEvent_t start, stop;

    hipEventCreate(&start);

    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    

    cudaStatus = hipMemcpy(temp3, input_data1->data(), num1 * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
            std::cerr << "hipMemcpyToSymbol failed89: " << hipGetErrorString(cudaStatus) << std::endl;
        }
    cudaStatus = hipMemcpy(temp4, decp_data1->data(), num1 * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
            std::cerr << "hipMemcpyToSymbol failed17: " << hipGetErrorString(cudaStatus) << std::endl;
        }
    cudaStatus = hipMemcpy(d_data, low->data(), num1 * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
            std::cerr << "hipMemcpyToSymbol failed27: " << hipGetErrorString(cudaStatus) << std::endl;
        }
    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // 计算这次迭代的时间并加到总时间上
    hipEventElapsedTime(&elapsedTime, start, stop);
    datatransfer+=elapsedTime;
    
    int *d_temp;  // 用于在主机端临时存储设备内存地址
    size_t size = num1 * sizeof(int);

    // 为设备端数组分配内存
    hipMalloc(&d_temp, size);

    // 将设备端内存地址复制到设备端全局指针
    
    hipEventRecord(start, 0);
    hipMemcpyToSymbol(HIP_SYMBOL(all_max), &d_temp, sizeof(int*));
    hipMemcpyToSymbol(HIP_SYMBOL(lowgradientindices), &d_data, sizeof(int*));
    
    int *d_temp1;  // 用于在主机端临时存储设备内存地址
    size_t size1 = num1 * sizeof(int);

    // 为设备端数组分配内存
    hipMalloc(&d_temp1, size1);

    // 将设备端内存地址复制到设备端全局指针
    hipMemcpyToSymbol(HIP_SYMBOL(all_min), &d_temp1, sizeof(int*));

    int *d_temp2;  // 用于在主机端临时存储设备内存地址
    size_t size4 = num1 * sizeof(int);
    // 为设备端数组分配内存
    hipMalloc(&d_temp2, size4);

    // 将设备端内存地址复制到设备端全局指针
    hipMemcpyToSymbol(HIP_SYMBOL(de_direction_as), &d_temp2, sizeof(int*));

    int *d_temp3;  // 用于在主机端临时存储设备内存地址
    size_t size3 = num1 * sizeof(int);

    // 为设备端数组分配内存
    hipMalloc(&d_temp3, size3);

    // 将设备端内存地址复制到设备端全局指针
    cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(de_direction_ds), &d_temp3, sizeof(int*));
    if (cudaStatus != hipSuccess) {
            std::cerr << "hipMemcpyToSymbol failed87: " << hipGetErrorString(cudaStatus) << std::endl;
        }
    cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(or_maxi), &temp, sizeof(int*));
    if (cudaStatus != hipSuccess) {
            std::cerr << "hipMemcpyToSymbol failed83: " << hipGetErrorString(cudaStatus) << std::endl;
        }
    cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(or_mini), &temp1, sizeof(int*));
    if (cudaStatus != hipSuccess) {
            std::cerr << "hipMemcpyToSymbol failed84: " << hipGetErrorString(cudaStatus) << std::endl;
        }
    hipMemcpyToSymbol(HIP_SYMBOL(input_data), &temp3, sizeof(double*));
    hipMemcpyToSymbol(HIP_SYMBOL(decp_data), &temp4, sizeof(double*));
    
    
    dim3 blockSize(1024);
    
    dim3 gridSize((num1 + blockSize.x - 1) / blockSize.x);
    
    int* tempDevicePtr = nullptr;
    size_t arraySize = num1*6; // 确定所需的大小
    cudaStatus = hipMalloc(&tempDevicePtr, arraySize * sizeof(int));
    
    cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(adjacency), &tempDevicePtr, sizeof(tempDevicePtr));
    if (cudaStatus != hipSuccess) {
            std::cerr << "hipMemcpyToSymbol failed81: " << hipGetErrorString(cudaStatus) << std::endl;
        }
    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // 计算这次迭代的时间并加到总时间上
    hipEventElapsedTime(&elapsedTime, start, stop);
    datatransfer+=elapsedTime;
    hipEventRecord(start, 0);
    computeAdjacency<<<gridSize, blockSize>>>();
    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // 计算这次迭代的时间并加到总时间上
    hipEventElapsedTime(&elapsedTime, start, stop);
    cout<<"comupte_adjacency: "<<elapsedTime<<endl;
    // cout<<"出发"<<endl;
    hipEventRecord(start, 0);
    // for(int i =0;i<1000;i++){
    find_direction<<<gridSize, blockSize>>>(1);
    //     cudaEventRecord(stop, 0);
    //     cudaEventSynchronize(stop);
    //     cudaEventElapsedTime(&elapsedTime, start, stop);
    //     cout<<"1次finddirection: "<<elapsedTime<<endl;
    // }
    
    
    // cout<<"1000次finddirection: "<<elapsedTime<<endl;
    find_direction<<<gridSize, blockSize>>>();
    // cout<<"出发"<<endl;
    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // 计算这次迭代的时间并加到总时间上
    hipEventElapsedTime(&elapsedTime, start, stop);
    finddirection+=elapsedTime;
    hipEventRecord(start, 0);
    hipMemcpy(a->data(), temp, num1 * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(b->data(), temp1, num1 * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(c->data(), d_temp2, num1 * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(d->data(), d_temp3, num1 * sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // 计算这次迭代的时间并加到总时间上
    hipEventElapsedTime(&elapsedTime, start, stop);
    datatransfer+=elapsedTime;
    cout<<"data_transfer:"<<datatransfer<<endl;
    cout<<"findd_dierction: "<<find_direction<<endl;
    // cudaFree(temp);
    // cudaFree(temp1);
    // cudaFree(temp3);
    // cudaFree(tempDevicePtr);
    
    return;
}
__global__ void copyDeviceVarToDeviceMem(int *deviceMem,int *deviceMem1) {
    if (threadIdx.x == 0) {  // 只在一个线程上执行
        *deviceMem = *de_direction_as;
        *deviceMem1 = *de_direction_ds;
    }
}
__global__ void getlabel(int *label, int *un_sign_ds, int *un_sign_as, int type=0){
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int *direction_as;
    int *direction_ds;
    
    if(i>=num or lowgradientindices[i]==1){
        // printf("%d\n",num);
        
        return;
    }
    
    if(type==0){
        direction_as = de_direction_as;
        direction_ds = de_direction_ds;
    }
    else{
        direction_as = or_maxi;
        direction_ds = or_mini;
    }
    
    int cur = label[i*2+1];
    
    
    int next_vertex;
    // cur!=-1就说明它首先不是cp，direction_as[cur]也说明他不是cp
    if (cur!=-1 and direction_as[cur]!=-1){
        
        int direc = direction_as[cur];
        // 找到他的下一个邻居
        
        next_vertex = from_direction_to_index1(cur, direc);
        
        // 检查下一个邻居是否为cp，如果是，直接把label换成邻居
        if(label[next_vertex*2+1] == -1){
            label[i*2+1] = next_vertex;
            
        }
        
        else{
            
            label[i*2+1] = label[next_vertex*2+1];
            
            
        }
        
        if (direction_as[label[i*2+1]] != -1){
            *un_sign_as+=1;  
        }
        
    }
    
    
    
    
    cur = label[i*2];
    int next_vertex1;
    
    
    if (cur!=-1 and label[cur*2]!=-1){
        
        int direc = direction_ds[cur];
        // 找到他的下一个邻居
        next_vertex1 = from_direction_to_index1(cur, direc);
        // 检查下一个邻居是否为cp，如果是，直接把label换成邻居
        if(label[next_vertex1*2] == -1){
            label[i*2] = next_vertex1;
            
        }
        // 如果不是cp，检查邻居是否找到cp，如果找到了，就换成邻居的label
        else if(label[label[next_vertex1*2]*2] == -1){
            label[i*2] = label[next_vertex1*2];  
        }
        
        else if(direction_ds[i]!=-1){
            // 如果邻居不是cp，那就替换成邻居的当前邻居
            if(label[next_vertex1*2]!=-1){
                label[i*2] = label[next_vertex1*2];
            }
            // 否则：下一个邻居是cp, 那么他的cp就是下一个邻居
            else{

                label[i*2] = next_vertex1;
            }
            
            
        }
        // if(i==66590){
        //     printf("%d %d %d %d %d\n",next_vertex,de_direction_as[next_vertex],de_direction_as[label[next_vertex*2+1]],label[next_vertex*2+1],label[i*2+1]);
        // }
        if (direction_ds[label[i*2]]!=-1){
            *un_sign_ds+=1;
        }
    } 

}

void fix_process(std::vector<int> *c,std::vector<int> *d,std::vector<double> *decp_data1,float &datatransfer, float &finddirection, float &getfcp, float &fixtime_cp){
    auto total_start2 = std::chrono::high_resolution_clock::now();
    int num1;
    hipMemcpyFromSymbol(&num1, HIP_SYMBOL(num), sizeof(int), 0, hipMemcpyDeviceToHost);
    
    double* temp5;
    float elapsedTime;
    
    hipEvent_t start, stop;
    
    hipEventCreate(&start);
    
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    
    hipError_t cudaStatus = hipMalloc((void**)&temp5, num1 * sizeof(double));
    
    cudaStatus = hipMemcpy(temp5, decp_data1->data(), num1 * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
            std::cerr << "hipMemcpyToSymbol failed7: " << hipGetErrorString(cudaStatus) << std::endl;
    }
    
    
    cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(decp_data), &temp5, sizeof(double*));
    if (cudaStatus != hipSuccess) {
            std::cerr << "hipMemcpyToSymbol failed73: " << hipGetErrorString(cudaStatus) << std::endl;
    }
   
    
    
    
    

    hipDeviceSynchronize();
    

    
    
    int* hostArray;
    cudaStatus = hipMalloc((void**)&hostArray, num1 * sizeof(int));
    if (cudaStatus != hipSuccess) {
            std::cerr << "hipMemcpyToSymbol failed70: " << hipGetErrorString(cudaStatus) << std::endl;
        }
    
    // 从设备内存复制数据到主机内存
    hipMemcpyToSymbol(HIP_SYMBOL(de_direction_as), &hostArray, sizeof(int*));
    
    int* hostArray1;

    
    cudaStatus = hipMalloc((void**)&hostArray1, num1 * sizeof(int));
    if (cudaStatus != hipSuccess) {
            std::cerr << "hipMemcpyToSymbol failed71: " << hipGetErrorString(cudaStatus) << std::endl;
        }
    cudaStatus =  hipMemcpyToSymbol(HIP_SYMBOL(de_direction_ds), &hostArray1, sizeof(int*));
    if (cudaStatus != hipSuccess) {
            std::cerr << "hipMemcpyToSymbol failed72: " << hipGetErrorString(cudaStatus) << std::endl;
    }
    
    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    
    hipEventElapsedTime(&elapsedTime, start, stop);
    datatransfer+=elapsedTime;

    dim3 blockSize(1024);
    dim3 gridSize((num1 + blockSize.x - 1) / blockSize.x);
    hipEventRecord(start, 0);

    find_direction<<<gridSize,blockSize>>>();
    
    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    
    hipEventElapsedTime(&elapsedTime, start, stop);
    // cout<<"1000次finddirection:"<<elapsedTime<<endl;
    
    finddirection+=elapsedTime;

    hipEventRecord(start, 0);
    
    iscriticle<<<gridSize,blockSize>>>();
    
    
    
    hipDeviceSynchronize();

    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    
    hipEventElapsedTime(&elapsedTime, start, stop);
    // cout<<"1000cigetfcp: "<<elapsedTime;
    getfcp+=elapsedTime;
    
    
    int host_count_f_max;
    cudaStatus = hipMemcpyFromSymbol(&host_count_f_max, HIP_SYMBOL(count_f_max), sizeof(int), 0, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipMemcpyToSymbol failed11: " << hipGetErrorString(cudaStatus) << std::endl;
    }
    int host_count_f_min;
    cudaStatus = hipMemcpyFromSymbol(&host_count_f_min, HIP_SYMBOL(count_f_min), sizeof(int), 0, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
            std::cerr << "hipMemcpyToSymbol failed12: " << hipGetErrorString(cudaStatus) << std::endl;
    }
    // cout<<host_count_f_max<<", "<<host_count_f_min<<num1<<endl;
    // return;
    
    
    // cout<<"wrong: "<<(host_count_f_max+host_count_f_min)/num1<<endl;

    while(host_count_f_max>0 or host_count_f_min>0){
        
        // cout<<host_count_f_max<<", "<<host_count_f_min<<endl;

        
        dim3 blockSize1(1024);
        dim3 gridSize1((host_count_f_max + blockSize1.x - 1) / blockSize1.x);
        // cudaEventRecord(start, 0);
        hipEventRecord(start, 0);
        fix_maxi_critical1<<<gridSize1, blockSize1>>>(0);
        
        // cudaDeviceSynchronize();

        dim3 blocknum(1024);
        dim3 gridnum((host_count_f_min + blocknum.x - 1) / blocknum.x);
        
        
        fix_maxi_critical1<<<gridnum, blocknum>>>(1);
        // cout<<"wanc"<<endl;
        hipDeviceSynchronize();
        
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);

        // 计算这次迭代的时间并加到总时间上
        hipEventElapsedTime(&elapsedTime, start, stop);
        fixtime_cp+=elapsedTime;
        // 重新检查错误cp个数
        int initialValue = 0;
        cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(count_f_max), &initialValue, sizeof(int));
        // if (cudaStatus != cudaSuccess) {
        //     std::cerr << "cudaMemcpyToSymbol failed4: " << cudaGetErrorString(cudaStatus) << std::endl;
        // }
        // int initialValue = 0;
        cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(count_f_min), &initialValue, sizeof(int));

        // if (cudaStatus != cudaSuccess) {
         //     std::cerr << "cudaMemcpyToSymbol failed5: " << cudaGetErrorString(cudaStatus) << std::endl;
        // }
        
        // std::cout << "Average Time Per Iteration = " << elapsedTime << " ms" << std::endl;
        hipEventRecord(start, 0);

        iscriticle<<<gridSize, blockSize>>>();
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);

        // 计算这次迭代的时间并加到总时间上
        hipEventElapsedTime(&elapsedTime, start, stop);
        getfcp+=elapsedTime;

        hipEventRecord(start, 0);
        find_direction<<<gridSize,blockSize>>>();
        
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsedTime, start, stop);
        finddirection+=elapsedTime;
        // 计算这次迭代的时间并加到总时间上
        
        
        hipMemcpyFromSymbol(&host_count_f_max, HIP_SYMBOL(count_f_max), sizeof(int), 0, hipMemcpyDeviceToHost);
        
        hipMemcpyFromSymbol(&host_count_f_min, HIP_SYMBOL(count_f_min), sizeof(int), 0, hipMemcpyDeviceToHost);
        // cout<<host_count_f_max<<", "<<host_count_f_min<<endl;
        hipDeviceSynchronize();
        
        // exit(0);
    }
    // cudaEventRecord(stop, 0);
    // cudaEventSynchronize(stop);
    
    hipEventRecord(start, 0);
    find_direction<<<gridSize,blockSize>>>();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    // finddirection+=elapsedTime;
    // cudaEventElapsedTime(&wholeTime, start1, stop);
    // cout<<"["<<totalElapsedTime/wholeTime<<", "<<totalElapsedTime_fcp/wholeTime<<", "<<totalElapsedTime_fd/wholeTime<<"],"<<endl;;
    // start2 = std::chrono::high_resolution_clock::now();
    hipEventRecord(start, 0);
    cudaStatus = hipMemcpy(decp_data1->data(), temp5, num1 * sizeof(double), hipMemcpyDeviceToHost);
    

    


    

    
    // cudaMemcpy(hostArray1, de_direction_ds, num1 * sizeof(int), cudaMemcpyDeviceToHost);
    hipMemcpy(c->data(), hostArray, num1 * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(d->data(), hostArray1, num1 * sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    datatransfer+=elapsedTime;
    
    hipDeviceSynchronize();
    
    // delete[] hostArray;
    // delete[] hostArray1;
    // delete[] temp5;
    hipFree(temp5);
    hipFree(hostArray);
    hipFree(hostArray1);
    // cudaFree(num1);
    
    
    // printf("%f, ",time/duration2.count());
    

    return;
    
}

__global__ void copyDeviceToArray(int* hostArray,int* hostArray1) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < num) {
        
        hostArray[index] = de_direction_as[index];
        
        hostArray1[index] = de_direction_ds[index];
    }
    
}


__global__ void initializeWithIndex(int* label, int size, int type=0) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size) {
        int *direction_ds;
        int *direction_as;
        if(type==0){
            direction_ds = de_direction_ds;
            direction_as = de_direction_as;
        }
        else{
            
            direction_ds = or_mini;
            direction_as = or_maxi;
        
        }

        if(direction_ds[index]!=-1){
            label[index*2] = index;
        }
        else{
            label[index*2] = -1;
        }

        if(direction_as[index]!=-1){
            label[index*2+1] = index;
        }
        else{
            label[index*2+1] = -1;
        }
    }
}


void mappath1(std::vector<int> *label, std::vector<int> *direction_as, std::vector<int> *direction_ds, float &finddirection, float &mappath_path, float &datatransfer,int type=0){
    int num1;
    hipMemcpyFromSymbol(&num1, HIP_SYMBOL(num), sizeof(int), 0, hipMemcpyDeviceToHost);
    
    int *un_sign_as;
    hipMalloc((void**)&un_sign_as, sizeof(int));
    hipMemset(un_sign_as, 0, sizeof(int));

    int *un_sign_ds;
    hipMalloc((void**)&un_sign_ds, sizeof(int));
    hipMemset(un_sign_ds, 0, sizeof(int));

    
    
    
    dim3 blockSize1(1024);
    dim3 gridSize1((num1 + blockSize1.x - 1) / blockSize1.x);

    float elapsedTime;
    
    hipEvent_t start, stop;

    hipEventCreate(&start);
    
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    
    int* label_temp;
    hipError_t cudaStatus = hipMalloc((void**)&label_temp, num1*2 * sizeof(int));
    if (cudaStatus != hipSuccess) {
            std::cerr << "hipMemcpyToSymbol failed60: " << hipGetErrorString(cudaStatus) << std::endl;
    }
    
    
    
    int h_un_sign_as = num1;
    int h_un_sign_ds = num1;
    // int *un_sign_as = 0;
    // int *un_sign_ds = 0;
    int* hostArray;
    cudaStatus = hipMalloc((void**)&hostArray, num1 * sizeof(int));
    // cout<<num1<<"大小"<<endl;
    // cudaMemcpy(decp_data1->data(), temp5, num1 * sizeof(double), cudaMemcpyDeviceToHost);
    
    cudaStatus = hipMemcpy(hostArray,direction_as->data(), num1 * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
            std::cerr << "hipMemcpyToSymbol failed76: " << hipGetErrorString(cudaStatus) << std::endl;
    }

    int* hostArray1;
    cudaStatus = hipMalloc((void**)&hostArray1, num1 * sizeof(int));
    cudaStatus = hipMemcpy(hostArray1,direction_ds->data(),  num1 * sizeof(int), hipMemcpyHostToDevice);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    datatransfer+=elapsedTime;

    if (cudaStatus != hipSuccess) {
        std::cerr << "hipMemcpyToSymbol failed78: " << hipGetErrorString(cudaStatus) << std::endl;
    }
    if(type==0){
        // cout<<"集哪里"<<endl;
        
        // 从设备内存复制数据到主机内存
        hipEventRecord(start, 0);
        hipMemcpyToSymbol(HIP_SYMBOL(de_direction_as), &hostArray, sizeof(int*));
        
        
        cudaStatus =  hipMemcpyToSymbol(HIP_SYMBOL(de_direction_ds), &hostArray1, sizeof(int*));
        if (cudaStatus != hipSuccess) {
                std::cerr << "hipMemcpyToSymbol failed72: " << hipGetErrorString(cudaStatus) << std::endl;
        }
        
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsedTime, start, stop);
        datatransfer+=elapsedTime;
        
    }
    hipEventRecord(start, 0);
    // for(int i=0;i<1000;i++){
    initializeWithIndex<<<gridSize1, blockSize1>>>(label_temp, num1,type);
    hipDeviceSynchronize();
    
    // h_un_sign_as = num1;
    while(h_un_sign_as>0 or h_un_sign_ds>0){
        
        int zero = 0;
        int zero1 = 0;

        // cout<<"找path"<<h_un_sign_as<<", "<<h_un_sign_ds<<endl;
        hipMemcpy(un_sign_as, &zero, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(un_sign_ds, &zero1, sizeof(int), hipMemcpyHostToDevice);
        getlabel<<<gridSize1,blockSize1>>>(label_temp,un_sign_as,un_sign_ds,type);
        
        hipMemcpy(&h_un_sign_as, un_sign_as, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&h_un_sign_ds, un_sign_ds, sizeof(int), hipMemcpyDeviceToHost);
        // exit(0);
        
        
    }   
        


    //     cudaDeviceSynchronize();
    // }
    hipDeviceSynchronize();
    

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    // cout<<"1000cimappath:"<<elapsedTime<<endl;
    mappath_path+=elapsedTime;

    hipEventRecord(start, 0);
    cudaStatus = hipMemcpy(label->data(), label_temp, num1 *2 * sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    datatransfer+=elapsedTime;
    if (cudaStatus != hipSuccess) {
            std::cerr << "hipMemcpyToSymbol failed61: " << hipGetErrorString(cudaStatus) << std::endl;
    }
    if(type==0){
        hipFree(label_temp);
        
    }
    
    hipFree(hostArray1);
    hipFree(hostArray);
    
    
    return;
};
